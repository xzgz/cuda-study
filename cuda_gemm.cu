#include "hip/hip_runtime.h"

struct DefaultMma<float, LayoutA, kAlignmentA, float, LayoutB,
                  kAlignmentB, float, layout::RowMajor,
                  arch::OpClassTensorOp, ArchTag, ThreadblockShape, WarpShape,
                  InstructionShape, 2, Operator, false, SharedMemoryClearOption::kNone,
                  GatherA, GatherB> {
  // Define the MmaCore components
  using MmaCore = typename cutlass::gemm::threadblock::DefaultMmaCore<
      ThreadblockShape, WarpShape, InstructionShape, float, LayoutA, float,
      LayoutB, float, layout::RowMajor, arch::OpClassTensorOp, 2,
      arch::OpMultiplyAddFastF16>;

  // Define iterators over tiles from the A operand
  using IteratorA =
      cutlass::transform::threadblock::PredicatedTileIterator<
          cutlass::MatrixShape<MmaCore::Shape::kM, MmaCore::Shape::kK>,
          float, LayoutA, 1, typename MmaCore::IteratorThreadMapA, kAlignmentA, GatherA>;

  // Define iterators over tiles from the B operand
  using IteratorB =
      cutlass::transform::threadblock::PredicatedTileIterator<
          cutlass::MatrixShape<MmaCore::Shape::kK, MmaCore::Shape::kN>,
          float, LayoutB, 0, typename MmaCore::IteratorThreadMapB, kAlignmentB, GatherB>;

  // Define the threadblock-scoped pipelined matrix multiply
  using ThreadblockMma = cutlass::gemm::threadblock::MmaPipelined<
      typename MmaCore::Shape, IteratorA, typename MmaCore::SmemIteratorA,
      IteratorB, typename MmaCore::SmemIteratorB, float,
      layout::RowMajor, typename MmaCore::MmaPolicy>;
};

template<typename WarpShape>
constexpr int simt_get_warp_threads_m() {
    return (WarpShape::kM > WarpShape::kN) ? 8 : 4;
}


cutlass::gemm::threadblock::DefaultMmaCore:

  // Define the warp-level op
  static const int WarpNumThreadsM = detail::simt_get_warp_threads_m<WarpShape>();
  static const int WarpNumThreadsN = kWarpSize / WarpNumThreadsM;
  static const int ThreadTileM = WarpShape::kM / WarpNumThreadsM;
  static const int ThreadTileN = WarpShape::kN / WarpNumThreadsN;
  static_assert(!(WarpShape::kM % WarpNumThreadsM) && !(WarpShape::kN % WarpNumThreadsN),
      "WarpShape must be divisible by ThreadTile shape.");
  static const int LaneLayout = ThreadTileM > 4 && ThreadTileN > 4 ? 2 : 1;
  static const int numElementsA = 128 / sizeof_bits<ElementA>::value;
  static const int numElementsB = 128 / sizeof_bits<ElementB>::value;
  static const int LaneM = cutlass::const_min(numElementsA, ThreadTileM);
  static const int LaneN = cutlass::const_min(numElementsB, ThreadTileN);

  static int const kPaddingM = detail::simt_transpose_padding(kWarpSize, Shape::kK, sizeof_bits<ElementA>::value);

  static_assert(!(kPaddingM % LaneM),
                "Padding must be divisible by Lane");

  // these should have max of thread tile also
  using LaneMmaShape = cutlass::gemm::GemmShape<
      LaneM,
      LaneN,
      1>;
  using Policy = cutlass::gemm::warp::MmaSimtPolicy<
      cutlass::MatrixShape<WarpNumThreadsM, WarpNumThreadsN>,   // WarpShape
      cutlass::layout::RowMajorInterleaved<LaneLayout>,         // LaneLayout
      LaneMmaShape
  >;

  using MmaWarpSimt = cutlass::gemm::warp::MmaSimt<
      WarpShape,    /// Size of the Gemm problem - concept: gemm::GemmShape<> 128, 128, 8
      ElementA,     /// Data type of A elements
      SmemLayoutA,  /// Layout of A matrix (concept: MatrixLayout)
      ElementB,     /// Data type of B elements
      SmemLayoutB,  /// Layout of B matrix (concept: MatrixLayout)
      ElementC,     /// Element type of C matrix
      LayoutC,      /// Layout of C matrix (concept: MatrixLayout)
      Policy        /// Policy describing warp-level MmaSimtOp (concept: MmaSimtOp policy)
  >;

  /// Policy used to define MmaPipelined 
  using MmaPolicy = MmaPolicy<
    MmaWarpSimt,
    MatrixShape<kPaddingM, 0>,    // skew for A matrix to avoid SMEM bank conflicts
    MatrixShape<0, 0>,
    WarpCount::kK
  >;
};


cutlass::gemm::warp::MmaSimt:

  /// Iterates over the A operand in memory
  using IteratorA = MmaSimtTileIterator<
    MatrixShape<Shape::kM, Policy::LaneMmaShape::kK>,
    Operand::kA,
    ElementA,
    LayoutA,
    Policy,
    PartitionsK,
    Shape::kK
  >;

  /// Storage for A tile
  using FragmentA = typename IteratorA::Fragment;

  /// Storage for transformed A tile
  using TransformedFragmentA = FragmentA;

  /// Iterates over the B operand in memory
  using IteratorB = MmaSimtTileIterator<
    MatrixShape<Policy::LaneMmaShape::kK, Shape::kN>,
    Operand::kB,
    ElementB,
    LayoutB,
    Policy,
    PartitionsK,
    Shape::kK
  >;

  /// Storage for B tile
  using FragmentB = typename IteratorB::Fragment;


MmaSimtTileIterator:

/// Operand tag
  static Operand const kOperand = Operand::kB;

  static_assert(!(Shape::kColumn % Policy::WarpShape::kColumn), 
    "The warp-level GEMM N size must be divisible by the number of threads arranged along the N dimension.");
  
  static_assert(Shape::kRow > 0, "Shape::kRow must be greater than zero.");
  static_assert(Shape::kColumn > 0, "Shape::kColumn must be greater than zero.");
  static_assert(Policy::WarpShape::kColumn > 0, "Policy::WarpShape::kColumn must be greater than zero.");
  static_assert(Shape::kColumn / Policy::WarpShape::kColumn > 0, "Shape::kColumn / Policy::WarpShape::kColumn must be greater than zero.");

  /// Thread-level shape of a fragment
  using ThreadShape = MatrixShape<
    Shape::kRow,
    Shape::kColumn / Policy::WarpShape::kColumn
  >;

  static_assert(!(ThreadShape::kColumn % Policy::LaneMmaShape::kN), 
    "Thread-level GEMM must be divisible by Policy::LaneMmaShape.");

  /// Number of individual loads
  using Iterations = MatrixShape<
    ThreadShape::kRow,
    ThreadShape::kColumn / Policy::LaneMmaShape::kN
  >;

  /// Fragment object holding a thread's part of a tile
  using Fragment = Array<Element, ThreadShape::kCount>;



  /// Operand tag
  static Operand const kOperand = Operand::kA;

  static_assert(!(Shape::kRow % Policy::WarpShape::kRow), 
    "The warp-level GEMM M size must be divisible by the number of threads arranged along the M dimension.");

  static_assert(Shape::kRow > 0, "Shape::kRow must be greater than zero.");
  static_assert(Shape::kColumn > 0, "Shape::kColumn must be greater than zero.");
  static_assert(Policy::WarpShape::kRow > 0, "Policy::WarpShape::kRow must be greater than zero.");
  static_assert(Shape::kRow / Policy::WarpShape::kRow > 0, "Shape::kRow / Policy::WarpShape::kRow must be greater than zero.");

  /// Thread-level shape of a fragment
  using ThreadShape = MatrixShape<
    Shape::kRow / Policy::WarpShape::kRow,
    Shape::kColumn
  >;

  static_assert(!(ThreadShape::kRow % Policy::LaneMmaShape::kM), 
    "Thread-level GEMM must be divisible by Policy::LaneMmaShape.");

  /// Number of individual loads (scalar loads)
  using Iterations = MatrixShape<
    ThreadShape::kRow / Policy::LaneMmaShape::kM,
    ThreadShape::kColumn
  >;

  /// Fragment object holding a thread's part of a tile
  using Fragment = Array<Element, ThreadShape::kCount>;



