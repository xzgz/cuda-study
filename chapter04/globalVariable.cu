#include "hip/hip_runtime.h"
#include <stdio.h>
// includes CUDA Runtime
#include <hip/hip_runtime.h>

#include "../include/common/common.h"

/*
 * An example of using a statically declared global variable (devData) to store
 * a floating-point value on the device.
 */

__device__ float devData;

__global__ void checkGlobalVariable()
{
  // display the original value
  printf("Device: the value of the global variable is %f\n", devData);

  // alter the value
  devData += 2.0f;
}

__global__ void checkGlobalMemoryVariable(float *dptr)
{
  // display the original value
  printf("Device: The value of the global memory variable is %f\n", *dptr);

  // alter the value
  *dptr += 2.0f;
}

int main(void)
{
  // initialize the global variable
  float value = 3.14f;
  CHECK(hipMemcpyToSymbol(HIP_SYMBOL(devData), &value, sizeof(float)));
  printf("Host:   copied %f to the global variable\n", value);
  printf("value address: %p\n", &value);
  printf("devData address: %p\n", &devData);

  // invoke the kernel
  checkGlobalVariable<<<1, 1>>>();

  // copy the global variable back to the host
  CHECK(hipMemcpyFromSymbol(&value, HIP_SYMBOL(devData), sizeof(float)));
  printf("Host:   the value changed by the checkGlobalVariable kernel to %f\n", value);

  value = 6;
  float *dptr = NULL;
  printf("value address: %p\n", &value);
  printf("devData address: %p\n", &devData);
  printf("dptr address: %p\n", dptr);
  CHECK(hipGetSymbolAddress((void **)&dptr, devData));
  CHECK(hipMemcpy((void *)dptr, &value, sizeof(float), hipMemcpyHostToDevice));
  printf("Host:   copied %f to the global memory variable\n", value);
  printf("value address: %p\n", &value);
  printf("devData address: %p\n", &devData);
  printf("dptr address: %p\n", dptr);

  // invoke the kernel
  checkGlobalMemoryVariable<<<1, 1>>>(dptr);

  // copy the global variable back to the host
  CHECK(hipMemcpy((void *)&value, dptr, sizeof(float), hipMemcpyDeviceToHost));
  printf("Host:   the value changed by the checkGlobalMemoryVariable kernel to %f\n", value);

  // invoke the kernel
  checkGlobalVariable<<<1, 1>>>();

  // copy the global variable back to the host
  CHECK(hipMemcpy((void *)&value, dptr, sizeof(float), hipMemcpyDeviceToHost));
  printf("Host:   the value changed by the checkGlobalVariable kernel to %f\n", value);
  printf("value address: %p\n", &value);
  printf("devData address: %p\n", &devData);
  printf("dptr address: %p\n", dptr);

  CHECK(hipDeviceReset());
  return EXIT_SUCCESS;
}
