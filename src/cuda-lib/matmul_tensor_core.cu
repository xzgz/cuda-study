#include "hip/hip_runtime.h"
#include "matmul_kernel.h"

using namespace nvcuda;

// The only dimensions currently supported by WMMA
const int WMMA_M = 16;
const int WMMA_N = 16;
const int WMMA_K = 16;

__global__ void wmma_sgemm_kernel(half *a, half *b, float *c, float *d,
        int M, int N, int K, float alpha, float beta) {
    // Leading dimensions. Packed with no transpositions.
    int lda = M;
    int ldb = K;
    int ldc = N;

    // Tile using a 2D grid
    int warpM = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize;
    int warpN = (blockIdx.y * blockDim.y + threadIdx.y);

    // Declare the fragments
    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> a_frag;
//    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> b_frag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> b_frag;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> acc_frag;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> c_frag;

    wmma::fill_fragment(acc_frag, 0.0f);

    // Loop over k
    for (int i = 0; i < K; i += WMMA_K) {
        int aCol = i;
        int aRow = warpM * WMMA_M;
//        int bRow = i;
//        int bCol = warpN * WMMA_N;
        int bCol = i;
        int bRow = warpN * WMMA_N;
        // Bounds checking
        if (aRow < M && aCol < K && bRow < K && bCol < N) {
            // Load the inputs
//            wmma::load_matrix_sync(a_frag, a + aRow * K + aCol, lda);
//            wmma::load_matrix_sync(b_frag, b + bRow * N + bCol, ldb);
            wmma::load_matrix_sync(a_frag, a + aRow * M + aCol, lda);
            wmma::load_matrix_sync(b_frag, b + bRow * K + bCol, ldb);

            // Perform the matrix multiplication
            wmma::mma_sync(acc_frag, a_frag, b_frag, acc_frag);
        }
    }

    // Load in the current value of c, scale it by beta, and add this our result scaled by alpha
    int cCol = warpN * WMMA_N;
    int cRow = warpM * WMMA_M;

    if (cRow < M && cCol < N) {
        wmma::load_matrix_sync(c_frag, c + cRow * N + cCol, ldc, wmma::mem_row_major);

        for(int i=0; i < c_frag.num_elements; i++) {
            c_frag.x[i] = alpha * acc_frag.x[i] + beta * c_frag.x[i];
        }

        // Store the output
        wmma::store_matrix_sync(d + cRow * N + cCol, c_frag, ldc, wmma::mem_row_major);
    }
}
