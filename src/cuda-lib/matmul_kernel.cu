#include "hip/hip_runtime.h"
#include <iostream>

#include "matmul_kernel.h"
#include <hip/hip_runtime.h>
#include <>

__global__ void cuda_kernel_warmup(float* p) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    float f = (float)idx;
    p[idx] = f * f * f;
}

// naive!!
__global__ void cuda_kernel_sgemm_0(
        float* a, float* b, float* c, size_t N, size_t M, size_t K, float alpha, float beta) {
    int ir = blockIdx.x * 32 + threadIdx.x; // row idx in global
    int ic = blockIdx.y * 32 + threadIdx.y; // col idx in global

    if (ir < M && ic < N) {
#define idx(ri, ci, nc) ((ri) * (nc) + (ci))
        float acc = 0.0f;
        for (int k = 0; k < K; ++k) {
            acc += a[idx(ir, k, K)] * b[idx(k, ic, N)];
        }
        c[idx(ir, ic, N)] = alpha * acc + beta * c[idx(ir, ic, N)];
#undef idx
    }
}

// use shared memory & tile
__global__ void cuda_kernel_sgemm_1(
        float* a, float* b, float* c, size_t N, size_t M, size_t K, float alpha, float beta) {
    int tr = threadIdx.x;                   // row idx in block
    int tc = threadIdx.y;                   // col idx in block
    int ir = blockIdx.x * 32 + threadIdx.x; // row idx in global
    int ic = blockIdx.y * 32 + threadIdx.y; // col idx in global

    __shared__ float a_sub[32][32 + 1];
    __shared__ float b_sub[32][32 + 1];

    int load_size = K / 32;
    if (K % 32 != 0) {
        load_size += 1;
    }
    float acc = 0.0f;
    int a_ir = ir;
    int b_ic = ic;
#define idx(ri, ci, nc) ((ri) * (nc) + (ci))
    for (int l = 0; l < load_size; ++l) {
        int a_ic = l * 32 + tc;
        int b_ir = l * 32 + tr;
        a_sub[tr][tc] = 0.0f;
        b_sub[tr][tc] = 0.0f;
        if (a_ir < M && a_ic < K)
            a_sub[tr][tc] = a[idx(a_ir, a_ic, K)];
        if (b_ir < K && b_ic < N)
            b_sub[tr][tc] = b[idx(b_ir, b_ic, N)];

        __syncthreads();

#pragma unroll
        for (int k = 0; k < 32; ++k) {
            acc += a_sub[tr][k] * b_sub[k][tc];
        }

        __syncthreads();
    }

    if (ir < M && ic < N)
        c[idx(ir, ic, N)] = alpha * acc + beta * c[idx(ir, ic, N)];
#undef idx
}

// use __ldg & avoid bank conflict
__global__ void cuda_kernel_sgemm_2(
        float* a, float* b, float* c, size_t N, size_t M, size_t K, float alpha, float beta) {
    int tr = threadIdx.x;                   // row idx in block
    int tc = threadIdx.y;                   // col idx in block
    int ir = blockIdx.x * 32 + threadIdx.x; // row idx in global
    int ic = blockIdx.y * 32 + threadIdx.y; // col idx in global

    __shared__ float a_sub[32][32 + 1]; // avoid bank conflict
    __shared__ float b_sub[32][32 + 1];

    int load_size = K / 32;
    if (K % 32 != 0) {
        load_size += 1;
    }
    float acc = 0.0f;
    int a_ir = ir;
    int b_ic = ic;
#define idx(ri, ci, nc) ((ri) * (nc) + (ci))
    for (int l = 0; l < load_size; ++l) {
        int a_ic = l * 32 + tc;
        int b_ir = l * 32 + tr;
        a_sub[tr][tc] = 0.0f;
        b_sub[tr][tc] = 0.0f;
        if (a_ir < M && a_ic < K) {
            a_sub[tr][tc] = __ldg(&a[idx(a_ir, a_ic, K)]); // cache
                                                           //            a_sub[tr][tc] = a[idx(a_ir, a_ic, K)];
        }
        if (b_ir < K && b_ic < N) {
            b_sub[tr][tc] = __ldg(&b[idx(b_ir, b_ic, N)]);
            //            b_sub[tr][tc] = b[idx(b_ir, b_ic, N)];
        }

        __syncthreads();

#pragma unroll
        for (int k = 0; k < 32; ++k) {
            acc += a_sub[tr][k] * b_sub[k][tc];
        }

        __syncthreads();
    }

    if (ir < M && ic < N)
        c[idx(ir, ic, N)] = alpha * acc + beta * c[idx(ir, ic, N)];
#undef idx
}

__global__ void cuda_kernel_sgemm_2_64x64(
        float* a, float* b, float* c, size_t N, size_t M, size_t K, float alpha, float beta) {
    int tr = 2 * threadIdx.x;                   // row idx in block
    int tc = 2 * threadIdx.y;                   // col idx in block
    int ir = blockIdx.x * 64 + 2 * threadIdx.x; // row idx in global
    int ic = blockIdx.y * 64 + 2 * threadIdx.y; // col idx in global

    __shared__ float a_sub[64][64 + 1]; // avoid bank conflict
    __shared__ float b_sub[64][64 + 1];

    int load_size = K / 64;
    if (K % 64 != 0) {
        load_size += 1;
    }
    float acc00 = 0.0f;
    float acc01 = 0.0f;
    float acc10 = 0.0f;
    float acc11 = 0.0f;
    int a_ir = ir;
    int b_ic = ic;
#define idx(ri, ci, nc) ((ri) * (nc) + (ci))
    for (int l = 0; l < load_size; ++l) {
        int a_ic = l * 64 + tc;
        int b_ir = l * 64 + tr;
        a_sub[tr][tc] = 0.0f;
        b_sub[tr][tc] = 0.0f;
        if (a_ir < M - 1 && a_ic < K - 1) {
            // a_sub[tr + 0][tc + 0] = __ldg(&a[idx(a_ir + 0, a_ic + 0, K)]);  // cache
            // a_sub[tr + 0][tc + 1] = __ldg(&a[idx(a_ir + 0, a_ic + 1, K)]);  // cache
            // a_sub[tr + 1][tc + 0] = __ldg(&a[idx(a_ir + 1, a_ic + 0, K)]);  // cache
            // a_sub[tr + 1][tc + 1] = __ldg(&a[idx(a_ir + 1, a_ic + 1, K)]);  // cache

            a_sub[tr + 0][tc + 0] = a[idx(a_ir + 0, a_ic + 0, K)];
            a_sub[tr + 0][tc + 1] = a[idx(a_ir + 0, a_ic + 1, K)];
            a_sub[tr + 1][tc + 0] = a[idx(a_ir + 1, a_ic + 0, K)];
            a_sub[tr + 1][tc + 1] = a[idx(a_ir + 1, a_ic + 1, K)];
        }
        if (b_ir < K - 1 && b_ic < N - 1) {
            // b_sub[tr + 0][tc + 0] = __ldg(&b[idx(b_ir + 0, b_ic + 0, N)]);  // cache
            // b_sub[tr + 0][tc + 1] = __ldg(&b[idx(b_ir + 0, b_ic + 1, N)]);  // cache
            // b_sub[tr + 1][tc + 0] = __ldg(&b[idx(b_ir + 1, b_ic + 0, N)]);  // cache
            // b_sub[tr + 1][tc + 1] = __ldg(&b[idx(b_ir + 1, b_ic + 1, N)]);  // cache

            b_sub[tr + 0][tc + 0] = b[idx(b_ir + 0, b_ic + 0, N)]; // cache
            b_sub[tr + 0][tc + 1] = b[idx(b_ir + 0, b_ic + 1, N)]; // cache
            b_sub[tr + 1][tc + 0] = b[idx(b_ir + 1, b_ic + 0, N)]; // cache
            b_sub[tr + 1][tc + 1] = b[idx(b_ir + 1, b_ic + 1, N)]; // cache
        }

        __syncthreads();

#pragma unroll
        for (int k = 0; k < 64; ++k) {
            acc00 += a_sub[tr + 0][k] * b_sub[k][tc + 0];
            acc01 += a_sub[tr + 0][k] * b_sub[k][tc + 1];
            acc10 += a_sub[tr + 1][k] * b_sub[k][tc + 0];
            acc11 += a_sub[tr + 1][k] * b_sub[k][tc + 1];
        }

        __syncthreads();
    }

    if (ir < M - 1 && ic < N - 1)
        c[idx(ir + 0, ic + 0, N)] = alpha * acc00 + beta * c[idx(ir + 0, ic + 0, N)];
    c[idx(ir + 0, ic + 1, N)] = alpha * acc01 + beta * c[idx(ir + 0, ic + 1, N)];
    c[idx(ir + 1, ic + 0, N)] = alpha * acc10 + beta * c[idx(ir + 1, ic + 0, N)];
    c[idx(ir + 1, ic + 1, N)] = alpha * acc11 + beta * c[idx(ir + 1, ic + 1, N)];
#undef idx
}

__device__ void sgemm_block_64x64(float* a, float* b, float* c, size_t M, size_t N, size_t K, float alpha, float beta) {

    __shared__ float a_b_shm[2 * 16 * 64];

    // int tid = threadIdx.x & 0x3f;
    int tid = threadIdx.x;
    int ldx = tid >= 32 ? N : M;
    int tid2 = (tid >> 4) & 1;
    // int tid15 = tid & 0xf;
    int tid15 = tid & 15;
    int track0 = tid2 * ldx + tid15 * 4;
    int track2 = track0 + 2 * ldx;
    int track4 = track0 + 4 * ldx;
    int track6 = track0 + 6 * ldx;
    int end = track0 + (K - 8) * ldx;
    int write_offset = tid2 * 64 + tid15 * 4;
    write_offset += tid >= 32 ? 512 : 0;

    int readAs = ((tid >> 1) & 7) << 2;
    int readBs = ((((tid & 0x30) >> 3) | (tid & 1)) << 2) + 512;

    float* read_addr = tid >= 32 ? b : a;

    float cbb00 = 0, cbb01 = 0, cbb02 = 0, cbb03 = 0;
    float cbb10 = 0, cbb11 = 0, cbb12 = 0, cbb13 = 0;
    float cbb20 = 0, cbb21 = 0, cbb22 = 0, cbb23 = 0;
    float cbb30 = 0, cbb31 = 0, cbb32 = 0, cbb33 = 0;
    float cba00 = 0, cba01 = 0, cba02 = 0, cba03 = 0;
    float cba10 = 0, cba11 = 0, cba12 = 0, cba13 = 0;
    float cba20 = 0, cba21 = 0, cba22 = 0, cba23 = 0;
    float cba30 = 0, cba31 = 0, cba32 = 0, cba33 = 0;
    float cab00 = 0, cab01 = 0, cab02 = 0, cab03 = 0;
    float cab10 = 0, cab11 = 0, cab12 = 0, cab13 = 0;
    float cab20 = 0, cab21 = 0, cab22 = 0, cab23 = 0;
    float cab30 = 0, cab31 = 0, cab32 = 0, cab33 = 0;
    float caa00 = 0, caa01 = 0, caa02 = 0, caa03 = 0;
    float caa10 = 0, caa11 = 0, caa12 = 0, caa13 = 0;
    float caa20 = 0, caa21 = 0, caa22 = 0, caa23 = 0;
    float caa30 = 0, caa31 = 0, caa32 = 0, caa33 = 0;

    // float cbb00, cbb01, cbb02, cbb03;
    // float cbb10, cbb11, cbb12, cbb13;
    // float cbb20, cbb21, cbb22, cbb23;
    // float cbb30, cbb31, cbb32, cbb33;
    // float cba00, cba01, cba02, cba03;
    // float cba10, cba11, cba12, cba13;
    // float cba20, cba21, cba22, cba23;
    // float cba30, cba31, cba32, cba33;
    // float cab00, cab01, cab02, cab03;
    // float cab10, cab11, cab12, cab13;
    // float cab20, cab21, cab22, cab23;
    // float cab30, cab31, cab32, cab33;
    // float caa00, caa01, caa02, caa03;
    // float caa10, caa11, caa12, caa13;
    // float caa20, caa21, caa22, caa23;
    // float caa30, caa31, caa32, caa33;

    float j0Ab00, j0Ab01, j0Ab02, j0Ab03;
    float j0Bb00, j0Bb01, j0Bb02, j0Bb03;
    float j0Aa00, j0Aa01, j0Aa02, j0Aa03;
    float j0Ba00, j0Ba01, j0Ba02, j0Ba03;
    // float j1Ab00, j1Ab01, j1Ab02, j1Ab03;
    // float j1Bb00, j1Bb01, j1Bb02, j1Bb03;
    // float j1Aa00, j1Aa01, j1Aa02, j1Aa03;
    // float j1Ba00, j1Ba01, j1Ba02, j1Ba03;

    // float j0Ab00=1, j0Ab01=1, j0Ab02=1, j0Ab03=1;
    // float j0Bb00=1, j0Bb01=1, j0Bb02=1, j0Bb03=1;
    // float j0Aa00=1, j0Aa01=1, j0Aa02=1, j0Aa03=1;
    // float j0Ba00=1, j0Ba01=1, j0Ba02=1, j0Ba03=1;
    // float j1Ab00=1, j1Ab01=1, j1Ab02=1, j1Ab03=1;
    // float j1Bb00=1, j1Bb01=1, j1Bb02=1, j1Bb03=1;
    // float j1Aa00=1, j1Aa01=1, j1Aa02=1, j1Aa03=1;
    // float j1Ba00=1, j1Ba01=1, j1Ba02=1, j1Ba03=1;

    while (track0 <= end) {
        a_b_shm[write_offset + 0 * 64 + 0] = read_addr[track0 + 0];
        a_b_shm[write_offset + 0 * 64 + 1] = read_addr[track0 + 1];
        a_b_shm[write_offset + 0 * 64 + 2] = read_addr[track0 + 2];
        a_b_shm[write_offset + 0 * 64 + 3] = read_addr[track0 + 3];

        a_b_shm[write_offset + 2 * 64 + 0] = read_addr[track2 + 0];
        a_b_shm[write_offset + 2 * 64 + 1] = read_addr[track2 + 1];
        a_b_shm[write_offset + 2 * 64 + 2] = read_addr[track2 + 2];
        a_b_shm[write_offset + 2 * 64 + 3] = read_addr[track2 + 3];

        a_b_shm[write_offset + 4 * 64 + 0] = read_addr[track4 + 0];
        a_b_shm[write_offset + 4 * 64 + 1] = read_addr[track4 + 1];
        a_b_shm[write_offset + 4 * 64 + 2] = read_addr[track4 + 2];
        a_b_shm[write_offset + 4 * 64 + 3] = read_addr[track4 + 3];

        a_b_shm[write_offset + 6 * 64 + 0] = read_addr[track6 + 0];
        a_b_shm[write_offset + 6 * 64 + 1] = read_addr[track6 + 1];
        a_b_shm[write_offset + 6 * 64 + 2] = read_addr[track6 + 2];
        a_b_shm[write_offset + 6 * 64 + 3] = read_addr[track6 + 3];

        __syncthreads();
        // __syncwarp(0xFFFFFFFF);

        write_offset ^= 16 * 64;
        track0 += 8 * ldx;
        track2 += 8 * ldx;
        track4 += 8 * ldx;
        track6 += 8 * ldx;

        for (int j = 0; j < 8; ++j) {
            // int prefetch = (j + 1) % 8;
            int prefetch = j;

            j0Ab00 = a_b_shm[readAs + prefetch * 64 + 0];
            j0Ab01 = a_b_shm[readAs + prefetch * 64 + 1];
            j0Ab02 = a_b_shm[readAs + prefetch * 64 + 2];
            j0Ab03 = a_b_shm[readAs + prefetch * 64 + 3];

            j0Bb00 = a_b_shm[readBs + prefetch * 64 + 0];
            j0Bb01 = a_b_shm[readBs + prefetch * 64 + 1];
            j0Bb02 = a_b_shm[readBs + prefetch * 64 + 2];
            j0Bb03 = a_b_shm[readBs + prefetch * 64 + 3];

            j0Aa00 = a_b_shm[readAs + prefetch * 64 + 32 + 0];
            j0Aa01 = a_b_shm[readAs + prefetch * 64 + 32 + 1];
            j0Aa02 = a_b_shm[readAs + prefetch * 64 + 32 + 2];
            j0Aa03 = a_b_shm[readAs + prefetch * 64 + 32 + 3];

            j0Ba00 = a_b_shm[readBs + prefetch * 64 + 32 + 0];
            j0Ba01 = a_b_shm[readBs + prefetch * 64 + 32 + 1];
            j0Ba02 = a_b_shm[readBs + prefetch * 64 + 32 + 2];
            j0Ba03 = a_b_shm[readBs + prefetch * 64 + 32 + 3];

            // cbb00 = j0Ab00;
            // cbb01 = j0Ab01;
            // cbb02 = j0Ab02;
            // cbb03 = j0Ab03;
            // cbb10 = j0Bb00;
            // cbb11 = j0Bb01;
            // cbb12 = j0Bb02;
            // cbb13 = j0Bb03;
            // cbb20 = j0Aa00;
            // cbb21 = j0Aa01;
            // cbb22 = j0Aa02;
            // cbb23 = j0Aa03;
            // cbb30 = j0Ba00;
            // cbb31 = j0Ba01;
            // cbb32 = j0Ba02;
            // cbb33 = j0Ba03;

            // cba00 = j0Ab00;
            // cba01 = j0Ab01;
            // cba02 = j0Ab02;
            // cba03 = j0Ab03;
            // cba10 = j0Bb00;
            // cba11 = j0Bb01;
            // cba12 = j0Bb02;
            // cba13 = j0Bb03;
            // cba20 = j0Aa00;
            // cba21 = j0Aa01;
            // cba22 = j0Aa02;
            // cba23 = j0Aa03;
            // cba30 = j0Ba00;
            // cba31 = j0Ba01;
            // cba32 = j0Ba02;
            // cba33 = j0Ba03;

            // cab00 = j0Ab00;
            // cab01 = j0Ab01;
            // cab02 = j0Ab02;
            // cab03 = j0Ab03;
            // cab10 = j0Bb00;
            // cab11 = j0Bb01;
            // cab12 = j0Bb02;
            // cab13 = j0Bb03;
            // cab20 = j0Aa00;
            // cab21 = j0Aa01;
            // cab22 = j0Aa02;
            // cab23 = j0Aa03;
            // cab30 = j0Ba00;
            // cab31 = j0Ba01;
            // cab32 = j0Ba02;
            // cab33 = j0Ba03;

            // caa00 = j0Ab00;
            // caa01 = j0Ab01;
            // caa02 = j0Ab02;
            // caa03 = j0Ab03;
            // caa10 = j0Bb00;
            // caa11 = j0Bb01;
            // caa12 = j0Bb02;
            // caa13 = j0Bb03;
            // caa20 = j0Aa00;
            // caa21 = j0Aa01;
            // caa22 = j0Aa02;
            // caa23 = j0Aa03;
            // caa30 = j0Ba00;
            // caa31 = j0Ba01;
            // caa32 = j0Ba02;
            // caa33 = j0Ba03;

            cbb00 += j0Bb00 * j0Ab00;
            cbb01 += j0Bb00 * j0Ab01;
            // j1Ab00 = a_b_shm[readAs + prefetch * 64 + 0];
            // j1Ab01 = a_b_shm[readAs + prefetch * 64 + 1];
            // j1Ab02 = a_b_shm[readAs + prefetch * 64 + 2];
            // j1Ab03 = a_b_shm[readAs + prefetch * 64 + 3];
            cbb02 += j0Bb00 * j0Ab02;
            cbb03 += j0Bb00 * j0Ab03;
            // j1Bb00 = a_b_shm[readBs + prefetch * 64 + 0];
            // j1Bb01 = a_b_shm[readBs + prefetch * 64 + 1];
            // j1Bb02 = a_b_shm[readBs + prefetch * 64 + 2];
            // j1Bb03 = a_b_shm[readBs + prefetch * 64 + 3];

            cbb10 += j0Bb01 * j0Ab00;
            cbb11 += j0Bb01 * j0Ab01;
            // j1Aa00 = a_b_shm[readAs + prefetch * 64 + 32 + 0];
            // j1Aa01 = a_b_shm[readAs + prefetch * 64 + 32 + 1];
            // j1Aa02 = a_b_shm[readAs + prefetch * 64 + 32 + 2];
            // j1Aa03 = a_b_shm[readAs + prefetch * 64 + 32 + 3];
            cbb12 += j0Bb01 * j0Ab02;
            cbb13 += j0Bb01 * j0Ab03;
            // j1Ba00 = a_b_shm[readBs + prefetch * 64 + 32 + 0];
            // j1Ba01 = a_b_shm[readBs + prefetch * 64 + 32 + 1];
            // j1Ba02 = a_b_shm[readBs + prefetch * 64 + 32 + 2];
            // j1Ba03 = a_b_shm[readBs + prefetch * 64 + 32 + 3];

            cbb20 += j0Bb02 * j0Ab00;
            cbb21 += j0Bb02 * j0Ab01;
            cbb22 += j0Bb02 * j0Ab02;
            cbb23 += j0Bb02 * j0Ab03;

            cbb30 += j0Bb03 * j0Ab00;
            cbb31 += j0Bb03 * j0Ab01;
            cbb32 += j0Bb03 * j0Ab02;
            cbb33 += j0Bb03 * j0Ab03;

            cba00 += j0Ba00 * j0Ab00;
            cba01 += j0Ba00 * j0Ab01;
            cba02 += j0Ba00 * j0Ab02;
            cba03 += j0Ba00 * j0Ab03;

            cba10 += j0Ba01 * j0Ab00;
            cba11 += j0Ba01 * j0Ab01;
            cba12 += j0Ba01 * j0Ab02;
            cba13 += j0Ba01 * j0Ab03;

            cba20 += j0Ba02 * j0Ab00;
            cba21 += j0Ba02 * j0Ab01;
            cba22 += j0Ba02 * j0Ab02;
            cba23 += j0Ba02 * j0Ab03;

            cba30 += j0Ba03 * j0Ab00;
            cba31 += j0Ba03 * j0Ab01;
            cba32 += j0Ba03 * j0Ab02;
            cba33 += j0Ba03 * j0Ab03;

            cab00 += j0Bb00 * j0Aa00;
            cab01 += j0Bb00 * j0Aa01;
            cab02 += j0Bb00 * j0Aa02;
            cab03 += j0Bb00 * j0Aa03;

            cab10 += j0Bb01 * j0Aa00;
            cab11 += j0Bb01 * j0Aa01;
            cab12 += j0Bb01 * j0Aa02;
            cab13 += j0Bb01 * j0Aa03;

            cab20 += j0Bb02 * j0Aa00;
            cab21 += j0Bb02 * j0Aa01;
            cab22 += j0Bb02 * j0Aa02;
            cab23 += j0Bb02 * j0Aa03;

            cab30 += j0Bb03 * j0Aa00;
            cab31 += j0Bb03 * j0Aa01;
            cab32 += j0Bb03 * j0Aa02;
            cab33 += j0Bb03 * j0Aa03;

            caa00 += j0Ba00 * j0Aa00;
            caa01 += j0Ba00 * j0Aa01;
            caa02 += j0Ba00 * j0Aa02;
            caa03 += j0Ba00 * j0Aa03;

            caa10 += j0Ba01 * j0Aa00;
            caa11 += j0Ba01 * j0Aa01;
            caa12 += j0Ba01 * j0Aa02;
            caa13 += j0Ba01 * j0Aa03;

            caa20 += j0Ba02 * j0Aa00;
            caa21 += j0Ba02 * j0Aa01;
            caa22 += j0Ba02 * j0Aa02;
            caa23 += j0Ba02 * j0Aa03;

            caa30 += j0Ba03 * j0Aa00;
            caa31 += j0Ba03 * j0Aa01;
            caa32 += j0Ba03 * j0Aa02;
            caa33 += j0Ba03 * j0Aa03;
        }

        readAs ^= 16 * 64;
        readBs ^= 16 * 64;
    }
    __syncthreads();

    int tid31 = tid & 31;
    int tid32 = tid & 32;
    int coord_x = readBs & 0x7f;
    int coord_y = readAs & 0x7f;
    int writeCs = coord_x / 4 * 64 + coord_y;
    int readCs = (tid32 << 3) + tid31;
    int ldc4 = M * 4;
    int Cy00 = (tid32 >> 1) * M + tid31;
    int Cy04 = Cy00 + ldc4;
    int Cy08 = Cy00 + 2 * ldc4;
    int Cy12 = Cy00 + 3 * ldc4;

    a_b_shm[writeCs + 0] = cbb00;
    a_b_shm[writeCs + 1] = cbb01;
    a_b_shm[writeCs + 2] = cbb02;
    a_b_shm[writeCs + 3] = cbb03;
    a_b_shm[writeCs + 32 + 0] = cab00;
    a_b_shm[writeCs + 32 + 1] = cab01;
    a_b_shm[writeCs + 32 + 2] = cab02;
    a_b_shm[writeCs + 32 + 3] = cab03;
    // if (threadIdx.x == 1) {
    //     printf("reg r0,  c4: %f\n", cbb00);
    // }
    // if (threadIdx.x == 18) {
    //     printf("reg r7,  c8: %f\n", cbb03);
    //     printf("reg r39, c8: %f\n", cab03);
    // }

    cbb00 = a_b_shm[readCs + 0 * 64 + 0];
    cbb01 = a_b_shm[readCs + 0 * 64 + 32];
    cbb02 = a_b_shm[readCs + 1 * 64 + 0];
    cbb03 = a_b_shm[readCs + 1 * 64 + 32];
    cab00 = a_b_shm[readCs + 2 * 64 + 0];
    cab01 = a_b_shm[readCs + 2 * 64 + 32];
    cab02 = a_b_shm[readCs + 3 * 64 + 0];
    cab03 = a_b_shm[readCs + 3 * 64 + 32];
    c[Cy00 + 0] = cbb00;
    c[Cy00 + 32] = cbb01;
    c[Cy04 + 0] = cbb02;
    c[Cy04 + 32] = cbb03;
    c[Cy08 + 0] = cab00;
    c[Cy08 + 32] = cab01;
    c[Cy12 + 0] = cab02;
    c[Cy12 + 32] = cab03;

    Cy00 += M;
    Cy04 += M;
    Cy08 += M;
    Cy12 += M;
    a_b_shm[writeCs + 0] = cbb10;
    a_b_shm[writeCs + 1] = cbb11;
    a_b_shm[writeCs + 2] = cbb12;
    a_b_shm[writeCs + 3] = cbb13;
    a_b_shm[writeCs + 32 + 0] = cab10;
    a_b_shm[writeCs + 32 + 1] = cab11;
    a_b_shm[writeCs + 32 + 2] = cab12;
    a_b_shm[writeCs + 32 + 3] = cab13;

    cbb10 = a_b_shm[readCs + 0 * 64 + 0];
    cbb11 = a_b_shm[readCs + 0 * 64 + 32];
    cbb12 = a_b_shm[readCs + 1 * 64 + 0];
    cbb13 = a_b_shm[readCs + 1 * 64 + 32];
    cab10 = a_b_shm[readCs + 2 * 64 + 0];
    cab11 = a_b_shm[readCs + 2 * 64 + 32];
    cab12 = a_b_shm[readCs + 3 * 64 + 0];
    cab13 = a_b_shm[readCs + 3 * 64 + 32];
    c[Cy00 + 0] = cbb10;
    c[Cy00 + 32] = cbb11;
    c[Cy04 + 0] = cbb12;
    c[Cy04 + 32] = cbb13;
    c[Cy08 + 0] = cab10;
    c[Cy08 + 32] = cab11;
    c[Cy12 + 0] = cab12;
    c[Cy12 + 32] = cab13;

    Cy00 += M;
    Cy04 += M;
    Cy08 += M;
    Cy12 += M;
    a_b_shm[writeCs + 0] = cbb20;
    a_b_shm[writeCs + 1] = cbb21;
    a_b_shm[writeCs + 2] = cbb22;
    a_b_shm[writeCs + 3] = cbb23;
    a_b_shm[writeCs + 32 + 0] = cab20;
    a_b_shm[writeCs + 32 + 1] = cab21;
    a_b_shm[writeCs + 32 + 2] = cab22;
    a_b_shm[writeCs + 32 + 3] = cab23;

    cbb20 = a_b_shm[readCs + 0 * 64 + 0];
    cbb21 = a_b_shm[readCs + 0 * 64 + 32];
    cbb22 = a_b_shm[readCs + 1 * 64 + 0];
    cbb23 = a_b_shm[readCs + 1 * 64 + 32];
    cab20 = a_b_shm[readCs + 2 * 64 + 0];
    cab21 = a_b_shm[readCs + 2 * 64 + 32];
    cab22 = a_b_shm[readCs + 3 * 64 + 0];
    cab23 = a_b_shm[readCs + 3 * 64 + 32];
    c[Cy00 + 0] = cbb20;
    c[Cy00 + 32] = cbb21;
    c[Cy04 + 0] = cbb22;
    c[Cy04 + 32] = cbb23;
    c[Cy08 + 0] = cab20;
    c[Cy08 + 32] = cab21;
    c[Cy12 + 0] = cab22;
    c[Cy12 + 32] = cab23;

    Cy00 += M;
    Cy04 += M;
    Cy08 += M;
    Cy12 += M;
    a_b_shm[writeCs + 0] = cbb30;
    a_b_shm[writeCs + 1] = cbb31;
    a_b_shm[writeCs + 2] = cbb32;
    a_b_shm[writeCs + 3] = cbb33;
    a_b_shm[writeCs + 32 + 0] = cab30;
    a_b_shm[writeCs + 32 + 1] = cab31;
    a_b_shm[writeCs + 32 + 2] = cab32;
    a_b_shm[writeCs + 32 + 3] = cab33;

    cbb30 = a_b_shm[readCs + 0 * 64 + 0];
    cbb31 = a_b_shm[readCs + 0 * 64 + 32];
    cbb32 = a_b_shm[readCs + 1 * 64 + 0];
    cbb33 = a_b_shm[readCs + 1 * 64 + 32];
    cab30 = a_b_shm[readCs + 2 * 64 + 0];
    cab31 = a_b_shm[readCs + 2 * 64 + 32];
    cab32 = a_b_shm[readCs + 3 * 64 + 0];
    cab33 = a_b_shm[readCs + 3 * 64 + 32];
    c[Cy00 + 0] = cbb30;
    c[Cy00 + 32] = cbb31;
    c[Cy04 + 0] = cbb32;
    c[Cy04 + 32] = cbb33;
    c[Cy08 + 0] = cab30;
    c[Cy08 + 32] = cab31;
    c[Cy12 + 0] = cab32;
    c[Cy12 + 32] = cab33;

    Cy00 += M;
    Cy04 += M;
    Cy08 += M;
    Cy12 += M;

    Cy00 += 28 * M;
    Cy04 += 28 * M;
    Cy08 += 28 * M;
    Cy12 += 28 * M;

    a_b_shm[writeCs + 0] = cba00;
    a_b_shm[writeCs + 1] = cba01;
    a_b_shm[writeCs + 2] = cba02;
    a_b_shm[writeCs + 3] = cba03;
    a_b_shm[writeCs + 32 + 0] = caa00;
    a_b_shm[writeCs + 32 + 1] = caa01;
    a_b_shm[writeCs + 32 + 2] = caa02;
    a_b_shm[writeCs + 32 + 3] = caa03;

    cba00 = a_b_shm[readCs + 0 * 64 + 0];
    cba01 = a_b_shm[readCs + 0 * 64 + 32];
    cba02 = a_b_shm[readCs + 1 * 64 + 0];
    cba03 = a_b_shm[readCs + 1 * 64 + 32];
    caa00 = a_b_shm[readCs + 2 * 64 + 0];
    caa01 = a_b_shm[readCs + 2 * 64 + 32];
    caa02 = a_b_shm[readCs + 3 * 64 + 0];
    caa03 = a_b_shm[readCs + 3 * 64 + 32];
    c[Cy00 + 0] = cba00;
    c[Cy00 + 32] = cba01;
    c[Cy04 + 0] = cba02;
    c[Cy04 + 32] = cba03;
    c[Cy08 + 0] = caa00;
    c[Cy08 + 32] = caa01;
    c[Cy12 + 0] = caa02;
    c[Cy12 + 32] = caa03;

    Cy00 += M;
    Cy04 += M;
    Cy08 += M;
    Cy12 += M;
    a_b_shm[writeCs + 0] = cba10;
    a_b_shm[writeCs + 1] = cba11;
    a_b_shm[writeCs + 2] = cba12;
    a_b_shm[writeCs + 3] = cba13;
    a_b_shm[writeCs + 32 + 0] = caa10;
    a_b_shm[writeCs + 32 + 1] = caa11;
    a_b_shm[writeCs + 32 + 2] = caa12;
    a_b_shm[writeCs + 32 + 3] = caa13;

    cba10 = a_b_shm[readCs + 0 * 64 + 0];
    cba11 = a_b_shm[readCs + 0 * 64 + 32];
    cba12 = a_b_shm[readCs + 1 * 64 + 0];
    cba13 = a_b_shm[readCs + 1 * 64 + 32];
    caa10 = a_b_shm[readCs + 2 * 64 + 0];
    caa11 = a_b_shm[readCs + 2 * 64 + 32];
    caa12 = a_b_shm[readCs + 3 * 64 + 0];
    caa13 = a_b_shm[readCs + 3 * 64 + 32];
    c[Cy00 + 0] = cba10;
    c[Cy00 + 32] = cba11;
    c[Cy04 + 0] = cba12;
    c[Cy04 + 32] = cba13;
    c[Cy08 + 0] = caa10;
    c[Cy08 + 32] = caa11;
    c[Cy12 + 0] = caa12;
    c[Cy12 + 32] = caa13;

    Cy00 += M;
    Cy04 += M;
    Cy08 += M;
    Cy12 += M;
    a_b_shm[writeCs + 0] = cba20;
    a_b_shm[writeCs + 1] = cba21;
    a_b_shm[writeCs + 2] = cba22;
    a_b_shm[writeCs + 3] = cba23;
    a_b_shm[writeCs + 32 + 0] = caa20;
    a_b_shm[writeCs + 32 + 1] = caa21;
    a_b_shm[writeCs + 32 + 2] = caa22;
    a_b_shm[writeCs + 32 + 3] = caa23;

    cba20 = a_b_shm[readCs + 0 * 64 + 0];
    cba21 = a_b_shm[readCs + 0 * 64 + 32];
    cba22 = a_b_shm[readCs + 1 * 64 + 0];
    cba23 = a_b_shm[readCs + 1 * 64 + 32];
    caa20 = a_b_shm[readCs + 2 * 64 + 0];
    caa21 = a_b_shm[readCs + 2 * 64 + 32];
    caa22 = a_b_shm[readCs + 3 * 64 + 0];
    caa23 = a_b_shm[readCs + 3 * 64 + 32];
    c[Cy00 + 0] = cba20;
    c[Cy00 + 32] = cba21;
    c[Cy04 + 0] = cba22;
    c[Cy04 + 32] = cba23;
    c[Cy08 + 0] = caa20;
    c[Cy08 + 32] = caa21;
    c[Cy12 + 0] = caa22;
    c[Cy12 + 32] = caa23;

    Cy00 += M;
    Cy04 += M;
    Cy08 += M;
    Cy12 += M;
    a_b_shm[writeCs + 0] = cba30;
    a_b_shm[writeCs + 1] = cba31;
    a_b_shm[writeCs + 2] = cba32;
    a_b_shm[writeCs + 3] = cba33;
    a_b_shm[writeCs + 32 + 0] = caa30;
    a_b_shm[writeCs + 32 + 1] = caa31;
    a_b_shm[writeCs + 32 + 2] = caa32;
    a_b_shm[writeCs + 32 + 3] = caa33;

    cba30 = a_b_shm[readCs + 0 * 64 + 0];
    cba31 = a_b_shm[readCs + 0 * 64 + 32];
    cba32 = a_b_shm[readCs + 1 * 64 + 0];
    cba33 = a_b_shm[readCs + 1 * 64 + 32];
    caa30 = a_b_shm[readCs + 2 * 64 + 0];
    caa31 = a_b_shm[readCs + 2 * 64 + 32];
    caa32 = a_b_shm[readCs + 3 * 64 + 0];
    caa33 = a_b_shm[readCs + 3 * 64 + 32];
    c[Cy00 + 0] = cba30;
    c[Cy00 + 32] = cba31;
    c[Cy04 + 0] = cba32;
    c[Cy04 + 32] = cba33;
    c[Cy08 + 0] = caa30;
    c[Cy08 + 32] = caa31;
    c[Cy12 + 0] = caa32;
    c[Cy12 + 32] = caa33;
}

__global__ void cuda_kernel_sgemm_100(
        float* a, float* b, float* c, size_t M, size_t N, size_t K, float alpha, float beta) {
    int block_x = blockIdx.x;
    int block_y = blockIdx.y;
    float* block_a = a + block_y * 64;
    float* block_b = b + block_x * 64;
    float* block_c = c + block_x * 64 * M + block_y * 64;
    sgemm_block_64x64(block_a, block_b, block_c, M, N, K, alpha, beta);
}

__global__ void ReferenceGemm_kernel(int M, int N, int K, float alpha, float const* A, int lda, float const* B, int ldb,
        float beta, float* C, int ldc) {

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    if (i < M && j < N) {
        float accumulator = 0;

        for (int k = 0; k < K; ++k) {
            accumulator += A[i * lda + k] * B[k * ldb + j];
            //            accumulator += A[i + k * lda] * B[k + j * ldb];
        }

        C[i * ldc + j] = alpha * accumulator + beta * C[i * ldc + j];
        //        C[i + j * ldc] = alpha * accumulator + beta * C[i + j * ldc];
    }
}

hipError_t ReferenceGemm(int M, int N, int K, float alpha, float const* A, int lda, float const* B, int ldb,
        float beta, float* C, int ldc) {

    dim3 block(16, 16);
    dim3 grid((M + block.x - 1) / block.x, (N + block.y - 1) / block.y);

    ReferenceGemm_kernel<<<grid, block>>>(M, N, K, alpha, A, lda, B, ldb, beta, C, ldc);

    return hipGetLastError();
}
