#include "hip/hip_runtime.h"
#include "matmul_kernel.h"
#include "prof.h"
#include "openblas/cblas.h"

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <assert.h>
#include <math.h>
#include <omp.h>
#include <iostream>

using namespace std;

#define WMMA_M 16
#define WMMA_N 16
#define WMMA_K 16

__global__ void cuda_kernel_sgemm_100_tex(
        float *a, float *b, float *c,
        size_t M, size_t N, size_t K,
        float alpha, float beta);

typedef texture<float, hipTextureType1D, hipReadModeElementType> floatTex;
texture<float, hipTextureType1D, hipReadModeElementType> tex1DRefA(0, hipFilterModePoint, hipAddressModeBorder);
texture<float, hipTextureType1D, hipReadModeElementType> tex1DRefB(0, hipFilterModePoint, hipAddressModeBorder);
#define     USE_TEXTURE     1


void gpu_sgemm(
        float *a, float *b, float *c,
        size_t N, size_t M, size_t K,
        float alpha, float beta, int kernel_type) {
    float *dev_a = NULL;
    float *dev_at = NULL;
    float *dev_b = NULL;
    float *dev_c = NULL;
    half *A = NULL;
    half *B = NULL;
    half *B_ht = NULL;
    float *C = NULL;
    float *D = NULL;
    float flop = 2 * (float)M * (float)N * (float)K;
    hipblasHandle_t handle;

    int lda = K;
    int ldb = N;
    int ldc = N;
//    int lda = M;
//    int ldb = K;
//    int ldc = M;

    if (kernel_type == 'b') hipblasCreate(&handle);
    if (kernel_type == 't') {
        half *B_h = (half*)b;
        B_ht = (half*)malloc(sizeof(half) * K * N);
        for (int i = 0; i < N; ++i) {
            for (int j = 0; j < K; ++j) {
                B_ht[i * K + j] = B_h[j * N + i];
            }
        }

        checkCudaErrors(hipMalloc((void**)&A, sizeof(half) * M * K));
        checkCudaErrors(hipMalloc((void**)&B, sizeof(half) * N * K));
        checkCudaErrors(hipMalloc((void**)&C, sizeof(float) * M * N));
        checkCudaErrors(hipMalloc((void**)&D, sizeof(float) * M * N));
//        assert((int(A)) % 128 == 0);
        assert(((unsigned long long)A) % 128 == 0);
        assert(((unsigned long long)B) % 128 == 0);
        assert(((unsigned long long)C) % 128 == 0);
        assert(((unsigned long long)D) % 128 == 0);
        checkCudaErrors(hipMemcpy(A, a, sizeof(half) * M * K, hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(B, B_ht, sizeof(half) * N * K, hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(C, c, sizeof(float) * M * N, hipMemcpyHostToDevice));
        checkCudaErrors(hipMemset(D, 0, sizeof(float) * M * N));
    }

    float* at = (float*)malloc(M * K * sizeof(float));
    for (int i = 0; i < M; ++i) {
        for (int j = 0; j < K; ++j) {
            at[j * M + i] = a[i * K + j];
        }
    }
    hipMalloc((void **)&dev_a, M * K * sizeof(float));
    hipMalloc((void **)&dev_at, M * K * sizeof(float));
    hipMalloc((void **)&dev_b, K * N * sizeof(float));
    hipMalloc((void **)&dev_c, M * N * sizeof(float));
    hipMemcpy(dev_a, a, M * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_at, at, M * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, K * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_c, c, M * N * sizeof(float), hipMemcpyHostToDevice);
    hipBindTexture(0, tex1DRefA, dev_at, M * K * sizeof(float));
    hipBindTexture(0, tex1DRefB, dev_b, K * N * sizeof(float));
    int cycle_count = 100;

//    hs_timer timer;
//    timer.tic("gpu sgemm");

    hipError_t result;
    hipEvent_t start;
    hipEvent_t stop;
    float msecTotal;
    hipEventCreate(&start);
    hipEventRecord(start, NULL);

    switch (kernel_type)
    {
        case 0:
        {
            int grid_r = M / 32;
            int grid_c = N / 32;
            if (M % 32 != 0)
                grid_r += 1;
            if (N % 32 != 0)
                grid_c += 1;
            dim3 grid_d(grid_r, grid_c, 1);
            dim3 block_d(32, 32, 1);
            cuda_kernel_sgemm_0<<<grid_d, block_d>>>(dev_a, dev_b, dev_c, N, M, K, alpha, beta);
            break;
        }
        case 1:
        {
            int grid_r = M / 32;
            int grid_c = N / 32;
            if (M % 32 != 0)
                grid_r += 1;
            if (N % 32 != 0)
                grid_c += 1;
            dim3 grid_d(grid_r, grid_c, 1);
            dim3 block_d(32, 32, 1);
            cuda_kernel_sgemm_1<<<grid_d, block_d>>>(dev_a, dev_b, dev_c, N, M, K, alpha, beta);
            break;
        }
        case 2:
        {
            int grid_r = M / 32;
            int grid_c = N / 32;
            if (M % 32 != 0)
                grid_r += 1;
            if (N % 32 != 0)
                grid_c += 1;
            dim3 grid_d(grid_r, grid_c, 1);
            dim3 block_d(32, 32, 1);
            for (int n = 0; n < cycle_count; ++n) {
                cuda_kernel_sgemm_2<<<grid_d, block_d>>>(dev_a, dev_b, dev_c, N, M, K, alpha, beta);
            }
            break;
        }
        case 20:
        {
            int grid_r = M / 64;
            int grid_c = N / 64;
            if (M % 64 != 0)
                grid_r += 1;
            if (N % 64 != 0)
                grid_c += 1;
            dim3 grid_d(grid_r, grid_c, 1);
            dim3 block_d(32, 32, 1);
            for (int n = 0; n < cycle_count; ++n) {
                cuda_kernel_sgemm_2_64x64<<<grid_d, block_d>>>(dev_a, dev_b, dev_c, N, M, K, alpha, beta);
            }
            break;
        }
        case 'b':
        {
            for (int n = 0; n < cycle_count; ++n) {
                hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, dev_b, N, dev_a, K, &beta, dev_c, N);
            }
            break;
        }
        case 'c':
        {
            for (int n = 0; n < cycle_count; ++n) {
                result = CutlassSgemmNN(M, N, K, alpha, dev_a, lda, dev_b, ldb, beta, dev_c, ldc);
            }
            if (result == hipSuccess) {
                cout << "CutlassSgemmNN success" << endl;
            }
            break;
        }
        case 'r':
        {
            hipError_t result;
            result = ReferenceGemm(M, N, K, alpha, dev_a, lda, dev_b, ldb, beta, dev_c, ldc);
            if (result == hipSuccess) {
                cout << "ReferenceGemm success" << endl;
            }
            break;
        }
        case 't':
        {
            dim3 gridDim;
            dim3 blockDim;

            // blockDim.x must be a multple of warpSize
            // 128x4 means we have 16 warps and a block computes a 64x64 output tile
            blockDim.x = 128;
            blockDim.y = 4;

            gridDim.x = (M + (WMMA_M * blockDim.x / 32 - 1)) / (WMMA_M * blockDim.x / 32);
            gridDim.y = (N + WMMA_N * blockDim.y - 1) / (WMMA_N * blockDim.y);

            wmma_sgemm_kernel<<<gridDim, blockDim>>>(A, B, C, D, M, N, K, alpha, beta);
        }
        case 100:
        {
            int stride_x = 64;
            int stride_y = 64;
            int grid_x = (N + stride_x - 1) / stride_x;
            int grid_y = (M + stride_y - 1) / stride_y;
            int block_x = stride_x;
            dim3 grid_d(grid_x, grid_y, 1);
            dim3 block_d(block_x, 1, 1);
            std::cout << grid_x << " " << grid_y << " " << block_x << std::endl;
            for (int n = 0; n < cycle_count; ++n) {
                cuda_kernel_sgemm_100<<<grid_d, block_d>>>(dev_at, dev_b, dev_c, M, N, K, alpha, beta);
                // cuda_kernel_sgemm_100_tex<<<grid_d, block_d>>>(dev_at, dev_b, dev_c, M, N, K, alpha, beta);
                // cuda_kernel_sgemm_100_v2<<<grid_d, block_d>>>(dev_at, dev_b, dev_c, M, N, K, alpha, beta);
            }
            break;
        }
    }
    hipEventCreate(&stop);
    hipEventRecord(stop, NULL);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&msecTotal, start, stop);
//    hipDeviceSynchronize();
    printf("Processing time: %f (ms), GFLOPS: %f \n", msecTotal, cycle_count * flop / msecTotal/ 1e+6);
//    timer.toc("gpu sgemm");

    float* ct = (float*)malloc(M * N * sizeof(float));
    if (kernel_type == 't') {
        hipMemcpy(c, D, M * N * sizeof(float), hipMemcpyDeviceToHost);
    } else {
        if (kernel_type == 100) {
            hipMemcpy(ct, dev_c, M * N * sizeof(float), hipMemcpyDeviceToHost);
            for (int i = 0; i < N; ++i) {
                for (int j = 0; j < M; ++j) {
                    c[j * N + i] = ct[i * M + j];
                }
            }
        } else {
            hipMemcpy(c, dev_c, M * N * sizeof(float), hipMemcpyDeviceToHost);
        }
    }

    if (kernel_type == 'b') hipblasDestroy(handle);
    if (kernel_type == 't') {
        free(B_ht);
        checkCudaErrors(hipFree((void*)A));
        checkCudaErrors(hipFree((void*)B));
        checkCudaErrors(hipFree((void*)C));
        checkCudaErrors(hipFree((void*)D));
    }

    free(at);
    free(ct);
    hipFree(dev_a);
    hipFree(dev_at);
    hipFree(dev_b);
    hipFree(dev_c);
}

void gpu_warmup()
{
    float *dev_p = 0;

    hs_timer timer;
    timer.tic("gpu warmup");

    hipMalloc((void **)&dev_p, 16 * 32 * sizeof(float));

    cuda_kernel_warmup<<<16, 32>>>(dev_p);

    hipDeviceSynchronize();

    hipFree(dev_p);

    timer.toc("gpu warmup");
}

//void cpu_kernel_sgemm_0(float *a, float *b, float *c, size_t N, size_t M, size_t K, float alpha, float beta) {
//    for (int m = 0; m < M; ++m) {
//        for (int n = 0; n < N; ++n) {
//            float acc = 0.0f;
//            for (int k = 0; k < K; ++k) {
//                acc += a[m * K + k] * b[k * N + n];
//            }
//            c[m * N + n] = alpha * acc + beta * c[m * N + n];
//        }
//    }
//}

void cpu_sgemm(
        float *a, float *b, float *c,
        size_t N, size_t M, size_t K,
        float alpha, float beta, int kernel_type)
{
    hs_timer timer;
    timer.tic("cpu sgemm");

    switch (kernel_type)
    {
        case 0:
        {
            cpu_kernel_sgemm_0(a, b, c, N, M, K, alpha, beta);
            break;
        }
        case 'm':
        {
            cblas_sgemm(CblasRowMajor, CblasNoTrans, CblasNoTrans, M, N, K, alpha, a, K, b, N, beta, c, N);
            break;
        }
    }
    timer.toc("cpu sgemm");
}

void cpu_warmup() {
    hs_timer timer;
    timer.tic("cpu warmup");

    const size_t arr_size = 1024;
    float *p = new float[arr_size];

#pragma omp parallel for simd
    for (size_t i = 0; i < arr_size; i++)
    {
        float f = (float)i;
        p[i] = f * f * f;
    }

    delete p;

    timer.toc("cpu warmup");
}


__device__ void sgemm_block_64x64_tex(
        float *a, float *b, float *c,
        size_t M, size_t N, size_t K,
        float alpha, float beta) {

    __shared__ float a_b_shm[2 * 16 * 64];

    int block_x = blockIdx.x;
    int block_y = blockIdx.y;
    // float* a = pa + block_y * 64;
    // float* b = pb + block_x * 64;
    // float* c = pc + block_x * 64 * M + block_y * 64;

    // int tid = threadIdx.x & 0x3f;
    int tid = threadIdx.x;
    int ldx = tid >= 32 ? N : M;
    int tid2 = (tid >> 4) & 1;
    // int tid15 = tid & 0xf;
    int tid15 = tid & 15;
    int a_b_offset = tid >= 32 ? block_x * 64 : block_y * 64;
    int track0 = a_b_offset + tid2 * ldx + tid15 * 4;
    int track2 = track0 + 2 * ldx;
    int track4 = track0 + 4 * ldx;
    int track6 = track0 + 6 * ldx;
    int end = track0 + (K - 8) * ldx;
    int write_offset = tid2 * 64 + tid15 * 4;
    write_offset += tid >= 32 ? 512 : 0;

    int readAs = ((tid >> 1) & 7) << 2;
    int readBs = ((((tid & 0x30) >> 3) | (tid & 1)) << 2) + 512;

#if USE_TEXTURE
    floatTex tex = tid >= 32 ? tex1DRefB : tex1DRefA;
#else
    float* read_addr = tid >= 32 ? b : a;
#endif

    float cbb00=0, cbb01=0, cbb02=0, cbb03=0;
    float cbb10=0, cbb11=0, cbb12=0, cbb13=0;
    float cbb20=0, cbb21=0, cbb22=0, cbb23=0;
    float cbb30=0, cbb31=0, cbb32=0, cbb33=0;
    float cba00=0, cba01=0, cba02=0, cba03=0;
    float cba10=0, cba11=0, cba12=0, cba13=0;
    float cba20=0, cba21=0, cba22=0, cba23=0;
    float cba30=0, cba31=0, cba32=0, cba33=0;
    float cab00=0, cab01=0, cab02=0, cab03=0;
    float cab10=0, cab11=0, cab12=0, cab13=0;
    float cab20=0, cab21=0, cab22=0, cab23=0;
    float cab30=0, cab31=0, cab32=0, cab33=0;
    float caa00=0, caa01=0, caa02=0, caa03=0;
    float caa10=0, caa11=0, caa12=0, caa13=0;
    float caa20=0, caa21=0, caa22=0, caa23=0;
    float caa30=0, caa31=0, caa32=0, caa33=0;

    // float cbb00, cbb01, cbb02, cbb03;
    // float cbb10, cbb11, cbb12, cbb13;
    // float cbb20, cbb21, cbb22, cbb23;
    // float cbb30, cbb31, cbb32, cbb33;
    // float cba00, cba01, cba02, cba03;
    // float cba10, cba11, cba12, cba13;
    // float cba20, cba21, cba22, cba23;
    // float cba30, cba31, cba32, cba33;
    // float cab00, cab01, cab02, cab03;
    // float cab10, cab11, cab12, cab13;
    // float cab20, cab21, cab22, cab23;
    // float cab30, cab31, cab32, cab33;
    // float caa00, caa01, caa02, caa03;
    // float caa10, caa11, caa12, caa13;
    // float caa20, caa21, caa22, caa23;
    // float caa30, caa31, caa32, caa33;

    float j0Ab00, j0Ab01, j0Ab02, j0Ab03;
    float j0Bb00, j0Bb01, j0Bb02, j0Bb03;
    float j0Aa00, j0Aa01, j0Aa02, j0Aa03;
    float j0Ba00, j0Ba01, j0Ba02, j0Ba03;
    // float j1Ab00, j1Ab01, j1Ab02, j1Ab03;
    // float j1Bb00, j1Bb01, j1Bb02, j1Bb03;
    // float j1Aa00, j1Aa01, j1Aa02, j1Aa03;
    // float j1Ba00, j1Ba01, j1Ba02, j1Ba03;

    // float j0Ab00=1, j0Ab01=1, j0Ab02=1, j0Ab03=1;
    // float j0Bb00=1, j0Bb01=1, j0Bb02=1, j0Bb03=1;
    // float j0Aa00=1, j0Aa01=1, j0Aa02=1, j0Aa03=1;
    // float j0Ba00=1, j0Ba01=1, j0Ba02=1, j0Ba03=1;
    // float j1Ab00=1, j1Ab01=1, j1Ab02=1, j1Ab03=1;
    // float j1Bb00=1, j1Bb01=1, j1Bb02=1, j1Bb03=1;
    // float j1Aa00=1, j1Aa01=1, j1Aa02=1, j1Aa03=1;
    // float j1Ba00=1, j1Ba01=1, j1Ba02=1, j1Ba03=1;

    while (track0 <= end) {
#if USE_TEXTURE
        a_b_shm[write_offset + 0 * 64 + 0] = tex1Dfetch(tex, track0 + 0);
        a_b_shm[write_offset + 0 * 64 + 1] = tex1Dfetch(tex, track0 + 1);
        a_b_shm[write_offset + 0 * 64 + 2] = tex1Dfetch(tex, track0 + 2);
        a_b_shm[write_offset + 0 * 64 + 3] = tex1Dfetch(tex, track0 + 3);
        a_b_shm[write_offset + 2 * 64 + 0] = tex1Dfetch(tex, track2 + 0);
        a_b_shm[write_offset + 2 * 64 + 1] = tex1Dfetch(tex, track2 + 1);
        a_b_shm[write_offset + 2 * 64 + 2] = tex1Dfetch(tex, track2 + 2);
        a_b_shm[write_offset + 2 * 64 + 3] = tex1Dfetch(tex, track2 + 3);
        a_b_shm[write_offset + 4 * 64 + 0] = tex1Dfetch(tex, track4 + 0);
        a_b_shm[write_offset + 4 * 64 + 1] = tex1Dfetch(tex, track4 + 1);
        a_b_shm[write_offset + 4 * 64 + 2] = tex1Dfetch(tex, track4 + 2);
        a_b_shm[write_offset + 4 * 64 + 3] = tex1Dfetch(tex, track4 + 3);
        a_b_shm[write_offset + 6 * 64 + 0] = tex1Dfetch(tex, track6 + 0);
        a_b_shm[write_offset + 6 * 64 + 1] = tex1Dfetch(tex, track6 + 1);
        a_b_shm[write_offset + 6 * 64 + 2] = tex1Dfetch(tex, track6 + 2);
        a_b_shm[write_offset + 6 * 64 + 3] = tex1Dfetch(tex, track6 + 3);
#else
        a_b_shm[write_offset + 0 * 64 + 0] = read_addr[track0 + 0];
        a_b_shm[write_offset + 0 * 64 + 1] = read_addr[track0 + 1];
        a_b_shm[write_offset + 0 * 64 + 2] = read_addr[track0 + 2];
        a_b_shm[write_offset + 0 * 64 + 3] = read_addr[track0 + 3];
        a_b_shm[write_offset + 2 * 64 + 0] = read_addr[track2 + 0];
        a_b_shm[write_offset + 2 * 64 + 1] = read_addr[track2 + 1];
        a_b_shm[write_offset + 2 * 64 + 2] = read_addr[track2 + 2];
        a_b_shm[write_offset + 2 * 64 + 3] = read_addr[track2 + 3];
        a_b_shm[write_offset + 4 * 64 + 0] = read_addr[track4 + 0];
        a_b_shm[write_offset + 4 * 64 + 1] = read_addr[track4 + 1];
        a_b_shm[write_offset + 4 * 64 + 2] = read_addr[track4 + 2];
        a_b_shm[write_offset + 4 * 64 + 3] = read_addr[track4 + 3];
        a_b_shm[write_offset + 6 * 64 + 0] = read_addr[track6 + 0];
        a_b_shm[write_offset + 6 * 64 + 1] = read_addr[track6 + 1];
        a_b_shm[write_offset + 6 * 64 + 2] = read_addr[track6 + 2];
        a_b_shm[write_offset + 6 * 64 + 3] = read_addr[track6 + 3];
#endif
        __syncthreads();
        // __syncwarp(0xFFFFFFFF);

        write_offset ^= 16 * 64;
        track0 += 8 * ldx;
        track2 += 8 * ldx;
        track4 += 8 * ldx;
        track6 += 8 * ldx;

        for (int j = 0; j < 8; ++j) {
            // int prefetch = (j + 1) % 8;
            int prefetch = j;

            j0Ab00 = a_b_shm[readAs + prefetch * 64 + 0];
            j0Ab01 = a_b_shm[readAs + prefetch * 64 + 1];
            j0Ab02 = a_b_shm[readAs + prefetch * 64 + 2];
            j0Ab03 = a_b_shm[readAs + prefetch * 64 + 3];

            j0Bb00 = a_b_shm[readBs + prefetch * 64 + 0];
            j0Bb01 = a_b_shm[readBs + prefetch * 64 + 1];
            j0Bb02 = a_b_shm[readBs + prefetch * 64 + 2];
            j0Bb03 = a_b_shm[readBs + prefetch * 64 + 3];

            j0Aa00 = a_b_shm[readAs + prefetch * 64 + 32 + 0];
            j0Aa01 = a_b_shm[readAs + prefetch * 64 + 32 + 1];
            j0Aa02 = a_b_shm[readAs + prefetch * 64 + 32 + 2];
            j0Aa03 = a_b_shm[readAs + prefetch * 64 + 32 + 3];

            j0Ba00 = a_b_shm[readBs + prefetch * 64 + 32 + 0];
            j0Ba01 = a_b_shm[readBs + prefetch * 64 + 32 + 1];
            j0Ba02 = a_b_shm[readBs + prefetch * 64 + 32 + 2];
            j0Ba03 = a_b_shm[readBs + prefetch * 64 + 32 + 3];

            cbb00 += j0Bb00 * j0Ab00;
            cbb01 += j0Bb00 * j0Ab01;
            // j1Ab00 = a_b_shm[readAs + prefetch * 64 + 0];
            // j1Ab01 = a_b_shm[readAs + prefetch * 64 + 1];
            // j1Ab02 = a_b_shm[readAs + prefetch * 64 + 2];
            // j1Ab03 = a_b_shm[readAs + prefetch * 64 + 3];
            cbb02 += j0Bb00 * j0Ab02;
            cbb03 += j0Bb00 * j0Ab03;
            // j1Bb00 = a_b_shm[readBs + prefetch * 64 + 0];
            // j1Bb01 = a_b_shm[readBs + prefetch * 64 + 1];
            // j1Bb02 = a_b_shm[readBs + prefetch * 64 + 2];
            // j1Bb03 = a_b_shm[readBs + prefetch * 64 + 3];

            cbb10 += j0Bb01 * j0Ab00;
            cbb11 += j0Bb01 * j0Ab01;
            // j1Aa00 = a_b_shm[readAs + prefetch * 64 + 32 + 0];
            // j1Aa01 = a_b_shm[readAs + prefetch * 64 + 32 + 1];
            // j1Aa02 = a_b_shm[readAs + prefetch * 64 + 32 + 2];
            // j1Aa03 = a_b_shm[readAs + prefetch * 64 + 32 + 3];
            cbb12 += j0Bb01 * j0Ab02;
            cbb13 += j0Bb01 * j0Ab03;
            // j1Ba00 = a_b_shm[readBs + prefetch * 64 + 32 + 0];
            // j1Ba01 = a_b_shm[readBs + prefetch * 64 + 32 + 1];
            // j1Ba02 = a_b_shm[readBs + prefetch * 64 + 32 + 2];
            // j1Ba03 = a_b_shm[readBs + prefetch * 64 + 32 + 3];

            cbb20 += j0Bb02 * j0Ab00;
            cbb21 += j0Bb02 * j0Ab01;
            cbb22 += j0Bb02 * j0Ab02;
            cbb23 += j0Bb02 * j0Ab03;

            cbb30 += j0Bb03 * j0Ab00;
            cbb31 += j0Bb03 * j0Ab01;
            cbb32 += j0Bb03 * j0Ab02;
            cbb33 += j0Bb03 * j0Ab03;

            cba00 += j0Ba00 * j0Ab00;
            cba01 += j0Ba00 * j0Ab01;
            cba02 += j0Ba00 * j0Ab02;
            cba03 += j0Ba00 * j0Ab03;

            cba10 += j0Ba01 * j0Ab00;
            cba11 += j0Ba01 * j0Ab01;
            cba12 += j0Ba01 * j0Ab02;
            cba13 += j0Ba01 * j0Ab03;

            cba20 += j0Ba02 * j0Ab00;
            cba21 += j0Ba02 * j0Ab01;
            cba22 += j0Ba02 * j0Ab02;
            cba23 += j0Ba02 * j0Ab03;

            cba30 += j0Ba03 * j0Ab00;
            cba31 += j0Ba03 * j0Ab01;
            cba32 += j0Ba03 * j0Ab02;
            cba33 += j0Ba03 * j0Ab03;

            cab00 += j0Bb00 * j0Aa00;
            cab01 += j0Bb00 * j0Aa01;
            cab02 += j0Bb00 * j0Aa02;
            cab03 += j0Bb00 * j0Aa03;

            cab10 += j0Bb01 * j0Aa00;
            cab11 += j0Bb01 * j0Aa01;
            cab12 += j0Bb01 * j0Aa02;
            cab13 += j0Bb01 * j0Aa03;

            cab20 += j0Bb02 * j0Aa00;
            cab21 += j0Bb02 * j0Aa01;
            cab22 += j0Bb02 * j0Aa02;
            cab23 += j0Bb02 * j0Aa03;

            cab30 += j0Bb03 * j0Aa00;
            cab31 += j0Bb03 * j0Aa01;
            cab32 += j0Bb03 * j0Aa02;
            cab33 += j0Bb03 * j0Aa03;

            caa00 += j0Ba00 * j0Aa00;
            caa01 += j0Ba00 * j0Aa01;
            caa02 += j0Ba00 * j0Aa02;
            caa03 += j0Ba00 * j0Aa03;

            caa10 += j0Ba01 * j0Aa00;
            caa11 += j0Ba01 * j0Aa01;
            caa12 += j0Ba01 * j0Aa02;
            caa13 += j0Ba01 * j0Aa03;

            caa20 += j0Ba02 * j0Aa00;
            caa21 += j0Ba02 * j0Aa01;
            caa22 += j0Ba02 * j0Aa02;
            caa23 += j0Ba02 * j0Aa03;

            caa30 += j0Ba03 * j0Aa00;
            caa31 += j0Ba03 * j0Aa01;
            caa32 += j0Ba03 * j0Aa02;
            caa33 += j0Ba03 * j0Aa03;
        }

        readAs ^= 16 * 64;
        readBs ^= 16 * 64;
    }
    __syncthreads();


    int tid31 = tid & 31;
    int tid32 = tid & 32;
    int coord_x = readBs & 0x7f;
    int coord_y = readAs & 0x7f;
    int writeCs = coord_x / 4 * 64 + coord_y;
    int readCs = (tid32 << 3) + tid31;
    int ldc4 = M * 4;
    int Cy00 = block_x * 64 * M + block_y * 64 + (tid32 >> 1) * M + tid31;
    int Cy04 = Cy00 + ldc4;
    int Cy08 = Cy00 + 2 * ldc4;
    int Cy12 = Cy00 + 3 * ldc4;

    a_b_shm[writeCs + 0] = cbb00;
    a_b_shm[writeCs + 1] = cbb01;
    a_b_shm[writeCs + 2] = cbb02;
    a_b_shm[writeCs + 3] = cbb03;
    a_b_shm[writeCs + 32 + 0] = cab00;
    a_b_shm[writeCs + 32 + 1] = cab01;
    a_b_shm[writeCs + 32 + 2] = cab02;
    a_b_shm[writeCs + 32 + 3] = cab03;
    // if (threadIdx.x == 1) {
    //     printf("reg r0,  c4: %f\n", cbb00);
    // }
    // if (threadIdx.x == 18) {
    //     printf("reg r7,  c8: %f\n", cbb03);
    //     printf("reg r39, c8: %f\n", cab03);
    // }

    cbb00 = a_b_shm[readCs + 0 * 64 + 0 ];
    cbb01 = a_b_shm[readCs + 0 * 64 + 32];
    cbb02 = a_b_shm[readCs + 1 * 64 + 0 ];
    cbb03 = a_b_shm[readCs + 1 * 64 + 32];
    cab00 = a_b_shm[readCs + 2 * 64 + 0 ];
    cab01 = a_b_shm[readCs + 2 * 64 + 32];
    cab02 = a_b_shm[readCs + 3 * 64 + 0 ];
    cab03 = a_b_shm[readCs + 3 * 64 + 32];
    c[Cy00 + 0 ] = cbb00;
    c[Cy00 + 32] = cbb01;
    c[Cy04 + 0 ] = cbb02;
    c[Cy04 + 32] = cbb03;
    c[Cy08 + 0 ] = cab00;
    c[Cy08 + 32] = cab01;
    c[Cy12 + 0 ] = cab02;
    c[Cy12 + 32] = cab03;

    Cy00 += M;
    Cy04 += M;
    Cy08 += M;
    Cy12 += M;
    a_b_shm[writeCs + 0] = cbb10;
    a_b_shm[writeCs + 1] = cbb11;
    a_b_shm[writeCs + 2] = cbb12;
    a_b_shm[writeCs + 3] = cbb13;
    a_b_shm[writeCs + 32 + 0] = cab10;
    a_b_shm[writeCs + 32 + 1] = cab11;
    a_b_shm[writeCs + 32 + 2] = cab12;
    a_b_shm[writeCs + 32 + 3] = cab13;

    cbb10 = a_b_shm[readCs + 0 * 64 + 0 ];
    cbb11 = a_b_shm[readCs + 0 * 64 + 32];
    cbb12 = a_b_shm[readCs + 1 * 64 + 0 ];
    cbb13 = a_b_shm[readCs + 1 * 64 + 32];
    cab10 = a_b_shm[readCs + 2 * 64 + 0 ];
    cab11 = a_b_shm[readCs + 2 * 64 + 32];
    cab12 = a_b_shm[readCs + 3 * 64 + 0 ];
    cab13 = a_b_shm[readCs + 3 * 64 + 32];
    c[Cy00 + 0 ] = cbb10;
    c[Cy00 + 32] = cbb11;
    c[Cy04 + 0 ] = cbb12;
    c[Cy04 + 32] = cbb13;
    c[Cy08 + 0 ] = cab10;
    c[Cy08 + 32] = cab11;
    c[Cy12 + 0 ] = cab12;
    c[Cy12 + 32] = cab13;

    Cy00 += M;
    Cy04 += M;
    Cy08 += M;
    Cy12 += M;
    a_b_shm[writeCs + 0] = cbb20;
    a_b_shm[writeCs + 1] = cbb21;
    a_b_shm[writeCs + 2] = cbb22;
    a_b_shm[writeCs + 3] = cbb23;
    a_b_shm[writeCs + 32 + 0] = cab20;
    a_b_shm[writeCs + 32 + 1] = cab21;
    a_b_shm[writeCs + 32 + 2] = cab22;
    a_b_shm[writeCs + 32 + 3] = cab23;

    cbb20 = a_b_shm[readCs + 0 * 64 + 0 ];
    cbb21 = a_b_shm[readCs + 0 * 64 + 32];
    cbb22 = a_b_shm[readCs + 1 * 64 + 0 ];
    cbb23 = a_b_shm[readCs + 1 * 64 + 32];
    cab20 = a_b_shm[readCs + 2 * 64 + 0 ];
    cab21 = a_b_shm[readCs + 2 * 64 + 32];
    cab22 = a_b_shm[readCs + 3 * 64 + 0 ];
    cab23 = a_b_shm[readCs + 3 * 64 + 32];
    c[Cy00 + 0 ] = cbb20;
    c[Cy00 + 32] = cbb21;
    c[Cy04 + 0 ] = cbb22;
    c[Cy04 + 32] = cbb23;
    c[Cy08 + 0 ] = cab20;
    c[Cy08 + 32] = cab21;
    c[Cy12 + 0 ] = cab22;
    c[Cy12 + 32] = cab23;

    Cy00 += M;
    Cy04 += M;
    Cy08 += M;
    Cy12 += M;
    a_b_shm[writeCs + 0] = cbb30;
    a_b_shm[writeCs + 1] = cbb31;
    a_b_shm[writeCs + 2] = cbb32;
    a_b_shm[writeCs + 3] = cbb33;
    a_b_shm[writeCs + 32 + 0] = cab30;
    a_b_shm[writeCs + 32 + 1] = cab31;
    a_b_shm[writeCs + 32 + 2] = cab32;
    a_b_shm[writeCs + 32 + 3] = cab33;

    cbb30 = a_b_shm[readCs + 0 * 64 + 0 ];
    cbb31 = a_b_shm[readCs + 0 * 64 + 32];
    cbb32 = a_b_shm[readCs + 1 * 64 + 0 ];
    cbb33 = a_b_shm[readCs + 1 * 64 + 32];
    cab30 = a_b_shm[readCs + 2 * 64 + 0 ];
    cab31 = a_b_shm[readCs + 2 * 64 + 32];
    cab32 = a_b_shm[readCs + 3 * 64 + 0 ];
    cab33 = a_b_shm[readCs + 3 * 64 + 32];
    c[Cy00 + 0 ] = cbb30;
    c[Cy00 + 32] = cbb31;
    c[Cy04 + 0 ] = cbb32;
    c[Cy04 + 32] = cbb33;
    c[Cy08 + 0 ] = cab30;
    c[Cy08 + 32] = cab31;
    c[Cy12 + 0 ] = cab32;
    c[Cy12 + 32] = cab33;

    Cy00 += M;
    Cy04 += M;
    Cy08 += M;
    Cy12 += M;

    Cy00 += 28 * M;
    Cy04 += 28 * M;
    Cy08 += 28 * M;
    Cy12 += 28 * M;

    a_b_shm[writeCs + 0] = cba00;
    a_b_shm[writeCs + 1] = cba01;
    a_b_shm[writeCs + 2] = cba02;
    a_b_shm[writeCs + 3] = cba03;
    a_b_shm[writeCs + 32 + 0] = caa00;
    a_b_shm[writeCs + 32 + 1] = caa01;
    a_b_shm[writeCs + 32 + 2] = caa02;
    a_b_shm[writeCs + 32 + 3] = caa03;

    cba00 = a_b_shm[readCs + 0 * 64 + 0 ];
    cba01 = a_b_shm[readCs + 0 * 64 + 32];
    cba02 = a_b_shm[readCs + 1 * 64 + 0 ];
    cba03 = a_b_shm[readCs + 1 * 64 + 32];
    caa00 = a_b_shm[readCs + 2 * 64 + 0 ];
    caa01 = a_b_shm[readCs + 2 * 64 + 32];
    caa02 = a_b_shm[readCs + 3 * 64 + 0 ];
    caa03 = a_b_shm[readCs + 3 * 64 + 32];
    c[Cy00 + 0 ] = cba00;
    c[Cy00 + 32] = cba01;
    c[Cy04 + 0 ] = cba02;
    c[Cy04 + 32] = cba03;
    c[Cy08 + 0 ] = caa00;
    c[Cy08 + 32] = caa01;
    c[Cy12 + 0 ] = caa02;
    c[Cy12 + 32] = caa03;

    Cy00 += M;
    Cy04 += M;
    Cy08 += M;
    Cy12 += M;
    a_b_shm[writeCs + 0] = cba10;
    a_b_shm[writeCs + 1] = cba11;
    a_b_shm[writeCs + 2] = cba12;
    a_b_shm[writeCs + 3] = cba13;
    a_b_shm[writeCs + 32 + 0] = caa10;
    a_b_shm[writeCs + 32 + 1] = caa11;
    a_b_shm[writeCs + 32 + 2] = caa12;
    a_b_shm[writeCs + 32 + 3] = caa13;

    cba10 = a_b_shm[readCs + 0 * 64 + 0 ];
    cba11 = a_b_shm[readCs + 0 * 64 + 32];
    cba12 = a_b_shm[readCs + 1 * 64 + 0 ];
    cba13 = a_b_shm[readCs + 1 * 64 + 32];
    caa10 = a_b_shm[readCs + 2 * 64 + 0 ];
    caa11 = a_b_shm[readCs + 2 * 64 + 32];
    caa12 = a_b_shm[readCs + 3 * 64 + 0 ];
    caa13 = a_b_shm[readCs + 3 * 64 + 32];
    c[Cy00 + 0 ] = cba10;
    c[Cy00 + 32] = cba11;
    c[Cy04 + 0 ] = cba12;
    c[Cy04 + 32] = cba13;
    c[Cy08 + 0 ] = caa10;
    c[Cy08 + 32] = caa11;
    c[Cy12 + 0 ] = caa12;
    c[Cy12 + 32] = caa13;

    Cy00 += M;
    Cy04 += M;
    Cy08 += M;
    Cy12 += M;
    a_b_shm[writeCs + 0] = cba20;
    a_b_shm[writeCs + 1] = cba21;
    a_b_shm[writeCs + 2] = cba22;
    a_b_shm[writeCs + 3] = cba23;
    a_b_shm[writeCs + 32 + 0] = caa20;
    a_b_shm[writeCs + 32 + 1] = caa21;
    a_b_shm[writeCs + 32 + 2] = caa22;
    a_b_shm[writeCs + 32 + 3] = caa23;

    cba20 = a_b_shm[readCs + 0 * 64 + 0 ];
    cba21 = a_b_shm[readCs + 0 * 64 + 32];
    cba22 = a_b_shm[readCs + 1 * 64 + 0 ];
    cba23 = a_b_shm[readCs + 1 * 64 + 32];
    caa20 = a_b_shm[readCs + 2 * 64 + 0 ];
    caa21 = a_b_shm[readCs + 2 * 64 + 32];
    caa22 = a_b_shm[readCs + 3 * 64 + 0 ];
    caa23 = a_b_shm[readCs + 3 * 64 + 32];
    c[Cy00 + 0 ] = cba20;
    c[Cy00 + 32] = cba21;
    c[Cy04 + 0 ] = cba22;
    c[Cy04 + 32] = cba23;
    c[Cy08 + 0 ] = caa20;
    c[Cy08 + 32] = caa21;
    c[Cy12 + 0 ] = caa22;
    c[Cy12 + 32] = caa23;

    Cy00 += M;
    Cy04 += M;
    Cy08 += M;
    Cy12 += M;
    a_b_shm[writeCs + 0] = cba30;
    a_b_shm[writeCs + 1] = cba31;
    a_b_shm[writeCs + 2] = cba32;
    a_b_shm[writeCs + 3] = cba33;
    a_b_shm[writeCs + 32 + 0] = caa30;
    a_b_shm[writeCs + 32 + 1] = caa31;
    a_b_shm[writeCs + 32 + 2] = caa32;
    a_b_shm[writeCs + 32 + 3] = caa33;

    cba30 = a_b_shm[readCs + 0 * 64 + 0 ];
    cba31 = a_b_shm[readCs + 0 * 64 + 32];
    cba32 = a_b_shm[readCs + 1 * 64 + 0 ];
    cba33 = a_b_shm[readCs + 1 * 64 + 32];
    caa30 = a_b_shm[readCs + 2 * 64 + 0 ];
    caa31 = a_b_shm[readCs + 2 * 64 + 32];
    caa32 = a_b_shm[readCs + 3 * 64 + 0 ];
    caa33 = a_b_shm[readCs + 3 * 64 + 32];
    c[Cy00 + 0 ] = cba30;
    c[Cy00 + 32] = cba31;
    c[Cy04 + 0 ] = cba32;
    c[Cy04 + 32] = cba33;
    c[Cy08 + 0 ] = caa30;
    c[Cy08 + 32] = caa31;
    c[Cy12 + 0 ] = caa32;
    c[Cy12 + 32] = caa33;
}

__global__ void cuda_kernel_sgemm_100_tex(
        float *a, float *b, float *c,
        size_t M, size_t N, size_t K,
        float alpha, float beta) {
    sgemm_block_64x64_tex(a, b, c, M, N, K, alpha, beta);
}
