#include "hip/hip_runtime.h"
#include <iostream>

#include "matmul_kernel.h"
#include <hip/hip_runtime.h>
#include <>

__device__ float aa[10];

__device__ void sgemm_block_64x64_v2(
        float *a, float *b, float *c,
        size_t M, size_t N, size_t K,
        float alpha, float beta) {

    // float* dev_p = nullptr;
    // hipMalloc((void **)&dev_p, 16 * 32 * sizeof(float));
    // float aa[100000];
    // for (int i = threadIdx.x; i < 10000; i += blockDim.x) {
    //     aa[i] = i;
    // }

    __shared__ float a_b_shm[2 * 16 * 64];

    // int tid = threadIdx.x & 0x3f;
    int tid = threadIdx.x;
    int ldx = tid >= 32 ? N : M;
    int tid2 = (tid >> 4) & 1;
    // int tid15 = tid & 0xf;
    int tid15 = tid & 15;
    int track0 = tid2 * ldx + tid15 * 4;
    int track2 = track0 + 2 * ldx;
    int track4 = track0 + 4 * ldx;
    int track6 = track0 + 6 * ldx;
    // int end = track0 + (K - 8) * ldx;
    int end = track0 + K * ldx;
    int write_offset = tid2 * 64 + tid15 * 4;
    write_offset += tid >= 32 ? 512 : 0;

    int readAs = ((tid >> 1) & 7) << 2;
    int readBs = ((((tid & 0x30) >> 3) | (tid & 1)) << 2) + 512;

    float* read_addr = tid >= 32 ? b : a;

    float cbb00=0, cbb01=0, cbb02=0, cbb03=0;
    float cbb10=0, cbb11=0, cbb12=0, cbb13=0;
    float cbb20=0, cbb21=0, cbb22=0, cbb23=0;
    float cbb30=0, cbb31=0, cbb32=0, cbb33=0;

    float cba00=0, cba01=0, cba02=0, cba03=0;
    float cba10=0, cba11=0, cba12=0, cba13=0;
    float cba20=0, cba21=0, cba22=0, cba23=0;
    float cba30=0, cba31=0, cba32=0, cba33=0;

    float cab00=0, cab01=0, cab02=0, cab03=0;
    float cab10=0, cab11=0, cab12=0, cab13=0;
    float cab20=0, cab21=0, cab22=0, cab23=0;
    float cab30=0, cab31=0, cab32=0, cab33=0;

    float caa00=0, caa01=0, caa02=0, caa03=0;
    float caa10=0, caa11=0, caa12=0, caa13=0;
    float caa20=0, caa21=0, caa22=0, caa23=0;
    float caa30=0, caa31=0, caa32=0, caa33=0;

    float j0Ab00, j0Ab01, j0Ab02, j0Ab03;
    float j0Bb00, j0Bb01, j0Bb02, j0Bb03;
    float j0Aa00, j0Aa01, j0Aa02, j0Aa03;
    float j0Ba00, j0Ba01, j0Ba02, j0Ba03;

    float j1Ab00, j1Ab01, j1Ab02, j1Ab03;
    float j1Bb00, j1Bb01, j1Bb02, j1Bb03;
    float j1Aa00, j1Aa01, j1Aa02, j1Aa03;
    float j1Ba00, j1Ba01, j1Ba02, j1Ba03;

    float loadX00, loadX01, loadX02, loadX03;
    float loadX20, loadX21, loadX22, loadX23;
    float loadX40, loadX41, loadX42, loadX43;
    float loadX60, loadX61, loadX62, loadX63;

    a_b_shm[write_offset + 0 * 64 + 0] = read_addr[track0 + 0];
    a_b_shm[write_offset + 0 * 64 + 1] = read_addr[track0 + 1];
    a_b_shm[write_offset + 0 * 64 + 2] = read_addr[track0 + 2];
    a_b_shm[write_offset + 0 * 64 + 3] = read_addr[track0 + 3];

    a_b_shm[write_offset + 2 * 64 + 0] = read_addr[track2 + 0];
    a_b_shm[write_offset + 2 * 64 + 1] = read_addr[track2 + 1];
    a_b_shm[write_offset + 2 * 64 + 2] = read_addr[track2 + 2];
    a_b_shm[write_offset + 2 * 64 + 3] = read_addr[track2 + 3];

    a_b_shm[write_offset + 4 * 64 + 0] = read_addr[track4 + 0];
    a_b_shm[write_offset + 4 * 64 + 1] = read_addr[track4 + 1];
    a_b_shm[write_offset + 4 * 64 + 2] = read_addr[track4 + 2];
    a_b_shm[write_offset + 4 * 64 + 3] = read_addr[track4 + 3];

    a_b_shm[write_offset + 6 * 64 + 0] = read_addr[track6 + 0];
    a_b_shm[write_offset + 6 * 64 + 1] = read_addr[track6 + 1];
    a_b_shm[write_offset + 6 * 64 + 2] = read_addr[track6 + 2];
    a_b_shm[write_offset + 6 * 64 + 3] = read_addr[track6 + 3];

    track0 += 8 * ldx;
    track2 += 8 * ldx;
    track4 += 8 * ldx;
    track6 += 8 * ldx;
    __syncthreads();
    write_offset ^= 16 * 64;

    j0Ab00 = a_b_shm[readAs + 0];
    j0Ab01 = a_b_shm[readAs + 1];
    j0Ab02 = a_b_shm[readAs + 2];
    j0Ab03 = a_b_shm[readAs + 3];

    j0Bb00 = a_b_shm[readBs + 0];
    j0Bb01 = a_b_shm[readBs + 1];
    j0Bb02 = a_b_shm[readBs + 2];
    j0Bb03 = a_b_shm[readBs + 3];

    j0Aa00 = a_b_shm[readAs + 32 + 0];
    j0Aa01 = a_b_shm[readAs + 32 + 1];
    j0Aa02 = a_b_shm[readAs + 32 + 2];
    j0Aa03 = a_b_shm[readAs + 32 + 3];

    j0Ba00 = a_b_shm[readBs + 32 + 0];
    j0Ba01 = a_b_shm[readBs + 32 + 1];
    j0Ba02 = a_b_shm[readBs + 32 + 2];
    j0Ba03 = a_b_shm[readBs + 32 + 3];

    while (track0 <= end) {
        // a_b_shm[write_offset + 0 * 64 + 0] = read_addr[track0 + 0];
        // a_b_shm[write_offset + 0 * 64 + 1] = read_addr[track0 + 1];
        // a_b_shm[write_offset + 0 * 64 + 2] = read_addr[track0 + 2];
        // a_b_shm[write_offset + 0 * 64 + 3] = read_addr[track0 + 3];

        // a_b_shm[write_offset + 2 * 64 + 0] = read_addr[track2 + 0];
        // a_b_shm[write_offset + 2 * 64 + 1] = read_addr[track2 + 1];
        // a_b_shm[write_offset + 2 * 64 + 2] = read_addr[track2 + 2];
        // a_b_shm[write_offset + 2 * 64 + 3] = read_addr[track2 + 3];

        // a_b_shm[write_offset + 4 * 64 + 0] = read_addr[track4 + 0];
        // a_b_shm[write_offset + 4 * 64 + 1] = read_addr[track4 + 1];
        // a_b_shm[write_offset + 4 * 64 + 2] = read_addr[track4 + 2];
        // a_b_shm[write_offset + 4 * 64 + 3] = read_addr[track4 + 3];

        // a_b_shm[write_offset + 6 * 64 + 0] = read_addr[track6 + 0];
        // a_b_shm[write_offset + 6 * 64 + 1] = read_addr[track6 + 1];
        // a_b_shm[write_offset + 6 * 64 + 2] = read_addr[track6 + 2];
        // a_b_shm[write_offset + 6 * 64 + 3] = read_addr[track6 + 3];

        // loadX00 = read_addr[track0 + 0];
        // loadX01 = read_addr[track0 + 1];
        // loadX02 = read_addr[track0 + 2];
        // loadX03 = read_addr[track0 + 3];
        // loadX20 = read_addr[track2 + 0];
        // loadX21 = read_addr[track2 + 1];
        // loadX22 = read_addr[track2 + 2];
        // loadX23 = read_addr[track2 + 3];
        // loadX40 = read_addr[track4 + 0];
        // loadX41 = read_addr[track4 + 1];
        // loadX42 = read_addr[track4 + 2];
        // loadX43 = read_addr[track4 + 3];
        // loadX60 = read_addr[track6 + 0];
        // loadX61 = read_addr[track6 + 1];
        // loadX62 = read_addr[track6 + 2];
        // loadX63 = read_addr[track6 + 3];

        // a_b_shm[write_offset + 0 * 64 + 0] = loadX00;
        // a_b_shm[write_offset + 0 * 64 + 1] = loadX01;
        // a_b_shm[write_offset + 0 * 64 + 2] = loadX02;
        // a_b_shm[write_offset + 0 * 64 + 3] = loadX03;
        // a_b_shm[write_offset + 2 * 64 + 0] = loadX20;
        // a_b_shm[write_offset + 2 * 64 + 1] = loadX21;
        // a_b_shm[write_offset + 2 * 64 + 2] = loadX22;
        // a_b_shm[write_offset + 2 * 64 + 3] = loadX23;
        // a_b_shm[write_offset + 4 * 64 + 0] = loadX40;
        // a_b_shm[write_offset + 4 * 64 + 1] = loadX41;
        // a_b_shm[write_offset + 4 * 64 + 2] = loadX42;
        // a_b_shm[write_offset + 4 * 64 + 3] = loadX43;
        // a_b_shm[write_offset + 6 * 64 + 0] = loadX60;
        // a_b_shm[write_offset + 6 * 64 + 1] = loadX61;
        // a_b_shm[write_offset + 6 * 64 + 2] = loadX62;
        // a_b_shm[write_offset + 6 * 64 + 3] = loadX63;

        cbb00 += j0Bb00 * j0Ab00;
        cbb01 += j0Bb00 * j0Ab01;
        j1Ab00 = a_b_shm[readAs + 1 * 64 + 0];
        j1Ab01 = a_b_shm[readAs + 1 * 64 + 1];
        j1Ab02 = a_b_shm[readAs + 1 * 64 + 2];
        j1Ab03 = a_b_shm[readAs + 1 * 64 + 3];
        cbb02 += j0Bb00 * j0Ab02;
        cbb03 += j0Bb00 * j0Ab03;
        j1Bb00 = a_b_shm[readBs + 1 * 64 + 0];
        j1Bb01 = a_b_shm[readBs + 1 * 64 + 1];
        j1Bb02 = a_b_shm[readBs + 1 * 64 + 2];
        j1Bb03 = a_b_shm[readBs + 1 * 64 + 3];
        cbb10 += j0Bb01 * j0Ab00;
        cbb11 += j0Bb01 * j0Ab01;
        j1Aa00 = a_b_shm[readAs + 1 * 64 + 32 + 0];
        j1Aa01 = a_b_shm[readAs + 1 * 64 + 32 + 1];
        j1Aa02 = a_b_shm[readAs + 1 * 64 + 32 + 2];
        j1Aa03 = a_b_shm[readAs + 1 * 64 + 32 + 3];
        cbb12 += j0Bb01 * j0Ab02;
        cbb13 += j0Bb01 * j0Ab03;
        j1Ba00 = a_b_shm[readBs + 1 * 64 + 32 + 0];
        j1Ba01 = a_b_shm[readBs + 1 * 64 + 32 + 1];
        j1Ba02 = a_b_shm[readBs + 1 * 64 + 32 + 2];
        j1Ba03 = a_b_shm[readBs + 1 * 64 + 32 + 3];

        cbb20 += j0Bb02 * j0Ab00;
        cbb21 += j0Bb02 * j0Ab01;
        cbb22 += j0Bb02 * j0Ab02;
        cbb23 += j0Bb02 * j0Ab03;
        cbb30 += j0Bb03 * j0Ab00;
        cbb31 += j0Bb03 * j0Ab01;
        cbb32 += j0Bb03 * j0Ab02;
        cbb33 += j0Bb03 * j0Ab03;
        cba00 += j0Ba00 * j0Ab00;
        cba01 += j0Ba00 * j0Ab01;
        cba02 += j0Ba00 * j0Ab02;
        cba03 += j0Ba00 * j0Ab03;
        cba10 += j0Ba01 * j0Ab00;
        cba11 += j0Ba01 * j0Ab01;
        cba12 += j0Ba01 * j0Ab02;
        cba13 += j0Ba01 * j0Ab03;
        cba20 += j0Ba02 * j0Ab00;
        cba21 += j0Ba02 * j0Ab01;
        cba22 += j0Ba02 * j0Ab02;
        cba23 += j0Ba02 * j0Ab03;
        cba30 += j0Ba03 * j0Ab00;
        cba31 += j0Ba03 * j0Ab01;
        cba32 += j0Ba03 * j0Ab02;
        cba33 += j0Ba03 * j0Ab03;

        loadX00 = read_addr[track0 + 0];
        loadX01 = read_addr[track0 + 1];
        loadX02 = read_addr[track0 + 2];
        loadX03 = read_addr[track0 + 3];
        cab00 += j0Bb00 * j0Aa00;
        cab01 += j0Bb00 * j0Aa01;
        loadX20 = read_addr[track2 + 0];
        loadX21 = read_addr[track2 + 1];
        loadX22 = read_addr[track2 + 2];
        loadX23 = read_addr[track2 + 3];
        cab02 += j0Bb00 * j0Aa02;
        cab03 += j0Bb00 * j0Aa03;
        cab10 += j0Bb01 * j0Aa00;
        cab11 += j0Bb01 * j0Aa01;
        cab12 += j0Bb01 * j0Aa02;
        cab13 += j0Bb01 * j0Aa03;
        cab20 += j0Bb02 * j0Aa00;
        cab21 += j0Bb02 * j0Aa01;
        cab22 += j0Bb02 * j0Aa02;
        cab23 += j0Bb02 * j0Aa03;
        cab30 += j0Bb03 * j0Aa00;
        cab31 += j0Bb03 * j0Aa01;
        cab32 += j0Bb03 * j0Aa02;
        cab33 += j0Bb03 * j0Aa03;
        caa00 += j0Ba00 * j0Aa00;
        caa01 += j0Ba00 * j0Aa01;
        caa02 += j0Ba00 * j0Aa02;
        caa03 += j0Ba00 * j0Aa03;
        caa10 += j0Ba01 * j0Aa00;
        caa11 += j0Ba01 * j0Aa01;
        caa12 += j0Ba01 * j0Aa02;
        caa13 += j0Ba01 * j0Aa03;
        caa20 += j0Ba02 * j0Aa00;
        caa21 += j0Ba02 * j0Aa01;
        caa22 += j0Ba02 * j0Aa02;
        caa23 += j0Ba02 * j0Aa03;
        caa30 += j0Ba03 * j0Aa00;
        caa31 += j0Ba03 * j0Aa01;
        caa32 += j0Ba03 * j0Aa02;
        caa33 += j0Ba03 * j0Aa03;


        cbb00 += j1Bb00 * j1Ab00;
        cbb01 += j1Bb00 * j1Ab01;
        j0Ab00 = a_b_shm[readAs + 2 * 64 + 0];
        j0Ab01 = a_b_shm[readAs + 2 * 64 + 1];
        j0Ab02 = a_b_shm[readAs + 2 * 64 + 2];
        j0Ab03 = a_b_shm[readAs + 2 * 64 + 3];
        cbb02 += j1Bb00 * j1Ab02;
        cbb03 += j1Bb00 * j1Ab03;
        j0Bb00 = a_b_shm[readBs + 2 * 64 + 0];
        j0Bb01 = a_b_shm[readBs + 2 * 64 + 1];
        j0Bb02 = a_b_shm[readBs + 2 * 64 + 2];
        j0Bb03 = a_b_shm[readBs + 2 * 64 + 3];
        cbb10 += j1Bb01 * j1Ab00;
        cbb11 += j1Bb01 * j1Ab01;
        j0Aa00 = a_b_shm[readAs + 2 * 64 + 32 + 0];
        j0Aa01 = a_b_shm[readAs + 2 * 64 + 32 + 1];
        j0Aa02 = a_b_shm[readAs + 2 * 64 + 32 + 2];
        j0Aa03 = a_b_shm[readAs + 2 * 64 + 32 + 3];
        cbb12 += j1Bb01 * j1Ab02;
        cbb13 += j1Bb01 * j1Ab03;
        j0Ba00 = a_b_shm[readBs + 2 * 64 + 32 + 0];
        j0Ba01 = a_b_shm[readBs + 2 * 64 + 32 + 1];
        j0Ba02 = a_b_shm[readBs + 2 * 64 + 32 + 2];
        j0Ba03 = a_b_shm[readBs + 2 * 64 + 32 + 3];

        cbb20 += j1Bb02 * j1Ab00;
        cbb21 += j1Bb02 * j1Ab01;
        cbb22 += j1Bb02 * j1Ab02;
        cbb23 += j1Bb02 * j1Ab03;
        cbb30 += j1Bb03 * j1Ab00;
        cbb31 += j1Bb03 * j1Ab01;
        cbb32 += j1Bb03 * j1Ab02;
        cbb33 += j1Bb03 * j1Ab03;
        cba00 += j1Ba00 * j1Ab00;
        cba01 += j1Ba00 * j1Ab01;
        cba02 += j1Ba00 * j1Ab02;
        cba03 += j1Ba00 * j1Ab03;
        cba10 += j1Ba01 * j1Ab00;
        cba11 += j1Ba01 * j1Ab01;
        cba12 += j1Ba01 * j1Ab02;
        cba13 += j1Ba01 * j1Ab03;
        cba20 += j1Ba02 * j1Ab00;
        cba21 += j1Ba02 * j1Ab01;
        cba22 += j1Ba02 * j1Ab02;
        cba23 += j1Ba02 * j1Ab03;
        cba30 += j1Ba03 * j1Ab00;
        cba31 += j1Ba03 * j1Ab01;
        cba32 += j1Ba03 * j1Ab02;
        cba33 += j1Ba03 * j1Ab03;

        loadX40 = read_addr[track4 + 0];
        loadX41 = read_addr[track4 + 1];
        loadX42 = read_addr[track4 + 2];
        loadX43 = read_addr[track4 + 3];
        cab00 += j1Bb00 * j1Aa00;
        cab01 += j1Bb00 * j1Aa01;
        loadX60 = read_addr[track6 + 0];
        loadX61 = read_addr[track6 + 1];
        loadX62 = read_addr[track6 + 2];
        loadX63 = read_addr[track6 + 3];
        cab02 += j1Bb00 * j1Aa02;
        cab03 += j1Bb00 * j1Aa03;
        cab10 += j1Bb01 * j1Aa00;
        cab11 += j1Bb01 * j1Aa01;
        cab12 += j1Bb01 * j1Aa02;
        cab13 += j1Bb01 * j1Aa03;
        cab20 += j1Bb02 * j1Aa00;
        cab21 += j1Bb02 * j1Aa01;
        cab22 += j1Bb02 * j1Aa02;
        cab23 += j1Bb02 * j1Aa03;
        cab30 += j1Bb03 * j1Aa00;
        cab31 += j1Bb03 * j1Aa01;
        cab32 += j1Bb03 * j1Aa02;
        cab33 += j1Bb03 * j1Aa03;
        caa00 += j1Ba00 * j1Aa00;
        caa01 += j1Ba00 * j1Aa01;
        caa02 += j1Ba00 * j1Aa02;
        caa03 += j1Ba00 * j1Aa03;
        caa10 += j1Ba01 * j1Aa00;
        caa11 += j1Ba01 * j1Aa01;
        caa12 += j1Ba01 * j1Aa02;
        caa13 += j1Ba01 * j1Aa03;
        caa20 += j1Ba02 * j1Aa00;
        caa21 += j1Ba02 * j1Aa01;
        caa22 += j1Ba02 * j1Aa02;
        caa23 += j1Ba02 * j1Aa03;
        caa30 += j1Ba03 * j1Aa00;
        caa31 += j1Ba03 * j1Aa01;
        caa32 += j1Ba03 * j1Aa02;
        caa33 += j1Ba03 * j1Aa03;


        cbb00 += j0Bb00 * j0Ab00;
        cbb01 += j0Bb00 * j0Ab01;
        j1Ab00 = a_b_shm[readAs + 3 * 64 + 0];
        j1Ab01 = a_b_shm[readAs + 3 * 64 + 1];
        j1Ab02 = a_b_shm[readAs + 3 * 64 + 2];
        j1Ab03 = a_b_shm[readAs + 3 * 64 + 3];
        cbb02 += j0Bb00 * j0Ab02;
        cbb03 += j0Bb00 * j0Ab03;
        j1Bb00 = a_b_shm[readBs + 3 * 64 + 0];
        j1Bb01 = a_b_shm[readBs + 3 * 64 + 1];
        j1Bb02 = a_b_shm[readBs + 3 * 64 + 2];
        j1Bb03 = a_b_shm[readBs + 3 * 64 + 3];
        cbb10 += j0Bb01 * j0Ab00;
        cbb11 += j0Bb01 * j0Ab01;
        j1Aa00 = a_b_shm[readAs + 3 * 64 + 32 + 0];
        j1Aa01 = a_b_shm[readAs + 3 * 64 + 32 + 1];
        j1Aa02 = a_b_shm[readAs + 3 * 64 + 32 + 2];
        j1Aa03 = a_b_shm[readAs + 3 * 64 + 32 + 3];
        cbb12 += j0Bb01 * j0Ab02;
        cbb13 += j0Bb01 * j0Ab03;
        j1Ba00 = a_b_shm[readBs + 3 * 64 + 32 + 0];
        j1Ba01 = a_b_shm[readBs + 3 * 64 + 32 + 1];
        j1Ba02 = a_b_shm[readBs + 3 * 64 + 32 + 2];
        j1Ba03 = a_b_shm[readBs + 3 * 64 + 32 + 3];

        cbb20 += j0Bb02 * j0Ab00;
        cbb21 += j0Bb02 * j0Ab01;
        cbb22 += j0Bb02 * j0Ab02;
        cbb23 += j0Bb02 * j0Ab03;
        cbb30 += j0Bb03 * j0Ab00;
        cbb31 += j0Bb03 * j0Ab01;
        cbb32 += j0Bb03 * j0Ab02;
        cbb33 += j0Bb03 * j0Ab03;
        cba00 += j0Ba00 * j0Ab00;
        cba01 += j0Ba00 * j0Ab01;
        cba02 += j0Ba00 * j0Ab02;
        cba03 += j0Ba00 * j0Ab03;
        cba10 += j0Ba01 * j0Ab00;
        cba11 += j0Ba01 * j0Ab01;
        cba12 += j0Ba01 * j0Ab02;
        cba13 += j0Ba01 * j0Ab03;
        cba20 += j0Ba02 * j0Ab00;
        cba21 += j0Ba02 * j0Ab01;
        cba22 += j0Ba02 * j0Ab02;
        cba23 += j0Ba02 * j0Ab03;
        cba30 += j0Ba03 * j0Ab00;
        cba31 += j0Ba03 * j0Ab01;
        cba32 += j0Ba03 * j0Ab02;
        cba33 += j0Ba03 * j0Ab03;
        cab00 += j0Bb00 * j0Aa00;
        cab01 += j0Bb00 * j0Aa01;
        cab02 += j0Bb00 * j0Aa02;
        cab03 += j0Bb00 * j0Aa03;
        cab10 += j0Bb01 * j0Aa00;
        cab11 += j0Bb01 * j0Aa01;
        cab12 += j0Bb01 * j0Aa02;
        cab13 += j0Bb01 * j0Aa03;
        cab20 += j0Bb02 * j0Aa00;
        cab21 += j0Bb02 * j0Aa01;
        cab22 += j0Bb02 * j0Aa02;
        cab23 += j0Bb02 * j0Aa03;
        cab30 += j0Bb03 * j0Aa00;
        cab31 += j0Bb03 * j0Aa01;
        cab32 += j0Bb03 * j0Aa02;
        cab33 += j0Bb03 * j0Aa03;
        caa00 += j0Ba00 * j0Aa00;
        caa01 += j0Ba00 * j0Aa01;
        caa02 += j0Ba00 * j0Aa02;
        caa03 += j0Ba00 * j0Aa03;
        caa10 += j0Ba01 * j0Aa00;
        caa11 += j0Ba01 * j0Aa01;
        caa12 += j0Ba01 * j0Aa02;
        caa13 += j0Ba01 * j0Aa03;
        caa20 += j0Ba02 * j0Aa00;
        caa21 += j0Ba02 * j0Aa01;
        caa22 += j0Ba02 * j0Aa02;
        caa23 += j0Ba02 * j0Aa03;
        caa30 += j0Ba03 * j0Aa00;
        caa31 += j0Ba03 * j0Aa01;
        caa32 += j0Ba03 * j0Aa02;
        caa33 += j0Ba03 * j0Aa03;


        cbb00 += j1Bb00 * j1Ab00;
        cbb01 += j1Bb00 * j1Ab01;
        j0Ab00 = a_b_shm[readAs + 4 * 64 + 0];
        j0Ab01 = a_b_shm[readAs + 4 * 64 + 1];
        j0Ab02 = a_b_shm[readAs + 4 * 64 + 2];
        j0Ab03 = a_b_shm[readAs + 4 * 64 + 3];
        cbb02 += j1Bb00 * j1Ab02;
        cbb03 += j1Bb00 * j1Ab03;
        j0Bb00 = a_b_shm[readBs + 4 * 64 + 0];
        j0Bb01 = a_b_shm[readBs + 4 * 64 + 1];
        j0Bb02 = a_b_shm[readBs + 4 * 64 + 2];
        j0Bb03 = a_b_shm[readBs + 4 * 64 + 3];
        cbb10 += j1Bb01 * j1Ab00;
        cbb11 += j1Bb01 * j1Ab01;
        j0Aa00 = a_b_shm[readAs + 4 * 64 + 32 + 0];
        j0Aa01 = a_b_shm[readAs + 4 * 64 + 32 + 1];
        j0Aa02 = a_b_shm[readAs + 4 * 64 + 32 + 2];
        j0Aa03 = a_b_shm[readAs + 4 * 64 + 32 + 3];
        cbb12 += j1Bb01 * j1Ab02;
        cbb13 += j1Bb01 * j1Ab03;
        j0Ba00 = a_b_shm[readBs + 4 * 64 + 32 + 0];
        j0Ba01 = a_b_shm[readBs + 4 * 64 + 32 + 1];
        j0Ba02 = a_b_shm[readBs + 4 * 64 + 32 + 2];
        j0Ba03 = a_b_shm[readBs + 4 * 64 + 32 + 3];

        cbb20 += j1Bb02 * j1Ab00;
        cbb21 += j1Bb02 * j1Ab01;
        cbb22 += j1Bb02 * j1Ab02;
        cbb23 += j1Bb02 * j1Ab03;
        cbb30 += j1Bb03 * j1Ab00;
        cbb31 += j1Bb03 * j1Ab01;
        cbb32 += j1Bb03 * j1Ab02;
        cbb33 += j1Bb03 * j1Ab03;
        cba00 += j1Ba00 * j1Ab00;
        cba01 += j1Ba00 * j1Ab01;
        cba02 += j1Ba00 * j1Ab02;
        cba03 += j1Ba00 * j1Ab03;
        cba10 += j1Ba01 * j1Ab00;
        cba11 += j1Ba01 * j1Ab01;
        cba12 += j1Ba01 * j1Ab02;
        cba13 += j1Ba01 * j1Ab03;
        cba20 += j1Ba02 * j1Ab00;
        cba21 += j1Ba02 * j1Ab01;
        cba22 += j1Ba02 * j1Ab02;
        cba23 += j1Ba02 * j1Ab03;
        cba30 += j1Ba03 * j1Ab00;
        cba31 += j1Ba03 * j1Ab01;
        cba32 += j1Ba03 * j1Ab02;
        cba33 += j1Ba03 * j1Ab03;
        cab00 += j1Bb00 * j1Aa00;
        cab01 += j1Bb00 * j1Aa01;
        cab02 += j1Bb00 * j1Aa02;
        cab03 += j1Bb00 * j1Aa03;
        cab10 += j1Bb01 * j1Aa00;
        cab11 += j1Bb01 * j1Aa01;
        cab12 += j1Bb01 * j1Aa02;
        cab13 += j1Bb01 * j1Aa03;
        cab20 += j1Bb02 * j1Aa00;
        cab21 += j1Bb02 * j1Aa01;
        cab22 += j1Bb02 * j1Aa02;
        cab23 += j1Bb02 * j1Aa03;
        cab30 += j1Bb03 * j1Aa00;
        cab31 += j1Bb03 * j1Aa01;
        cab32 += j1Bb03 * j1Aa02;
        cab33 += j1Bb03 * j1Aa03;
        caa00 += j1Ba00 * j1Aa00;
        caa01 += j1Ba00 * j1Aa01;
        caa02 += j1Ba00 * j1Aa02;
        caa03 += j1Ba00 * j1Aa03;
        caa10 += j1Ba01 * j1Aa00;
        caa11 += j1Ba01 * j1Aa01;
        caa12 += j1Ba01 * j1Aa02;
        caa13 += j1Ba01 * j1Aa03;
        caa20 += j1Ba02 * j1Aa00;
        caa21 += j1Ba02 * j1Aa01;
        caa22 += j1Ba02 * j1Aa02;
        caa23 += j1Ba02 * j1Aa03;
        caa30 += j1Ba03 * j1Aa00;
        caa31 += j1Ba03 * j1Aa01;
        caa32 += j1Ba03 * j1Aa02;
        caa33 += j1Ba03 * j1Aa03;


        cbb00 += j0Bb00 * j0Ab00;
        cbb01 += j0Bb00 * j0Ab01;
        j1Ab00 = a_b_shm[readAs + 5 * 64 + 0];
        j1Ab01 = a_b_shm[readAs + 5 * 64 + 1];
        j1Ab02 = a_b_shm[readAs + 5 * 64 + 2];
        j1Ab03 = a_b_shm[readAs + 5 * 64 + 3];
        cbb02 += j0Bb00 * j0Ab02;
        cbb03 += j0Bb00 * j0Ab03;
        j1Bb00 = a_b_shm[readBs + 5 * 64 + 0];
        j1Bb01 = a_b_shm[readBs + 5 * 64 + 1];
        j1Bb02 = a_b_shm[readBs + 5 * 64 + 2];
        j1Bb03 = a_b_shm[readBs + 5 * 64 + 3];
        cbb10 += j0Bb01 * j0Ab00;
        cbb11 += j0Bb01 * j0Ab01;
        j1Aa00 = a_b_shm[readAs + 5 * 64 + 32 + 0];
        j1Aa01 = a_b_shm[readAs + 5 * 64 + 32 + 1];
        j1Aa02 = a_b_shm[readAs + 5 * 64 + 32 + 2];
        j1Aa03 = a_b_shm[readAs + 5 * 64 + 32 + 3];
        cbb12 += j0Bb01 * j0Ab02;
        cbb13 += j0Bb01 * j0Ab03;
        j1Ba00 = a_b_shm[readBs + 5 * 64 + 32 + 0];
        j1Ba01 = a_b_shm[readBs + 5 * 64 + 32 + 1];
        j1Ba02 = a_b_shm[readBs + 5 * 64 + 32 + 2];
        j1Ba03 = a_b_shm[readBs + 5 * 64 + 32 + 3];

        cbb20 += j0Bb02 * j0Ab00;
        cbb21 += j0Bb02 * j0Ab01;
        cbb22 += j0Bb02 * j0Ab02;
        cbb23 += j0Bb02 * j0Ab03;
        cbb30 += j0Bb03 * j0Ab00;
        cbb31 += j0Bb03 * j0Ab01;
        cbb32 += j0Bb03 * j0Ab02;
        cbb33 += j0Bb03 * j0Ab03;
        cba00 += j0Ba00 * j0Ab00;
        cba01 += j0Ba00 * j0Ab01;
        cba02 += j0Ba00 * j0Ab02;
        cba03 += j0Ba00 * j0Ab03;
        cba10 += j0Ba01 * j0Ab00;
        cba11 += j0Ba01 * j0Ab01;
        cba12 += j0Ba01 * j0Ab02;
        cba13 += j0Ba01 * j0Ab03;
        cba20 += j0Ba02 * j0Ab00;
        cba21 += j0Ba02 * j0Ab01;
        cba22 += j0Ba02 * j0Ab02;
        cba23 += j0Ba02 * j0Ab03;
        cba30 += j0Ba03 * j0Ab00;
        cba31 += j0Ba03 * j0Ab01;
        cba32 += j0Ba03 * j0Ab02;
        cba33 += j0Ba03 * j0Ab03;
        cab00 += j0Bb00 * j0Aa00;
        cab01 += j0Bb00 * j0Aa01;
        cab02 += j0Bb00 * j0Aa02;
        cab03 += j0Bb00 * j0Aa03;
        cab10 += j0Bb01 * j0Aa00;
        cab11 += j0Bb01 * j0Aa01;
        cab12 += j0Bb01 * j0Aa02;
        cab13 += j0Bb01 * j0Aa03;
        cab20 += j0Bb02 * j0Aa00;
        cab21 += j0Bb02 * j0Aa01;
        cab22 += j0Bb02 * j0Aa02;
        cab23 += j0Bb02 * j0Aa03;
        cab30 += j0Bb03 * j0Aa00;
        cab31 += j0Bb03 * j0Aa01;
        cab32 += j0Bb03 * j0Aa02;
        cab33 += j0Bb03 * j0Aa03;
        caa00 += j0Ba00 * j0Aa00;
        caa01 += j0Ba00 * j0Aa01;
        caa02 += j0Ba00 * j0Aa02;
        caa03 += j0Ba00 * j0Aa03;
        caa10 += j0Ba01 * j0Aa00;
        caa11 += j0Ba01 * j0Aa01;
        caa12 += j0Ba01 * j0Aa02;
        caa13 += j0Ba01 * j0Aa03;
        caa20 += j0Ba02 * j0Aa00;
        caa21 += j0Ba02 * j0Aa01;
        caa22 += j0Ba02 * j0Aa02;
        caa23 += j0Ba02 * j0Aa03;
        caa30 += j0Ba03 * j0Aa00;
        caa31 += j0Ba03 * j0Aa01;
        caa32 += j0Ba03 * j0Aa02;
        caa33 += j0Ba03 * j0Aa03;


        cbb00 += j1Bb00 * j1Ab00;
        cbb01 += j1Bb00 * j1Ab01;
        j0Ab00 = a_b_shm[readAs + 6 * 64 + 0];
        j0Ab01 = a_b_shm[readAs + 6 * 64 + 1];
        j0Ab02 = a_b_shm[readAs + 6 * 64 + 2];
        j0Ab03 = a_b_shm[readAs + 6 * 64 + 3];
        cbb02 += j1Bb00 * j1Ab02;
        cbb03 += j1Bb00 * j1Ab03;
        j0Bb00 = a_b_shm[readBs + 6 * 64 + 0];
        j0Bb01 = a_b_shm[readBs + 6 * 64 + 1];
        j0Bb02 = a_b_shm[readBs + 6 * 64 + 2];
        j0Bb03 = a_b_shm[readBs + 6 * 64 + 3];
        cbb10 += j1Bb01 * j1Ab00;
        cbb11 += j1Bb01 * j1Ab01;
        j0Aa00 = a_b_shm[readAs + 6 * 64 + 32 + 0];
        j0Aa01 = a_b_shm[readAs + 6 * 64 + 32 + 1];
        j0Aa02 = a_b_shm[readAs + 6 * 64 + 32 + 2];
        j0Aa03 = a_b_shm[readAs + 6 * 64 + 32 + 3];
        cbb12 += j1Bb01 * j1Ab02;
        cbb13 += j1Bb01 * j1Ab03;
        j0Ba00 = a_b_shm[readBs + 6 * 64 + 32 + 0];
        j0Ba01 = a_b_shm[readBs + 6 * 64 + 32 + 1];
        j0Ba02 = a_b_shm[readBs + 6 * 64 + 32 + 2];
        j0Ba03 = a_b_shm[readBs + 6 * 64 + 32 + 3];

        cbb20 += j1Bb02 * j1Ab00;
        cbb21 += j1Bb02 * j1Ab01;
        cbb22 += j1Bb02 * j1Ab02;
        cbb23 += j1Bb02 * j1Ab03;
        cbb30 += j1Bb03 * j1Ab00;
        cbb31 += j1Bb03 * j1Ab01;
        cbb32 += j1Bb03 * j1Ab02;
        cbb33 += j1Bb03 * j1Ab03;
        cba00 += j1Ba00 * j1Ab00;
        cba01 += j1Ba00 * j1Ab01;
        cba02 += j1Ba00 * j1Ab02;
        cba03 += j1Ba00 * j1Ab03;
        cba10 += j1Ba01 * j1Ab00;
        cba11 += j1Ba01 * j1Ab01;
        cba12 += j1Ba01 * j1Ab02;
        cba13 += j1Ba01 * j1Ab03;
        cba20 += j1Ba02 * j1Ab00;
        cba21 += j1Ba02 * j1Ab01;
        cba22 += j1Ba02 * j1Ab02;
        cba23 += j1Ba02 * j1Ab03;
        cba30 += j1Ba03 * j1Ab00;
        cba31 += j1Ba03 * j1Ab01;
        cba32 += j1Ba03 * j1Ab02;
        cba33 += j1Ba03 * j1Ab03;

        a_b_shm[write_offset + 0 * 64 + 0] = loadX00;
        a_b_shm[write_offset + 0 * 64 + 1] = loadX01;
        a_b_shm[write_offset + 0 * 64 + 2] = loadX02;
        a_b_shm[write_offset + 0 * 64 + 3] = loadX03;
        cab00 += j1Bb00 * j1Aa00;
        cab01 += j1Bb00 * j1Aa01;
        a_b_shm[write_offset + 2 * 64 + 0] = loadX20;
        a_b_shm[write_offset + 2 * 64 + 1] = loadX21;
        a_b_shm[write_offset + 2 * 64 + 2] = loadX22;
        a_b_shm[write_offset + 2 * 64 + 3] = loadX23;
        cab02 += j1Bb00 * j1Aa02;
        cab03 += j1Bb00 * j1Aa03;
        cab10 += j1Bb01 * j1Aa00;
        cab11 += j1Bb01 * j1Aa01;
        cab12 += j1Bb01 * j1Aa02;
        cab13 += j1Bb01 * j1Aa03;
        cab20 += j1Bb02 * j1Aa00;
        cab21 += j1Bb02 * j1Aa01;
        cab22 += j1Bb02 * j1Aa02;
        cab23 += j1Bb02 * j1Aa03;
        cab30 += j1Bb03 * j1Aa00;
        cab31 += j1Bb03 * j1Aa01;
        cab32 += j1Bb03 * j1Aa02;
        cab33 += j1Bb03 * j1Aa03;
        caa00 += j1Ba00 * j1Aa00;
        caa01 += j1Ba00 * j1Aa01;
        caa02 += j1Ba00 * j1Aa02;
        caa03 += j1Ba00 * j1Aa03;
        caa10 += j1Ba01 * j1Aa00;
        caa11 += j1Ba01 * j1Aa01;
        caa12 += j1Ba01 * j1Aa02;
        caa13 += j1Ba01 * j1Aa03;
        caa20 += j1Ba02 * j1Aa00;
        caa21 += j1Ba02 * j1Aa01;
        caa22 += j1Ba02 * j1Aa02;
        caa23 += j1Ba02 * j1Aa03;
        caa30 += j1Ba03 * j1Aa00;
        caa31 += j1Ba03 * j1Aa01;
        caa32 += j1Ba03 * j1Aa02;
        caa33 += j1Ba03 * j1Aa03;


        cbb00 += j0Bb00 * j0Ab00;
        cbb01 += j0Bb00 * j0Ab01;
        j1Ab00 = a_b_shm[readAs + 7 * 64 + 0];
        j1Ab01 = a_b_shm[readAs + 7 * 64 + 1];
        j1Ab02 = a_b_shm[readAs + 7 * 64 + 2];
        j1Ab03 = a_b_shm[readAs + 7 * 64 + 3];
        cbb02 += j0Bb00 * j0Ab02;
        cbb03 += j0Bb00 * j0Ab03;
        j1Bb00 = a_b_shm[readBs + 7 * 64 + 0];
        j1Bb01 = a_b_shm[readBs + 7 * 64 + 1];
        j1Bb02 = a_b_shm[readBs + 7 * 64 + 2];
        j1Bb03 = a_b_shm[readBs + 7 * 64 + 3];
        cbb10 += j0Bb01 * j0Ab00;
        cbb11 += j0Bb01 * j0Ab01;
        j1Aa00 = a_b_shm[readAs + 7 * 64 + 32 + 0];
        j1Aa01 = a_b_shm[readAs + 7 * 64 + 32 + 1];
        j1Aa02 = a_b_shm[readAs + 7 * 64 + 32 + 2];
        j1Aa03 = a_b_shm[readAs + 7 * 64 + 32 + 3];
        cbb12 += j0Bb01 * j0Ab02;
        cbb13 += j0Bb01 * j0Ab03;
        j1Ba00 = a_b_shm[readBs + 7 * 64 + 32 + 0];
        j1Ba01 = a_b_shm[readBs + 7 * 64 + 32 + 1];
        j1Ba02 = a_b_shm[readBs + 7 * 64 + 32 + 2];
        j1Ba03 = a_b_shm[readBs + 7 * 64 + 32 + 3];

        cbb20 += j0Bb02 * j0Ab00;
        cbb21 += j0Bb02 * j0Ab01;
        cbb22 += j0Bb02 * j0Ab02;
        cbb23 += j0Bb02 * j0Ab03;
        cbb30 += j0Bb03 * j0Ab00;
        cbb31 += j0Bb03 * j0Ab01;
        cbb32 += j0Bb03 * j0Ab02;
        cbb33 += j0Bb03 * j0Ab03;
        cba00 += j0Ba00 * j0Ab00;
        cba01 += j0Ba00 * j0Ab01;
        cba02 += j0Ba00 * j0Ab02;
        cba03 += j0Ba00 * j0Ab03;
        cba10 += j0Ba01 * j0Ab00;
        cba11 += j0Ba01 * j0Ab01;
        cba12 += j0Ba01 * j0Ab02;
        cba13 += j0Ba01 * j0Ab03;
        cba20 += j0Ba02 * j0Ab00;
        cba21 += j0Ba02 * j0Ab01;
        cba22 += j0Ba02 * j0Ab02;
        cba23 += j0Ba02 * j0Ab03;
        cba30 += j0Ba03 * j0Ab00;
        cba31 += j0Ba03 * j0Ab01;
        cba32 += j0Ba03 * j0Ab02;
        cba33 += j0Ba03 * j0Ab03;

        a_b_shm[write_offset + 4 * 64 + 0] = loadX40;
        a_b_shm[write_offset + 4 * 64 + 1] = loadX41;
        a_b_shm[write_offset + 4 * 64 + 2] = loadX42;
        a_b_shm[write_offset + 4 * 64 + 3] = loadX43;
        cab00 += j0Bb00 * j0Aa00;
        cab01 += j0Bb00 * j0Aa01;
        a_b_shm[write_offset + 6 * 64 + 0] = loadX60;
        a_b_shm[write_offset + 6 * 64 + 1] = loadX61;
        a_b_shm[write_offset + 6 * 64 + 2] = loadX62;
        a_b_shm[write_offset + 6 * 64 + 3] = loadX63;
        cab02 += j0Bb00 * j0Aa02;
        cab03 += j0Bb00 * j0Aa03;
        cab10 += j0Bb01 * j0Aa00;
        cab11 += j0Bb01 * j0Aa01;
        cab12 += j0Bb01 * j0Aa02;
        cab13 += j0Bb01 * j0Aa03;
        cab20 += j0Bb02 * j0Aa00;
        cab21 += j0Bb02 * j0Aa01;
        cab22 += j0Bb02 * j0Aa02;
        cab23 += j0Bb02 * j0Aa03;
        cab30 += j0Bb03 * j0Aa00;
        cab31 += j0Bb03 * j0Aa01;
        cab32 += j0Bb03 * j0Aa02;
        cab33 += j0Bb03 * j0Aa03;
        caa00 += j0Ba00 * j0Aa00;
        caa01 += j0Ba00 * j0Aa01;
        caa02 += j0Ba00 * j0Aa02;
        caa03 += j0Ba00 * j0Aa03;
        caa10 += j0Ba01 * j0Aa00;
        caa11 += j0Ba01 * j0Aa01;
        caa12 += j0Ba01 * j0Aa02;
        caa13 += j0Ba01 * j0Aa03;
        caa20 += j0Ba02 * j0Aa00;
        caa21 += j0Ba02 * j0Aa01;
        caa22 += j0Ba02 * j0Aa02;
        caa23 += j0Ba02 * j0Aa03;
        caa30 += j0Ba03 * j0Aa00;
        caa31 += j0Ba03 * j0Aa01;
        caa32 += j0Ba03 * j0Aa02;
        caa33 += j0Ba03 * j0Aa03;

        __syncthreads();
        readAs ^= 16 * 64;
        readBs ^= 16 * 64;
        write_offset ^= 16 * 64;

        cbb00 += j1Bb00 * j1Ab00;
        cbb01 += j1Bb00 * j1Ab01;
        j0Ab00 = a_b_shm[readAs + 0 * 64 + 0];
        j0Ab01 = a_b_shm[readAs + 0 * 64 + 1];
        j0Ab02 = a_b_shm[readAs + 0 * 64 + 2];
        j0Ab03 = a_b_shm[readAs + 0 * 64 + 3];
        cbb02 += j1Bb00 * j1Ab02;
        cbb03 += j1Bb00 * j1Ab03;
        j0Bb00 = a_b_shm[readBs + 0 * 64 + 0];
        j0Bb01 = a_b_shm[readBs + 0 * 64 + 1];
        j0Bb02 = a_b_shm[readBs + 0 * 64 + 2];
        j0Bb03 = a_b_shm[readBs + 0 * 64 + 3];
        cbb10 += j1Bb01 * j1Ab00;
        cbb11 += j1Bb01 * j1Ab01;
        j0Aa00 = a_b_shm[readAs + 0 * 64 + 32 + 0];
        j0Aa01 = a_b_shm[readAs + 0 * 64 + 32 + 1];
        j0Aa02 = a_b_shm[readAs + 0 * 64 + 32 + 2];
        j0Aa03 = a_b_shm[readAs + 0 * 64 + 32 + 3];
        cbb12 += j1Bb01 * j1Ab02;
        cbb13 += j1Bb01 * j1Ab03;
        j0Ba00 = a_b_shm[readBs + 0 * 64 + 32 + 0];
        j0Ba01 = a_b_shm[readBs + 0 * 64 + 32 + 1];
        j0Ba02 = a_b_shm[readBs + 0 * 64 + 32 + 2];
        j0Ba03 = a_b_shm[readBs + 0 * 64 + 32 + 3];

        cbb20 += j1Bb02 * j1Ab00;
        cbb21 += j1Bb02 * j1Ab01;
        cbb22 += j1Bb02 * j1Ab02;
        cbb23 += j1Bb02 * j1Ab03;
        cbb30 += j1Bb03 * j1Ab00;
        cbb31 += j1Bb03 * j1Ab01;
        cbb32 += j1Bb03 * j1Ab02;
        cbb33 += j1Bb03 * j1Ab03;
        cba00 += j1Ba00 * j1Ab00;
        cba01 += j1Ba00 * j1Ab01;
        cba02 += j1Ba00 * j1Ab02;
        cba03 += j1Ba00 * j1Ab03;
        cba10 += j1Ba01 * j1Ab00;
        cba11 += j1Ba01 * j1Ab01;
        cba12 += j1Ba01 * j1Ab02;
        cba13 += j1Ba01 * j1Ab03;
        cba20 += j1Ba02 * j1Ab00;
        cba21 += j1Ba02 * j1Ab01;
        cba22 += j1Ba02 * j1Ab02;
        cba23 += j1Ba02 * j1Ab03;
        cba30 += j1Ba03 * j1Ab00;
        cba31 += j1Ba03 * j1Ab01;
        cba32 += j1Ba03 * j1Ab02;
        cba33 += j1Ba03 * j1Ab03;
        cab00 += j1Bb00 * j1Aa00;
        cab01 += j1Bb00 * j1Aa01;
        cab02 += j1Bb00 * j1Aa02;
        cab03 += j1Bb00 * j1Aa03;
        cab10 += j1Bb01 * j1Aa00;
        cab11 += j1Bb01 * j1Aa01;
        cab12 += j1Bb01 * j1Aa02;
        cab13 += j1Bb01 * j1Aa03;
        cab20 += j1Bb02 * j1Aa00;
        cab21 += j1Bb02 * j1Aa01;
        cab22 += j1Bb02 * j1Aa02;
        cab23 += j1Bb02 * j1Aa03;
        cab30 += j1Bb03 * j1Aa00;
        cab31 += j1Bb03 * j1Aa01;
        cab32 += j1Bb03 * j1Aa02;
        cab33 += j1Bb03 * j1Aa03;
        caa00 += j1Ba00 * j1Aa00;
        caa01 += j1Ba00 * j1Aa01;
        caa02 += j1Ba00 * j1Aa02;
        caa03 += j1Ba00 * j1Aa03;
        caa10 += j1Ba01 * j1Aa00;
        caa11 += j1Ba01 * j1Aa01;
        caa12 += j1Ba01 * j1Aa02;
        caa13 += j1Ba01 * j1Aa03;
        caa20 += j1Ba02 * j1Aa00;
        caa21 += j1Ba02 * j1Aa01;
        caa22 += j1Ba02 * j1Aa02;
        caa23 += j1Ba02 * j1Aa03;
        caa30 += j1Ba03 * j1Aa00;
        caa31 += j1Ba03 * j1Aa01;
        caa32 += j1Ba03 * j1Aa02;
        caa33 += j1Ba03 * j1Aa03;

        track0 += 8 * ldx;
        track2 += 8 * ldx;
        track4 += 8 * ldx;
        track6 += 8 * ldx;
    }
    __syncthreads();

    // j0Ab00 = a_b_shm[readAs + 0];
    // j0Ab01 = a_b_shm[readAs + 1];
    // j0Ab02 = a_b_shm[readAs + 2];
    // j0Ab03 = a_b_shm[readAs + 3];

    // j0Bb00 = a_b_shm[readBs + 0];
    // j0Bb01 = a_b_shm[readBs + 1];
    // j0Bb02 = a_b_shm[readBs + 2];
    // j0Bb03 = a_b_shm[readBs + 3];

    // j0Aa00 = a_b_shm[readAs + 32 + 0];
    // j0Aa01 = a_b_shm[readAs + 32 + 1];
    // j0Aa02 = a_b_shm[readAs + 32 + 2];
    // j0Aa03 = a_b_shm[readAs + 32 + 3];

    // j0Ba00 = a_b_shm[readBs + 32 + 0];
    // j0Ba01 = a_b_shm[readBs + 32 + 1];
    // j0Ba02 = a_b_shm[readBs + 32 + 2];
    // j0Ba03 = a_b_shm[readBs + 32 + 3];

    // for (int j = 0; j < 7; ++j) {
    //     int prefetch = (j + 1) % 8;

    //     if (!(j & 1)) {

    //         cbb00 += j0Bb00 * j0Ab00;
    //         cbb01 += j0Bb00 * j0Ab01;
    //         j1Ab00 = a_b_shm[readAs + prefetch * 64 + 0];
    //         j1Ab01 = a_b_shm[readAs + prefetch * 64 + 1];
    //         j1Ab02 = a_b_shm[readAs + prefetch * 64 + 2];
    //         j1Ab03 = a_b_shm[readAs + prefetch * 64 + 3];
    //         cbb02 += j0Bb00 * j0Ab02;
    //         cbb03 += j0Bb00 * j0Ab03;
    //         j1Bb00 = a_b_shm[readBs + prefetch * 64 + 0];
    //         j1Bb01 = a_b_shm[readBs + prefetch * 64 + 1];
    //         j1Bb02 = a_b_shm[readBs + prefetch * 64 + 2];
    //         j1Bb03 = a_b_shm[readBs + prefetch * 64 + 3];

    //         cbb10 += j0Bb01 * j0Ab00;
    //         cbb11 += j0Bb01 * j0Ab01;
    //         j1Aa00 = a_b_shm[readAs + prefetch * 64 + 32 + 0];
    //         j1Aa01 = a_b_shm[readAs + prefetch * 64 + 32 + 1];
    //         j1Aa02 = a_b_shm[readAs + prefetch * 64 + 32 + 2];
    //         j1Aa03 = a_b_shm[readAs + prefetch * 64 + 32 + 3];
    //         cbb12 += j0Bb01 * j0Ab02;
    //         cbb13 += j0Bb01 * j0Ab03;
    //         j1Ba00 = a_b_shm[readBs + prefetch * 64 + 32 + 0];
    //         j1Ba01 = a_b_shm[readBs + prefetch * 64 + 32 + 1];
    //         j1Ba02 = a_b_shm[readBs + prefetch * 64 + 32 + 2];
    //         j1Ba03 = a_b_shm[readBs + prefetch * 64 + 32 + 3];

    //         cbb20 += j0Bb02 * j0Ab00;
    //         cbb21 += j0Bb02 * j0Ab01;
    //         cbb22 += j0Bb02 * j0Ab02;
    //         cbb23 += j0Bb02 * j0Ab03;

    //         cbb30 += j0Bb03 * j0Ab00;
    //         cbb31 += j0Bb03 * j0Ab01;
    //         cbb32 += j0Bb03 * j0Ab02;
    //         cbb33 += j0Bb03 * j0Ab03;

    //         cba00 += j0Ba00 * j0Ab00;
    //         cba01 += j0Ba00 * j0Ab01;
    //         cba02 += j0Ba00 * j0Ab02;
    //         cba03 += j0Ba00 * j0Ab03;

    //         cba10 += j0Ba01 * j0Ab00;
    //         cba11 += j0Ba01 * j0Ab01;
    //         cba12 += j0Ba01 * j0Ab02;
    //         cba13 += j0Ba01 * j0Ab03;

    //         cba20 += j0Ba02 * j0Ab00;
    //         cba21 += j0Ba02 * j0Ab01;
    //         cba22 += j0Ba02 * j0Ab02;
    //         cba23 += j0Ba02 * j0Ab03;

    //         cba30 += j0Ba03 * j0Ab00;
    //         cba31 += j0Ba03 * j0Ab01;
    //         cba32 += j0Ba03 * j0Ab02;
    //         cba33 += j0Ba03 * j0Ab03;

    //         cab00 += j0Bb00 * j0Aa00;
    //         cab01 += j0Bb00 * j0Aa01;
    //         cab02 += j0Bb00 * j0Aa02;
    //         cab03 += j0Bb00 * j0Aa03;

    //         cab10 += j0Bb01 * j0Aa00;
    //         cab11 += j0Bb01 * j0Aa01;
    //         cab12 += j0Bb01 * j0Aa02;
    //         cab13 += j0Bb01 * j0Aa03;

    //         cab20 += j0Bb02 * j0Aa00;
    //         cab21 += j0Bb02 * j0Aa01;
    //         cab22 += j0Bb02 * j0Aa02;
    //         cab23 += j0Bb02 * j0Aa03;

    //         cab30 += j0Bb03 * j0Aa00;
    //         cab31 += j0Bb03 * j0Aa01;
    //         cab32 += j0Bb03 * j0Aa02;
    //         cab33 += j0Bb03 * j0Aa03;

    //         caa00 += j0Ba00 * j0Aa00;
    //         caa01 += j0Ba00 * j0Aa01;
    //         caa02 += j0Ba00 * j0Aa02;
    //         caa03 += j0Ba00 * j0Aa03;

    //         caa10 += j0Ba01 * j0Aa00;
    //         caa11 += j0Ba01 * j0Aa01;
    //         caa12 += j0Ba01 * j0Aa02;
    //         caa13 += j0Ba01 * j0Aa03;

    //         caa20 += j0Ba02 * j0Aa00;
    //         caa21 += j0Ba02 * j0Aa01;
    //         caa22 += j0Ba02 * j0Aa02;
    //         caa23 += j0Ba02 * j0Aa03;

    //         caa30 += j0Ba03 * j0Aa00;
    //         caa31 += j0Ba03 * j0Aa01;
    //         caa32 += j0Ba03 * j0Aa02;
    //         caa33 += j0Ba03 * j0Aa03;
    //     } else {
    //         // j1Ab00 = a_b_shm[readAs + prefetch * 64 + 0];
    //         // j1Ab01 = a_b_shm[readAs + prefetch * 64 + 1];
    //         // j1Ab02 = a_b_shm[readAs + prefetch * 64 + 2];
    //         // j1Ab03 = a_b_shm[readAs + prefetch * 64 + 3];

    //         // j1Bb00 = a_b_shm[readBs + prefetch * 64 + 0];
    //         // j1Bb01 = a_b_shm[readBs + prefetch * 64 + 1];
    //         // j1Bb02 = a_b_shm[readBs + prefetch * 64 + 2];
    //         // j1Bb03 = a_b_shm[readBs + prefetch * 64 + 3];

    //         // j1Aa00 = a_b_shm[readAs + prefetch * 64 + 32 + 0];
    //         // j1Aa01 = a_b_shm[readAs + prefetch * 64 + 32 + 1];
    //         // j1Aa02 = a_b_shm[readAs + prefetch * 64 + 32 + 2];
    //         // j1Aa03 = a_b_shm[readAs + prefetch * 64 + 32 + 3];

    //         // j1Ba00 = a_b_shm[readBs + prefetch * 64 + 32 + 0];
    //         // j1Ba01 = a_b_shm[readBs + prefetch * 64 + 32 + 1];
    //         // j1Ba02 = a_b_shm[readBs + prefetch * 64 + 32 + 2];
    //         // j1Ba03 = a_b_shm[readBs + prefetch * 64 + 32 + 3];

    //         cbb00 += j1Bb00 * j1Ab00;
    //         cbb01 += j1Bb00 * j1Ab01;
    //         j0Ab00 = a_b_shm[readAs + prefetch * 64 + 0];
    //         j0Ab01 = a_b_shm[readAs + prefetch * 64 + 1];
    //         j0Ab02 = a_b_shm[readAs + prefetch * 64 + 2];
    //         j0Ab03 = a_b_shm[readAs + prefetch * 64 + 3];
    //         cbb02 += j1Bb00 * j1Ab02;
    //         cbb03 += j1Bb00 * j1Ab03;
    //         j0Bb00 = a_b_shm[readBs + prefetch * 64 + 0];
    //         j0Bb01 = a_b_shm[readBs + prefetch * 64 + 1];
    //         j0Bb02 = a_b_shm[readBs + prefetch * 64 + 2];
    //         j0Bb03 = a_b_shm[readBs + prefetch * 64 + 3];

    //         cbb10 += j1Bb01 * j1Ab00;
    //         cbb11 += j1Bb01 * j1Ab01;
    //         j0Aa00 = a_b_shm[readAs + prefetch * 64 + 32 + 0];
    //         j0Aa01 = a_b_shm[readAs + prefetch * 64 + 32 + 1];
    //         j0Aa02 = a_b_shm[readAs + prefetch * 64 + 32 + 2];
    //         j0Aa03 = a_b_shm[readAs + prefetch * 64 + 32 + 3];
    //         cbb12 += j1Bb01 * j1Ab02;
    //         cbb13 += j1Bb01 * j1Ab03;
    //         j0Ba00 = a_b_shm[readBs + prefetch * 64 + 32 + 0];
    //         j0Ba01 = a_b_shm[readBs + prefetch * 64 + 32 + 1];
    //         j0Ba02 = a_b_shm[readBs + prefetch * 64 + 32 + 2];
    //         j0Ba03 = a_b_shm[readBs + prefetch * 64 + 32 + 3];

    //         cbb20 += j1Bb02 * j1Ab00;
    //         cbb21 += j1Bb02 * j1Ab01;
    //         cbb22 += j1Bb02 * j1Ab02;
    //         cbb23 += j1Bb02 * j1Ab03;

    //         cbb30 += j1Bb03 * j1Ab00;
    //         cbb31 += j1Bb03 * j1Ab01;
    //         cbb32 += j1Bb03 * j1Ab02;
    //         cbb33 += j1Bb03 * j1Ab03;

    //         cba00 += j1Ba00 * j1Ab00;
    //         cba01 += j1Ba00 * j1Ab01;
    //         cba02 += j1Ba00 * j1Ab02;
    //         cba03 += j1Ba00 * j1Ab03;

    //         cba10 += j1Ba01 * j1Ab00;
    //         cba11 += j1Ba01 * j1Ab01;
    //         cba12 += j1Ba01 * j1Ab02;
    //         cba13 += j1Ba01 * j1Ab03;

    //         cba20 += j1Ba02 * j1Ab00;
    //         cba21 += j1Ba02 * j1Ab01;
    //         cba22 += j1Ba02 * j1Ab02;
    //         cba23 += j1Ba02 * j1Ab03;

    //         cba30 += j1Ba03 * j1Ab00;
    //         cba31 += j1Ba03 * j1Ab01;
    //         cba32 += j1Ba03 * j1Ab02;
    //         cba33 += j1Ba03 * j1Ab03;

    //         cab00 += j1Bb00 * j1Aa00;
    //         cab01 += j1Bb00 * j1Aa01;
    //         cab02 += j1Bb00 * j1Aa02;
    //         cab03 += j1Bb00 * j1Aa03;

    //         cab10 += j1Bb01 * j1Aa00;
    //         cab11 += j1Bb01 * j1Aa01;
    //         cab12 += j1Bb01 * j1Aa02;
    //         cab13 += j1Bb01 * j1Aa03;

    //         cab20 += j1Bb02 * j1Aa00;
    //         cab21 += j1Bb02 * j1Aa01;
    //         cab22 += j1Bb02 * j1Aa02;
    //         cab23 += j1Bb02 * j1Aa03;

    //         cab30 += j1Bb03 * j1Aa00;
    //         cab31 += j1Bb03 * j1Aa01;
    //         cab32 += j1Bb03 * j1Aa02;
    //         cab33 += j1Bb03 * j1Aa03;

    //         caa00 += j1Ba00 * j1Aa00;
    //         caa01 += j1Ba00 * j1Aa01;
    //         caa02 += j1Ba00 * j1Aa02;
    //         caa03 += j1Ba00 * j1Aa03;

    //         caa10 += j1Ba01 * j1Aa00;
    //         caa11 += j1Ba01 * j1Aa01;
    //         caa12 += j1Ba01 * j1Aa02;
    //         caa13 += j1Ba01 * j1Aa03;

    //         caa20 += j1Ba02 * j1Aa00;
    //         caa21 += j1Ba02 * j1Aa01;
    //         caa22 += j1Ba02 * j1Aa02;
    //         caa23 += j1Ba02 * j1Aa03;

    //         caa30 += j1Ba03 * j1Aa00;
    //         caa31 += j1Ba03 * j1Aa01;
    //         caa32 += j1Ba03 * j1Aa02;
    //         caa33 += j1Ba03 * j1Aa03;
    //     }

    // }

    // cbb00 += j1Bb00 * j1Ab00;
    // cbb01 += j1Bb00 * j1Ab01;
    // cbb02 += j1Bb00 * j1Ab02;
    // cbb03 += j1Bb00 * j1Ab03;
    // cbb10 += j1Bb01 * j1Ab00;
    // cbb11 += j1Bb01 * j1Ab01;
    // cbb12 += j1Bb01 * j1Ab02;
    // cbb13 += j1Bb01 * j1Ab03;
    // cbb20 += j1Bb02 * j1Ab00;
    // cbb21 += j1Bb02 * j1Ab01;
    // cbb22 += j1Bb02 * j1Ab02;
    // cbb23 += j1Bb02 * j1Ab03;
    // cbb30 += j1Bb03 * j1Ab00;
    // cbb31 += j1Bb03 * j1Ab01;
    // cbb32 += j1Bb03 * j1Ab02;
    // cbb33 += j1Bb03 * j1Ab03;
    // cba00 += j1Ba00 * j1Ab00;
    // cba01 += j1Ba00 * j1Ab01;
    // cba02 += j1Ba00 * j1Ab02;
    // cba03 += j1Ba00 * j1Ab03;
    // cba10 += j1Ba01 * j1Ab00;
    // cba11 += j1Ba01 * j1Ab01;
    // cba12 += j1Ba01 * j1Ab02;
    // cba13 += j1Ba01 * j1Ab03;
    // cba20 += j1Ba02 * j1Ab00;
    // cba21 += j1Ba02 * j1Ab01;
    // cba22 += j1Ba02 * j1Ab02;
    // cba23 += j1Ba02 * j1Ab03;
    // cba30 += j1Ba03 * j1Ab00;
    // cba31 += j1Ba03 * j1Ab01;
    // cba32 += j1Ba03 * j1Ab02;
    // cba33 += j1Ba03 * j1Ab03;
    // cab00 += j1Bb00 * j1Aa00;
    // cab01 += j1Bb00 * j1Aa01;
    // cab02 += j1Bb00 * j1Aa02;
    // cab03 += j1Bb00 * j1Aa03;
    // cab10 += j1Bb01 * j1Aa00;
    // cab11 += j1Bb01 * j1Aa01;
    // cab12 += j1Bb01 * j1Aa02;
    // cab13 += j1Bb01 * j1Aa03;
    // cab20 += j1Bb02 * j1Aa00;
    // cab21 += j1Bb02 * j1Aa01;
    // cab22 += j1Bb02 * j1Aa02;
    // cab23 += j1Bb02 * j1Aa03;
    // cab30 += j1Bb03 * j1Aa00;
    // cab31 += j1Bb03 * j1Aa01;
    // cab32 += j1Bb03 * j1Aa02;
    // cab33 += j1Bb03 * j1Aa03;
    // caa00 += j1Ba00 * j1Aa00;
    // caa01 += j1Ba00 * j1Aa01;
    // caa02 += j1Ba00 * j1Aa02;
    // caa03 += j1Ba00 * j1Aa03;
    // caa10 += j1Ba01 * j1Aa00;
    // caa11 += j1Ba01 * j1Aa01;
    // caa12 += j1Ba01 * j1Aa02;
    // caa13 += j1Ba01 * j1Aa03;
    // caa20 += j1Ba02 * j1Aa00;
    // caa21 += j1Ba02 * j1Aa01;
    // caa22 += j1Ba02 * j1Aa02;
    // caa23 += j1Ba02 * j1Aa03;
    // caa30 += j1Ba03 * j1Aa00;
    // caa31 += j1Ba03 * j1Aa01;
    // caa32 += j1Ba03 * j1Aa02;
    // caa33 += j1Ba03 * j1Aa03;

    int tid31 = tid & 31;
    int tid32 = tid & 32;
    int coord_x = readBs & 0x7f;
    int coord_y = readAs & 0x7f;
    int writeCs = coord_x / 4 * 64 + coord_y;
    int readCs = (tid32 << 3) + tid31;
    int ldc4 = M * 4;
    int Cy00 = (tid32 >> 1) * M + tid31;
    int Cy04 = Cy00 + ldc4;
    int Cy08 = Cy00 + 2 * ldc4;
    int Cy12 = Cy00 + 3 * ldc4;

    a_b_shm[writeCs + 0] = cbb00;
    a_b_shm[writeCs + 1] = cbb01;
    a_b_shm[writeCs + 2] = cbb02;
    a_b_shm[writeCs + 3] = cbb03;
    a_b_shm[writeCs + 32 + 0] = cab00;
    a_b_shm[writeCs + 32 + 1] = cab01;
    a_b_shm[writeCs + 32 + 2] = cab02;
    a_b_shm[writeCs + 32 + 3] = cab03;
    // if (threadIdx.x == 1) {
    //     printf("reg r0,  c4: %f\n", cbb00);
    // }
    // if (threadIdx.x == 18) {
    //     printf("reg r7,  c8: %f\n", cbb03);
    //     printf("reg r39, c8: %f\n", cab03);
    // }

    cbb00 = a_b_shm[readCs + 0 * 64 + 0 ];
    cbb01 = a_b_shm[readCs + 0 * 64 + 32];
    cbb02 = a_b_shm[readCs + 1 * 64 + 0 ];
    cbb03 = a_b_shm[readCs + 1 * 64 + 32];
    cab00 = a_b_shm[readCs + 2 * 64 + 0 ];
    cab01 = a_b_shm[readCs + 2 * 64 + 32];
    cab02 = a_b_shm[readCs + 3 * 64 + 0 ];
    cab03 = a_b_shm[readCs + 3 * 64 + 32];
    c[Cy00 + 0 ] = cbb00;
    c[Cy00 + 32] = cbb01;
    c[Cy04 + 0 ] = cbb02;
    c[Cy04 + 32] = cbb03;
    c[Cy08 + 0 ] = cab00;
    c[Cy08 + 32] = cab01;
    c[Cy12 + 0 ] = cab02;
    c[Cy12 + 32] = cab03;

    Cy00 += M;
    Cy04 += M;
    Cy08 += M;
    Cy12 += M;
    a_b_shm[writeCs + 0] = cbb10;
    a_b_shm[writeCs + 1] = cbb11;
    a_b_shm[writeCs + 2] = cbb12;
    a_b_shm[writeCs + 3] = cbb13;
    a_b_shm[writeCs + 32 + 0] = cab10;
    a_b_shm[writeCs + 32 + 1] = cab11;
    a_b_shm[writeCs + 32 + 2] = cab12;
    a_b_shm[writeCs + 32 + 3] = cab13;

    cbb10 = a_b_shm[readCs + 0 * 64 + 0 ];
    cbb11 = a_b_shm[readCs + 0 * 64 + 32];
    cbb12 = a_b_shm[readCs + 1 * 64 + 0 ];
    cbb13 = a_b_shm[readCs + 1 * 64 + 32];
    cab10 = a_b_shm[readCs + 2 * 64 + 0 ];
    cab11 = a_b_shm[readCs + 2 * 64 + 32];
    cab12 = a_b_shm[readCs + 3 * 64 + 0 ];
    cab13 = a_b_shm[readCs + 3 * 64 + 32];
    c[Cy00 + 0 ] = cbb10;
    c[Cy00 + 32] = cbb11;
    c[Cy04 + 0 ] = cbb12;
    c[Cy04 + 32] = cbb13;
    c[Cy08 + 0 ] = cab10;
    c[Cy08 + 32] = cab11;
    c[Cy12 + 0 ] = cab12;
    c[Cy12 + 32] = cab13;

    Cy00 += M;
    Cy04 += M;
    Cy08 += M;
    Cy12 += M;
    a_b_shm[writeCs + 0] = cbb20;
    a_b_shm[writeCs + 1] = cbb21;
    a_b_shm[writeCs + 2] = cbb22;
    a_b_shm[writeCs + 3] = cbb23;
    a_b_shm[writeCs + 32 + 0] = cab20;
    a_b_shm[writeCs + 32 + 1] = cab21;
    a_b_shm[writeCs + 32 + 2] = cab22;
    a_b_shm[writeCs + 32 + 3] = cab23;

    cbb20 = a_b_shm[readCs + 0 * 64 + 0 ];
    cbb21 = a_b_shm[readCs + 0 * 64 + 32];
    cbb22 = a_b_shm[readCs + 1 * 64 + 0 ];
    cbb23 = a_b_shm[readCs + 1 * 64 + 32];
    cab20 = a_b_shm[readCs + 2 * 64 + 0 ];
    cab21 = a_b_shm[readCs + 2 * 64 + 32];
    cab22 = a_b_shm[readCs + 3 * 64 + 0 ];
    cab23 = a_b_shm[readCs + 3 * 64 + 32];
    c[Cy00 + 0 ] = cbb20;
    c[Cy00 + 32] = cbb21;
    c[Cy04 + 0 ] = cbb22;
    c[Cy04 + 32] = cbb23;
    c[Cy08 + 0 ] = cab20;
    c[Cy08 + 32] = cab21;
    c[Cy12 + 0 ] = cab22;
    c[Cy12 + 32] = cab23;

    Cy00 += M;
    Cy04 += M;
    Cy08 += M;
    Cy12 += M;
    a_b_shm[writeCs + 0] = cbb30;
    a_b_shm[writeCs + 1] = cbb31;
    a_b_shm[writeCs + 2] = cbb32;
    a_b_shm[writeCs + 3] = cbb33;
    a_b_shm[writeCs + 32 + 0] = cab30;
    a_b_shm[writeCs + 32 + 1] = cab31;
    a_b_shm[writeCs + 32 + 2] = cab32;
    a_b_shm[writeCs + 32 + 3] = cab33;

    cbb30 = a_b_shm[readCs + 0 * 64 + 0 ];
    cbb31 = a_b_shm[readCs + 0 * 64 + 32];
    cbb32 = a_b_shm[readCs + 1 * 64 + 0 ];
    cbb33 = a_b_shm[readCs + 1 * 64 + 32];
    cab30 = a_b_shm[readCs + 2 * 64 + 0 ];
    cab31 = a_b_shm[readCs + 2 * 64 + 32];
    cab32 = a_b_shm[readCs + 3 * 64 + 0 ];
    cab33 = a_b_shm[readCs + 3 * 64 + 32];
    c[Cy00 + 0 ] = cbb30;
    c[Cy00 + 32] = cbb31;
    c[Cy04 + 0 ] = cbb32;
    c[Cy04 + 32] = cbb33;
    c[Cy08 + 0 ] = cab30;
    c[Cy08 + 32] = cab31;
    c[Cy12 + 0 ] = cab32;
    c[Cy12 + 32] = cab33;

    Cy00 += M;
    Cy04 += M;
    Cy08 += M;
    Cy12 += M;

    Cy00 += 28 * M;
    Cy04 += 28 * M;
    Cy08 += 28 * M;
    Cy12 += 28 * M;

    a_b_shm[writeCs + 0] = cba00;
    a_b_shm[writeCs + 1] = cba01;
    a_b_shm[writeCs + 2] = cba02;
    a_b_shm[writeCs + 3] = cba03;
    a_b_shm[writeCs + 32 + 0] = caa00;
    a_b_shm[writeCs + 32 + 1] = caa01;
    a_b_shm[writeCs + 32 + 2] = caa02;
    a_b_shm[writeCs + 32 + 3] = caa03;

    cba00 = a_b_shm[readCs + 0 * 64 + 0 ];
    cba01 = a_b_shm[readCs + 0 * 64 + 32];
    cba02 = a_b_shm[readCs + 1 * 64 + 0 ];
    cba03 = a_b_shm[readCs + 1 * 64 + 32];
    caa00 = a_b_shm[readCs + 2 * 64 + 0 ];
    caa01 = a_b_shm[readCs + 2 * 64 + 32];
    caa02 = a_b_shm[readCs + 3 * 64 + 0 ];
    caa03 = a_b_shm[readCs + 3 * 64 + 32];
    c[Cy00 + 0 ] = cba00;
    c[Cy00 + 32] = cba01;
    c[Cy04 + 0 ] = cba02;
    c[Cy04 + 32] = cba03;
    c[Cy08 + 0 ] = caa00;
    c[Cy08 + 32] = caa01;
    c[Cy12 + 0 ] = caa02;
    c[Cy12 + 32] = caa03;

    Cy00 += M;
    Cy04 += M;
    Cy08 += M;
    Cy12 += M;
    a_b_shm[writeCs + 0] = cba10;
    a_b_shm[writeCs + 1] = cba11;
    a_b_shm[writeCs + 2] = cba12;
    a_b_shm[writeCs + 3] = cba13;
    a_b_shm[writeCs + 32 + 0] = caa10;
    a_b_shm[writeCs + 32 + 1] = caa11;
    a_b_shm[writeCs + 32 + 2] = caa12;
    a_b_shm[writeCs + 32 + 3] = caa13;

    cba10 = a_b_shm[readCs + 0 * 64 + 0 ];
    cba11 = a_b_shm[readCs + 0 * 64 + 32];
    cba12 = a_b_shm[readCs + 1 * 64 + 0 ];
    cba13 = a_b_shm[readCs + 1 * 64 + 32];
    caa10 = a_b_shm[readCs + 2 * 64 + 0 ];
    caa11 = a_b_shm[readCs + 2 * 64 + 32];
    caa12 = a_b_shm[readCs + 3 * 64 + 0 ];
    caa13 = a_b_shm[readCs + 3 * 64 + 32];
    c[Cy00 + 0 ] = cba10;
    c[Cy00 + 32] = cba11;
    c[Cy04 + 0 ] = cba12;
    c[Cy04 + 32] = cba13;
    c[Cy08 + 0 ] = caa10;
    c[Cy08 + 32] = caa11;
    c[Cy12 + 0 ] = caa12;
    c[Cy12 + 32] = caa13;

    Cy00 += M;
    Cy04 += M;
    Cy08 += M;
    Cy12 += M;
    a_b_shm[writeCs + 0] = cba20;
    a_b_shm[writeCs + 1] = cba21;
    a_b_shm[writeCs + 2] = cba22;
    a_b_shm[writeCs + 3] = cba23;
    a_b_shm[writeCs + 32 + 0] = caa20;
    a_b_shm[writeCs + 32 + 1] = caa21;
    a_b_shm[writeCs + 32 + 2] = caa22;
    a_b_shm[writeCs + 32 + 3] = caa23;

    cba20 = a_b_shm[readCs + 0 * 64 + 0 ];
    cba21 = a_b_shm[readCs + 0 * 64 + 32];
    cba22 = a_b_shm[readCs + 1 * 64 + 0 ];
    cba23 = a_b_shm[readCs + 1 * 64 + 32];
    caa20 = a_b_shm[readCs + 2 * 64 + 0 ];
    caa21 = a_b_shm[readCs + 2 * 64 + 32];
    caa22 = a_b_shm[readCs + 3 * 64 + 0 ];
    caa23 = a_b_shm[readCs + 3 * 64 + 32];
    c[Cy00 + 0 ] = cba20;
    c[Cy00 + 32] = cba21;
    c[Cy04 + 0 ] = cba22;
    c[Cy04 + 32] = cba23;
    c[Cy08 + 0 ] = caa20;
    c[Cy08 + 32] = caa21;
    c[Cy12 + 0 ] = caa22;
    c[Cy12 + 32] = caa23;

    Cy00 += M;
    Cy04 += M;
    Cy08 += M;
    Cy12 += M;
    a_b_shm[writeCs + 0] = cba30;
    a_b_shm[writeCs + 1] = cba31;
    a_b_shm[writeCs + 2] = cba32;
    a_b_shm[writeCs + 3] = cba33;
    a_b_shm[writeCs + 32 + 0] = caa30;
    a_b_shm[writeCs + 32 + 1] = caa31;
    a_b_shm[writeCs + 32 + 2] = caa32;
    a_b_shm[writeCs + 32 + 3] = caa33;

    cba30 = a_b_shm[readCs + 0 * 64 + 0 ];
    cba31 = a_b_shm[readCs + 0 * 64 + 32];
    cba32 = a_b_shm[readCs + 1 * 64 + 0 ];
    cba33 = a_b_shm[readCs + 1 * 64 + 32];
    caa30 = a_b_shm[readCs + 2 * 64 + 0 ];
    caa31 = a_b_shm[readCs + 2 * 64 + 32];
    caa32 = a_b_shm[readCs + 3 * 64 + 0 ];
    caa33 = a_b_shm[readCs + 3 * 64 + 32];
    c[Cy00 + 0 ] = cba30;
    c[Cy00 + 32] = cba31;
    c[Cy04 + 0 ] = cba32;
    c[Cy04 + 32] = cba33;
    c[Cy08 + 0 ] = caa30;
    c[Cy08 + 32] = caa31;
    c[Cy12 + 0 ] = caa32;
    c[Cy12 + 32] = caa33;
}

__global__ void cuda_kernel_sgemm_100_v2(
        float *a, float *b, float *c,
        size_t M, size_t N, size_t K,
        float alpha, float beta) {
    int block_x = blockIdx.x;
    int block_y = blockIdx.y;
    float* block_a = a + block_y * 64;
    float* block_b = b + block_x * 64;
    float* block_c = c + block_x * 64 * M + block_y * 64;
    sgemm_block_64x64_v2(block_a, block_b, block_c, M, N, K, alpha, beta);
}

__global__ void sgemm_block_test(
        float *a, float *b, float *c,
        size_t M, size_t N, size_t K,
        float alpha, float beta) {

    __shared__ float a_b_shm[2 * 16 * 64];

    int tid = threadIdx.x & 0x3f;
    int ldx = N;
    int tid2 = (tid >> 4) & 1;
    // int tid15 = tid & 0xf;
    int tid15 = tid & 15;
    int track0 = 8;
    // int track2 = track0 + 2 * ldx;
    // int track4 = track0 + 4 * ldx;
    // int track6 = track0 + 6 * ldx;
    int end = track0 + (K - 8) * ldx;
    int write_offset = tid2 * 64 + tid15 * 4;

    // int readAs = ((tid >> 1) & 7) << 2;
    // int readBs = ((((tid & 0x30) >> 3) | (tid & 1)) << 2) + 512;
    int readAs = 2;
    int readBs = 3;

    // float* read_addr = tid >= 32 ? b : a;
    float* read_addr = b;

    while (track0 <= end) {
        a_b_shm[write_offset + 0 * 64 + 0] = read_addr[track0 + 0];
        __syncthreads();

        write_offset ^= 16 * 64;
        track0 += 8 * ldx;
        // track2 += 8 * ldx;
        // track4 += 8 * ldx;
        // track6 += 8 * ldx;

        readAs ^= 16 * 64;
        // readBs ^= 16 * 64;

        a[track0] = a_b_shm[readAs];
        // b[track2] = a_b_shm[readBs];
        // c[track6] = c[track4] + a[write_offset] + b[write_offset];
    }

    // while (track0 < end) {
    //     a_b_shm[write_offset + 2 * 64 + 0] = read_addr[track2 + 0];
    //     __syncwarp(0xFFFFFFFF);

    //     write_offset ^= 16 * 64;
    //     readAs ^= 16 * 64;
    //     readBs ^= 16 * 64;

    //     a[track0] = a_b_shm[readAs];
    //     b[track2] = a_b_shm[readBs];
    //     c[track6] = c[track4] + a[write_offset] + b[write_offset];
    // }

}
