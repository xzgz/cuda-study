#include "cutlass/gemm/device/gemm.h"
#include "matmul_kernel.h"

// #define USE_TENSOR_CORE

hipError_t CutlassSgemmNN(int M, int N, int K, float alpha, float const* A, int lda, float const* B, int ldb,
        float beta, float* C, int ldc, int cycle_count) {
    // for (int i = 0; i < cycle_count; ++i) {

#ifndef USE_TENSOR_CORE

    using CutlassGemm = cutlass::gemm::device::Gemm<float, // Data-type of A matrix
            cutlass::layout::RowMajor,                     // Layout of A matrix
            float,                                         // Data-type of B matrix
            cutlass::layout::RowMajor,                     // Layout of B matrix
            float,                                         // Data-type of C matrix
            cutlass::layout::RowMajor, float, cutlass::arch::OpClassSimt, cutlass::arch::Sm80,
            cutlass::gemm::GemmShape<256, 128, 8>, cutlass::gemm::GemmShape<64, 64, 8>,
            cutlass::gemm::GemmShape<1, 1, 1>
            // cutlass::gemm::GemmShape<32, 32, 8>,
            // cutlass::gemm::GemmShape<16, 16, 8>,
            // cutlass::gemm::GemmShape<1, 1, 1>,
            >;

    // Define a CUTLASS GEMM type
    CutlassGemm gemm_operator;
    CutlassGemm::Arguments args({M, N, K}, // Gemm Problem dimensions
            {A, lda},                      // Tensor-ref for source matrix A
            {B, ldb},                      // Tensor-ref for source matrix B
            {C, ldc},                      // Tensor-ref for source matrix C
            {C, ldc},       // Tensor-ref for destination matrix D (may be different memory than source C matrix)
            {alpha, beta}); // Scalars used in the Epilogue

#else

    using ElementAccumulator = float;                  // <- data type of accumulator
    using ElementComputeEpilogue = ElementAccumulator; // <- data type of epilogue operations
    using ElementInputA = cutlass::half_t;             // <- data type of elements in input matrix A
    using ElementInputB = cutlass::half_t;             // <- data type of elements in input matrix B
    using ElementOutput = float;                       // <- data type of elements in output matrix D

    // The code section below describes matrix layout of input and output matrices. Column Major for
    // Matrix A, Row Major for Matrix B and Row Major for Matrix C
    using LayoutInputA = cutlass::layout::RowMajor;
    using LayoutInputB = cutlass::layout::RowMajor;
    using LayoutOutput = cutlass::layout::RowMajor;

    // This code section describes whether you want to use tensor cores or regular SIMT cores on GPU SM
    using MMAOp = cutlass::arch::OpClassTensorOp;

    // This code section describes CUDA SM architecture number
    using SmArch = cutlass::arch::Sm80;

    // This code section describes the tile size a thread block will compute
    using ShapeMMAThreadBlock = cutlass::gemm::GemmShape<128, 128, 16>; // <- threadblock tile M = 128, N = 128,
    // This code section describes tile size a warp will compute
    using ShapeMMAWarp = cutlass::gemm::GemmShape<64, 64, 16>; // <- warp tile M = 64, N = 64, K = 16
    // This code section describes the size of MMA op
    using ShapeMMAOp = cutlass::gemm::GemmShape<16, 8, 8>; // <- MMA Op tile M = 16, N = 8, K = 8

    // This code section describes how threadblocks are scheduled on GPU
    using SwizzleThreadBlock = cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>; // <- ??

    // This code section describes the epilogue part of the kernel
    using EpilogueOp = cutlass::epilogue::thread::LinearCombination<ElementOutput, // <- data type of output matrix
            128 / cutlass::sizeof_bits<ElementOutput>::value, // <- the number of elements per vectorized
                                                              // memory access. For a byte, it's 16
                                                              // elements. This becomes the vector width of
                                                              // math instructions in the epilogue too
            ElementAccumulator,                               // <- data type of accumulator
            ElementComputeEpilogue>; // <- data type for alpha/beta in linear combination function

    // Number of pipelines you want to use
    constexpr int NumStages = 4;

    using CutlassGemm = cutlass::gemm::device::Gemm<ElementInputA, LayoutInputA, ElementInputB, LayoutInputB,
            ElementOutput, LayoutOutput, ElementAccumulator, MMAOp, SmArch, ShapeMMAThreadBlock, ShapeMMAWarp,
            ShapeMMAOp, EpilogueOp, SwizzleThreadBlock, NumStages>;

    // Define a CUTLASS GEMM type
    CutlassGemm gemm_operator;

    CutlassGemm::Arguments args({M, N, K}, // Gemm Problem dimensions
            {(ElementInputA*)A, lda},      // Tensor-ref for source matrix A
            {(ElementInputB*)B, ldb},      // Tensor-ref for source matrix B
            {C, ldc},                      // Tensor-ref for source matrix C
            {C, ldc},       // Tensor-ref for destination matrix D (may be different memory than source C matrix)
            {alpha, beta}); // Scalars used in the Epilogue

#endif

    // Launch the CUTLASS GEMM kernel.
    cutlass::Status status = gemm_operator(args);

    if (status != cutlass::Status::kSuccess) {
        return hipErrorUnknown;
    }
    // }

    return hipSuccess;
}
