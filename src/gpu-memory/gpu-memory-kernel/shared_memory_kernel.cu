// Copyright 2022.
// All rights reserved.
// @author heyanguang
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <fstream>
#include <iostream>

#define CUDA_CHECK(condition)                                    \
    /* Code block avoids redefinition of cudaError_t error */    \
    do {                                                         \
        hipError_t error = condition;                           \
        if (error != hipSuccess) {                              \
            std::cout << hipGetErrorString(error) << std::endl; \
        }                                                        \
    } while (0)

#define DIVUP(m, n) (((m) / (n)) + ((m) % (n) > 0))

#define CUDA_1D_KERNEL_LOOP(i, n) \
    for (int32_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x)

namespace memory_test {

template <typename T = float>
__global__ void GlobalToDynamicShared(const T *input, T *output) {
    extern __shared__ float shared_memory_pool[];
    T *shared_memory_addr = reinterpret_cast<T*>(shared_memory_pool);

    int32_t block_thread_count = blockDim.x * blockDim.y;
    uint64_t shared_memory_element_count = gridDim.y * block_thread_count;
    uint64_t global_addr = blockIdx.x * shared_memory_element_count + blockIdx.y * block_thread_count
            + threadIdx.y * blockDim.x + threadIdx.x;
    uint64_t shared_addr = blockIdx.y * block_thread_count + threadIdx.y * blockDim.x + threadIdx.x;

    shared_memory_addr[shared_addr] = input[global_addr];
//    shared_memory_addr[shared_addr] = 0.5;

//    T var;
//    var = shared_memory_addr[shared_addr];
//    var = input[global_addr];
//    shared_memory_addr[shared_addr] = var;
}

template <typename T = float>
__global__ void GlobalToGlobal(const T *input, T *output) {
    int32_t block_thread_count = blockDim.x * blockDim.y;

//    uint64_t shared_memory_element_count = gridDim.y * block_thread_count;
//    uint64_t global_addr = blockIdx.x * shared_memory_element_count + blockIdx.y * block_thread_count
//            + threadIdx.y * blockDim.x + threadIdx.x;

    uint64_t shared_memory_element_count = gridDim.x * block_thread_count;
    uint64_t global_addr = blockIdx.y * shared_memory_element_count + blockIdx.x * block_thread_count
            + threadIdx.y * blockDim.x + threadIdx.x;

    output[global_addr] = input[global_addr];
    // T val = input[global_addr];
    // T val = 0.5;
    // output[global_addr] = val;

    // T var;
    // var = input[global_addr];
    // output[global_addr] = var;
}

template <typename T = float>
__global__ void GlobalToGlobalV4(const T *input, T *output) {
    int32_t block_thread_count = blockDim.x * blockDim.y;
    uint64_t shared_memory_element_count = gridDim.x * block_thread_count;
    uint64_t global_addr = blockIdx.y * shared_memory_element_count + blockIdx.x * block_thread_count
        + threadIdx.y * blockDim.x + threadIdx.x;

//    output[global_addr * 4 + 0] = input[global_addr * 4 + 0];
//    output[global_addr * 4 + 1] = input[global_addr * 4 + 1];
//    output[global_addr * 4 + 2] = input[global_addr * 4 + 2];
//    output[global_addr * 4 + 3] = input[global_addr * 4 + 3];

    asm volatile(
    "{\n\t"
    ".reg.f32 a<4>;\n\t"
    ".reg.u64 rd, wr;\n\t"
    "add.u64 rd, %0, %2;\n\t"
    "add.u64 wr, %1, %2;\n\t"
    "ld.global.v4.f32 { a0, a1, a2, a3 }, [rd];\n\t"
    "st.global.v4.f32 [wr], { a0, a1, a2, a3 };\n\t"
    "}"
    :
    : "l"(input), "l"(output), "l"(global_addr * 16)
    : "memory"
    );
}

template <typename T = float>
__global__ void GlobalToDynamicSharedToGlobal(const T *input, const int32_t sm_element_count, T *output) {
    extern __shared__ float shared_memory_pool[];
    // __shared__ float other[32 * 1024];
    T *shared_memory_addr = reinterpret_cast<T*>(shared_memory_pool);

    int32_t block_thread_count = blockDim.x * blockDim.y;
    uint64_t shared_memory_element_count = gridDim.x * block_thread_count;
    uint64_t global_addr = blockIdx.y * shared_memory_element_count + blockIdx.x * block_thread_count
            + threadIdx.y * blockDim.x + threadIdx.x;
    uint64_t shared_addr = blockIdx.x * block_thread_count + threadIdx.y * blockDim.x + threadIdx.x;

    // shared_memory_addr[shared_addr] = input[global_addr];
    // output[global_addr] = shared_memory_addr[shared_addr];

    if (shared_addr < sm_element_count) {
        // T val = input[global_addr];
        // shared_memory_addr[shared_addr] = val;
        // other[shared_addr] = T(1.0) - val;
        // output[global_addr] = other[shared_addr] + shared_memory_addr[shared_addr];

        shared_memory_addr[shared_addr] = input[global_addr];
        output[global_addr] = shared_memory_addr[shared_addr];
    } else {
        output[global_addr] = input[global_addr];
    }
}

template <typename T = float>
__global__ void SharedToShared(const T *input, T *output) {
    extern __shared__ float shared_memory_pool[];
    int32_t block_thread_count = blockDim.x * blockDim.y;
    uint64_t shared_memory_element_count = gridDim.y * block_thread_count;
    T *shared_memory_addr1 = reinterpret_cast<T*>(shared_memory_pool);
    T *shared_memory_addr2 = shared_memory_addr1 + shared_memory_element_count;

    uint64_t shared_addr = blockIdx.y * block_thread_count + threadIdx.y * blockDim.x + threadIdx.x;

    shared_memory_addr2[shared_addr] = shared_memory_addr1[shared_addr];
//    shared_memory_addr2[shared_addr] = 0.5;
}

template <typename T = float>
__global__ void SharedToSharedV4(const T *input, T *output) {
    extern __shared__ float shared_memory_pool[];
    int32_t block_thread_count = blockDim.x * blockDim.y;
    uint64_t shared_memory_element_count = gridDim.y * block_thread_count;
    T *shared_memory_addr1 = reinterpret_cast<T*>(shared_memory_pool);
    T *shared_memory_addr2 = shared_memory_addr1 + shared_memory_element_count;

    uint64_t shared_addr = blockIdx.y * block_thread_count + threadIdx.y * blockDim.x + threadIdx.x;

//    shared_memory_addr2[shared_addr] = shared_memory_addr1[shared_addr];
//    shared_memory_addr2[shared_addr] = 0.5;

//    asm volatile (
//    "{\n\t"
//    ".reg.f32 a<4>;\n\t"
//    ".reg.u32 smem_ptr32_0, smem_ptr32_1, rd, wr;\n\t"
//    ".reg.u64 smem_ptr64_0, smem_ptr64_1;\n\t"
//    "cvta.to.shared.u64 smem_ptr64_0, %0;\n\t"
//    "cvta.to.shared.u64 smem_ptr64_1, %1;\n\t"
//    "cvt.u32.u64 smem_ptr32_0, smem_ptr64_0;\n\t"
//    "cvt.u32.u64 smem_ptr32_1, smem_ptr64_1;\n\t"
//    "add.u32 smem_ptr32_0, smem_ptr32_0, %2;\n\t"
//    "add.u32 smem_ptr32_1, smem_ptr32_1, %2;\n\t"
//    "ld.shared.v4.f32 { a0, a1, a2, a3 }, [smem_ptr32_0];\n\t"
//    "st.shared.v4.f32 [smem_ptr32_1], { a0, a1, a2, a3 };\n\t"
//    "}"
//    :
//    : "l"(shared_memory_addr1), "l"(shared_memory_addr2), "r"(uint32_t(shared_addr * 16))
//    : "memory"
//    );

    asm volatile (
    "{\n\t"
    ".reg.f32 a<4>;\n\t"
    ".reg.u64 smem_ptr64_0, smem_ptr64_1;\n\t"
    "cvta.to.shared.u64 smem_ptr64_0, %0;\n\t"
    "cvta.to.shared.u64 smem_ptr64_1, %1;\n\t"
    "add.u64 smem_ptr64_0, smem_ptr64_0, %2;\n\t"
    "add.u64 smem_ptr64_1, smem_ptr64_1, %2;\n\t"
    "ld.shared.v4.f32 { a0, a1, a2, a3 }, [smem_ptr64_0];\n\t"
    "st.shared.v4.f32 [smem_ptr64_1], { a0, a1, a2, a3 };\n\t"
    "}"
    :
    : "l"(shared_memory_addr1), "l"(shared_memory_addr2), "l"(shared_addr * 16)
    : "memory"
    );

//    printf("addr1=%p, addr2=%p\n", shared_memory_addr1, shared_memory_addr2);
}

namespace functor {

template <typename T>
int32_t LaunchGlobalToDynamicShared(hipStream_t stream, const T *input, T *output,
        const uint64_t shared_memory_size, const uint64_t cycle_count) {
    int32_t block_count = shared_memory_size / (1024 * sizeof(T));
    dim3 block(32, 32);
    dim3 grid(cycle_count, block_count);

    GlobalToDynamicShared<T><<<grid, block, shared_memory_size, stream>>>(input, output);

    return 1;
}

template <typename T>
int32_t LaunchGlobalToGlobal(hipStream_t stream, const T *input, T *output,
    const uint64_t shared_memory_size, const uint64_t block_size, const uint64_t grid_y) {
    int32_t sm_element_count = shared_memory_size / sizeof(T);
    int32_t grid_x = (sm_element_count + block_size - 1) / block_size;

    dim3 block(1024);
    // dim3 block(32, 32);
    // dim3 grid(grid_x, grid_y);
    dim3 grid(grid_y, grid_x);

    GlobalToGlobal<T><<<grid, block, 0, stream>>>(input, output);

    return 1;
}

template <typename T>
int32_t LaunchGlobalToGlobalV4(hipStream_t stream, const T *input, T *output,
    const uint64_t shared_memory_size, const uint64_t block_size, const uint64_t grid_y) {
    int32_t sm_element_count = shared_memory_size / sizeof(T);
    int32_t grid_x = (sm_element_count + block_size - 1) / block_size;

    dim3 block(1024);
    // dim3 block(32, 32);
    dim3 grid(grid_x, grid_y / 4);

    GlobalToGlobalV4<T><<<grid, block, 0, stream>>>(input, output);

    return 1;
}

template <typename T>
int32_t LaunchGlobalToDynamicSharedToGlobal(hipStream_t stream, const T *input, T *output,
    const uint64_t shared_memory_size, const uint64_t block_size, const uint64_t grid_y) {
    int32_t sm_element_count = shared_memory_size / sizeof(T);
    int32_t grid_x = (sm_element_count + block_size - 1) / block_size;

    dim3 block(32, 32);
    dim3 grid(grid_x, grid_y);

    if (shared_memory_size > 48 * 1024) {
        hipFuncSetAttribute(reinterpret_cast<const void*>(GlobalToDynamicSharedToGlobal<T>), hipFuncAttributeMaxDynamicSharedMemorySize, shared_memory_size);
    }
    GlobalToDynamicSharedToGlobal<T><<<grid, block, shared_memory_size, stream>>>(input, sm_element_count, output);

    return 1;
}

template <typename T>
int32_t LaunchSharedToShared(hipStream_t stream, const T *input, T *output,
        const uint64_t shared_memory_size, const uint64_t cycle_count) {
    int32_t block_count = shared_memory_size / (2 * 1024 * sizeof(T));
    dim3 block(32, 32);
    dim3 grid(cycle_count, block_count);
//    dim3 grid(block_count, cycle_count);

    SharedToShared<T><<<grid, block, shared_memory_size, stream>>>(input, output);

    return 1;
}

template <typename T>
int32_t LaunchSharedToSharedV4(hipStream_t stream, const T *input, T *output,
                               const uint64_t shared_memory_size, const uint64_t cycle_count) {
    int32_t block_count = shared_memory_size / (2 * 1024 * sizeof(T) * 4);
    dim3 block(128, 8);
    dim3 grid(cycle_count, block_count);
//    dim3 grid(block_count, cycle_count);

    SharedToSharedV4<T><<<grid, block, shared_memory_size, stream>>>(input, output);

    return 1;
}

template int32_t LaunchGlobalToDynamicShared(hipStream_t stream, const float *input, float *output,
        const uint64_t shared_memory_size, const uint64_t cycle_count);
template int32_t LaunchGlobalToGlobal(hipStream_t stream, const float *input, float *output,
        const uint64_t shared_memory_size, const uint64_t block_size, const uint64_t grid_y);
template int32_t LaunchGlobalToGlobalV4(hipStream_t stream, const float *input, float *output,
        const uint64_t shared_memory_size, const uint64_t block_size, const uint64_t grid_y);
template int32_t LaunchGlobalToDynamicSharedToGlobal(hipStream_t stream, const float *input, float *output,
        const uint64_t shared_memory_size, const uint64_t block_size, const uint64_t grid_y);
template int32_t LaunchSharedToShared(hipStream_t stream, const float *input, float *output,
        const uint64_t shared_memory_size, const uint64_t cycle_count);
template int32_t LaunchSharedToSharedV4(hipStream_t stream, const float *input, float *output,
        const uint64_t shared_memory_size, const uint64_t cycle_count);
}
}
