#include "hip/hip_runtime.h"
// Copyright 2020, Tencent Inc.
// All rights reserved.
//
// @author shaorunwang <shaorunwang@tencent.com>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <thrust/extrema.h>

#include <fstream>
#include <iostream>

//#include "NvInfer.h"
//#include "sparse_fwffm_plugin.h"

#define CUDA_CHECK(condition)                                    \
/* Code block avoids redefinition of hipError_t error */    \
do {                                                         \
hipError_t error = condition;                           \
if (error != hipSuccess) {                              \
std::cout << hipGetErrorString(error) << std::endl; \
}                                                        \
} while (0)

#define DIVUP(m, n) (((m) / (n)) + ((m) % (n) > 0))

#define CUDA_1D_KERNEL_LOOP(i, n) \
for (int32_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x)

namespace nvinfer1 {

namespace sparse_fwffm {

__global__ void ComputeBatchBoundary(const int32_t* index_tensor, int32_t total_feature_num,
                                     int32_t batch_size, int32_t* sample_feature_start_addr,
                                     int32_t* sample_feature_end_addr) {
    int32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < total_feature_num) {
        int32_t idx = index_tensor[tid];
        // atomicMin(sample_feature_start_addr + idx, tid);
        // atomicMax(sample_feature_end_addr + idx, tid + 1);
        if (tid > 0) {
            int32_t pre_idx = index_tensor[tid - 1];
            for (int32_t i = idx; i > pre_idx; --i) {
                sample_feature_start_addr[i] = tid;
            }
        } else {
            int32_t first_idx = index_tensor[0];
            for (int32_t i = 0; i <= first_idx; ++i) {
                sample_feature_start_addr[i] = 0;
            }
            int32_t last_idx = index_tensor[total_feature_num - 1];
            for (int32_t i = batch_size - 1; i > last_idx; --i) {
                sample_feature_start_addr[i] = total_feature_num;
            }
            sample_feature_start_addr[batch_size] = total_feature_num;
        }
    }
}

template <typename T = float, int32_t warp_num = 32>
__global__ void ProcessCommonPart(int32_t embedding_size, int32_t field_num, int32_t fw_field_num,
                                  int32_t* sample_feature_start_addr, const T* weight_tensor,
                                  const int32_t* field_tensor, T* gmem_fw_cross_mean_sum,
                                  T* gmem_fw_cross_square_sum, int32_t* gmem_fw_field_map) {
    int32_t warp_id = threadIdx.y;
    int32_t lane_id = threadIdx.x;

    int32_t global_warp_id = blockIdx.x * warp_num + warp_id;
    int32_t total_global_warp_num = gridDim.x * warp_num;

    int32_t tid = threadIdx.x + threadIdx.y * blockDim.x;
    int32_t total_thread = warp_num * 32;
    int32_t common_feature_num = sample_feature_start_addr[0];

    for (int32_t wid = global_warp_id; wid < common_feature_num; wid += total_global_warp_num) {
        int32_t field_1 = field_tensor[wid * 2] - 1;
        int32_t fw_field_1 = field_tensor[wid * 2 + 1] - 1;

        if (fw_field_1 < 0 || fw_field_1 >= fw_field_num || field_1 < 0 || field_1 >= field_num)
            continue;

        if (lane_id == 0) gmem_fw_field_map[fw_field_1] = field_1;

#pragma unroll
        for (int32_t field_2 = 0; field_2 < field_num; field_2++) {
            int32_t mem_field_offset = (field_2 * fw_field_num + fw_field_1) * embedding_size;
#pragma unroll
            for (int32_t n = 0; n < (embedding_size + 31) / 32 * 32; n += 32) {
                if (n + lane_id < embedding_size) {
                    T reg = weight_tensor[wid * embedding_size * field_num +
                                          field_2 * embedding_size + n + lane_id];
                    atomicAdd(gmem_fw_cross_mean_sum + mem_field_offset + n + lane_id, reg);
                }
            }
        }

        int32_t mem_field_offset = fw_field_1 * embedding_size;
#pragma unroll
        for (int32_t n = 0; n < (embedding_size + 31) / 32 * 32; n += 32) {
            if (n + lane_id < embedding_size) {
                T reg = weight_tensor[wid * embedding_size * field_num + field_1 * embedding_size +
                                      n + lane_id];
                float square = reg * reg;
                atomicAdd(gmem_fw_cross_square_sum + mem_field_offset + n + lane_id, square);
            }
        }
    }
}

template <typename T = float>
__global__ void BroadcastCommonPart(int32_t batch, int32_t embedding_size, int32_t field_num,
                                    int32_t fw_field_num, T* gmem_fw_cross_mean_sum,
                                    T* gmem_fw_cross_square_sum, T* output) {

    int32_t lane_id = threadIdx.x;
    int32_t fw_field_id = blockIdx.x % fw_field_num;
    int32_t tid = lane_id + fw_field_id * embedding_size;
    int32_t bid = blockIdx.x / fw_field_num;
    // printf("bid : %d, tid: %d\n",bid, tid);
    T Reg_square = gmem_fw_cross_square_sum[tid];
    output[bid * (embedding_size * (field_num + 1) * fw_field_num) +
    embedding_size * field_num * fw_field_num + tid] = Reg_square;

    T Reg_mean_0 = gmem_fw_cross_mean_sum[tid];
    T Reg_mean_1 = gmem_fw_cross_mean_sum[embedding_size * fw_field_num + tid];
    output[bid * (embedding_size * (field_num + 1) * fw_field_num) + tid] = Reg_mean_0;
    output[bid * (embedding_size * (field_num + 1) * fw_field_num) + embedding_size * fw_field_num +
    tid] = Reg_mean_1;
}

template <typename T = float, int32_t warp_num = 32>
__device__ void ProcessSamplePart(
int32_t* smem_fw_field_map, int32_t* smem_fw_map_idx, T* gmem_cross_mean_sum,
T* gmem_cross_square_sum, T* gmem_fw_cross_mean_sum, T* gmem_fw_cross_square_sum,
int32_t* gmem_fw_field_map, int32_t embedding_size, int32_t field_num, int32_t fw_field_num,
int32_t this_sample_feature_num, int32_t this_sample_feature_start_addr,
int32_t sample_0_feature_num, int32_t sample_0_feature_start_addr, const T* weight_tensor,
const int* field_tensor, int32_t shared_mem_elements) {

    constexpr int32_t total_thread = warp_num * 32;

    int32_t warp_id = threadIdx.y;
    int32_t lane_id = threadIdx.x;

    int32_t tid = threadIdx.x + threadIdx.y * 32;

    for (int32_t i = 0; i < fw_field_num; i += warp_num * 32) {
        if (i + tid < fw_field_num) {
            smem_fw_field_map[i + tid] = gmem_fw_field_map[i + tid];
        }
    }
    for (int32_t i = fw_field_num; i < fw_field_num * 2; i += warp_num * 32) {
        if (i + tid < fw_field_num * 2) {
            smem_fw_field_map[i + tid] = -1;
        }
    }

    __syncthreads();

    // patch
    if (blockIdx.x != 0) {
        int32_t sample_0_start_row = warp_id + sample_0_feature_start_addr;
        int32_t sample_0_end_row = sample_0_feature_num + sample_0_feature_start_addr;
        for (int32_t wid = sample_0_start_row; wid < sample_0_end_row; wid += warp_num) {
            int32_t field_1 = field_tensor[wid * 2] - 1;
            int32_t fw_field_1 = field_tensor[wid * 2 + 1] - 1;
            if (fw_field_1 < 0 || fw_field_1 >= fw_field_num || field_1 < 0 || field_1 >= field_num)
                continue;

            if (lane_id == 0) smem_fw_field_map[fw_field_1] = field_1;
        }
    }

    // sample feature phase

    int32_t common_fw_field_map_offset_for_ad = blockIdx.x > 0 ? fw_field_num : 0;

    int32_t sample_start_row = warp_id + this_sample_feature_start_addr;
    int32_t sample_end_row = this_sample_feature_num + this_sample_feature_start_addr;
    for (int32_t wid = sample_start_row; wid < sample_end_row; wid += warp_num) {
        int32_t field_1 = field_tensor[wid * 2] - 1;
        int32_t fw_field_1 = field_tensor[wid * 2 + 1] - 1;
        if (fw_field_1 < 0 || fw_field_1 >= fw_field_num || field_1 < 0 || field_1 >= field_num)
            continue;

        if (lane_id == 0) {
            smem_fw_field_map[fw_field_1 + common_fw_field_map_offset_for_ad] = field_1;
        }

#pragma unroll
        for (int32_t field_2 = 0; field_2 < field_num; field_2++) {
            int32_t mem_field_offset = (field_2 * fw_field_num + fw_field_1) * embedding_size;
#pragma unroll
            for (int32_t n = 0; n < (embedding_size + 31) / 32 * 32; n += 32) {
                T reg = T(0);
                int32_t rd_offset =
                wid * embedding_size * field_num + field_2 * embedding_size + n + lane_id;
                T* wr_ptr = gmem_cross_mean_sum + mem_field_offset + n + lane_id;

                if (n + lane_id < embedding_size) {
                    reg = weight_tensor[rd_offset];
                    atomicAdd(wr_ptr, reg);
                }
            }
        }
        int32_t mem_field_offset = fw_field_1 * embedding_size;
#pragma unroll
        for (int32_t n = 0; n < (embedding_size + 31) / 32 * 32; n += 32) {
            T reg = T(0);
            T square = T(0);
            int32_t rd_offset =
            wid * embedding_size * field_num + field_1 * embedding_size + n + lane_id;
            T* wr_ptr = gmem_cross_square_sum + mem_field_offset + n + lane_id;

            if (n + lane_id < embedding_size) {
                reg = weight_tensor[rd_offset];
                square = reg * reg;
                atomicAdd(wr_ptr, square);
            }
        }
    }

    __syncthreads();
    int32_t i = 0;
    for (int32_t i = 0; i < fw_field_num; i += total_thread) {
        if (i + tid < fw_field_num) {
            int32_t field_1 = smem_fw_field_map[common_fw_field_map_offset_for_ad + i + tid];
            int32_t field_1_part1 = smem_fw_field_map[i + tid];
            if (field_1 < 0 && field_1_part1 != 1) {
                field_1 = field_1_part1;
            }
            smem_fw_field_map[i + tid] = field_1;
        }
    }
    __syncthreads();
    if (warp_id == 0 && lane_id == 0) {
        int32_t cnt = 0;
        for (int32_t i = 0; i < fw_field_num; i++) {
            if (smem_fw_field_map[i] >= 0) {
                smem_fw_map_idx[cnt++] = i;
            }
        }
        smem_fw_map_idx[fw_field_num] = cnt;
    }
    __syncthreads();
}

template <typename T = float, int32_t warp_num = 32>
__device__ void ProcessSamplePart_share(
int32_t* smem_fw_field_map, int32_t* smem_fw_map_idx, T* smem_cross_mean_sum,
T* smem_cross_square_sum, T* gmem_fw_cross_mean_sum, T* gmem_fw_cross_square_sum,
int32_t* gmem_fw_field_map, int32_t embedding_size, int32_t field_num, int32_t fw_field_num,
int32_t this_sample_feature_num, int32_t this_sample_feature_start_addr,
int32_t sample_0_feature_num, int32_t sample_0_feature_start_addr, const T* weight_tensor,
const int* field_tensor, int32_t shared_mem_elements) {

    constexpr int32_t total_thread = warp_num * 32;

    int32_t warp_id = threadIdx.y;
    int32_t lane_id = threadIdx.x;

    int32_t tid = threadIdx.x + threadIdx.y * 32;

    for (int32_t i = 0; i < fw_field_num; i += warp_num * 32) {
        if (i + tid < fw_field_num) {
            smem_fw_field_map[i + tid] = gmem_fw_field_map[i + tid];
        }
    }
    for (int32_t i = fw_field_num; i < fw_field_num * 2; i += warp_num * 32) {
        if (i + tid < fw_field_num * 2) {
            smem_fw_field_map[i + tid] = -1;
        }
    }

    for (int32_t i = tid; i < embedding_size * field_num * fw_field_num; i += warp_num * 32) {
        smem_cross_mean_sum[i] = gmem_fw_cross_mean_sum[i];
        if (i < embedding_size * fw_field_num) {
            smem_cross_square_sum[i] = gmem_fw_cross_square_sum[i];
        }
    }

    __syncthreads();

    // patch
    if (blockIdx.x != 0) {
        int32_t sample_0_start_row = warp_id + sample_0_feature_start_addr;
        int32_t sample_0_end_row = sample_0_feature_num + sample_0_feature_start_addr;
        for (int32_t wid = sample_0_start_row; wid < sample_0_end_row; wid += warp_num) {
            int32_t field_1 = field_tensor[wid * 2] - 1;
            int32_t fw_field_1 = field_tensor[wid * 2 + 1] - 1;
            if (fw_field_1 < 0 || fw_field_1 >= fw_field_num || field_1 < 0 || field_1 >= field_num)
                continue;

            if (lane_id == 0) smem_fw_field_map[fw_field_1] = field_1;
        }
    }

    // sample feature phase

    int32_t common_fw_field_map_offset_for_ad = blockIdx.x > 0 ? fw_field_num : 0;

    int32_t sample_start_row = warp_id + this_sample_feature_start_addr;
    int32_t sample_end_row = this_sample_feature_num + this_sample_feature_start_addr;
    for (int32_t wid = sample_start_row; wid < sample_end_row; wid += warp_num) {
        int32_t field_1 = field_tensor[wid * 2] - 1;
        int32_t fw_field_1 = field_tensor[wid * 2 + 1] - 1;
        if (fw_field_1 < 0 || fw_field_1 >= fw_field_num || field_1 < 0 || field_1 >= field_num)
            continue;

        if (lane_id == 0) {
            smem_fw_field_map[fw_field_1 + common_fw_field_map_offset_for_ad] = field_1;
        }

#pragma unroll
        for (int32_t field_2 = 0; field_2 < field_num; field_2++) {
            int32_t mem_field_offset = (field_2 * fw_field_num + fw_field_1) * embedding_size;
#pragma unroll
            for (int32_t n = 0; n < (embedding_size + 31) / 32 * 32; n += 32) {
                T reg = T(0);
                int32_t rd_offset =
                wid * embedding_size * field_num + field_2 * embedding_size + n + lane_id;
                T* wr_ptr = smem_cross_mean_sum + mem_field_offset + n + lane_id;

                if (n + lane_id < embedding_size) {
                    reg = weight_tensor[rd_offset];
                    atomicAdd(wr_ptr, reg);
                }
            }
        }
        int32_t mem_field_offset = fw_field_1 * embedding_size;
#pragma unroll
        for (int32_t n = 0; n < (embedding_size + 31) / 32 * 32; n += 32) {
            T reg = T(0);
            T square = T(0);
            int32_t rd_offset =
            wid * embedding_size * field_num + field_1 * embedding_size + n + lane_id;
            T* wr_ptr = smem_cross_square_sum + mem_field_offset + n + lane_id;

            if (n + lane_id < embedding_size) {
                reg = weight_tensor[rd_offset];
                square = reg * reg;
                atomicAdd(wr_ptr, square);
            }
        }
    }

    __syncthreads();
    int32_t i = 0;
    for (int32_t i = 0; i < fw_field_num; i += total_thread) {
        if (i + tid < fw_field_num) {
            int32_t field_1 = smem_fw_field_map[common_fw_field_map_offset_for_ad + i + tid];
            int32_t field_1_part1 = smem_fw_field_map[i + tid];
            if (field_1 < 0 && field_1_part1 != 1) {
                field_1 = field_1_part1;
            }
            smem_fw_field_map[i + tid] = field_1;
        }
    }
    __syncthreads();
    if (warp_id == 0 && lane_id == 0) {
        int32_t cnt = 0;
        for (int32_t i = 0; i < fw_field_num; i++) {
            if (smem_fw_field_map[i] >= 0) {
                smem_fw_map_idx[cnt++] = i;
            }
        }
        smem_fw_map_idx[fw_field_num] = cnt;
    }
    __syncthreads();
}
template <typename T = float, int32_t warp_num = 32>
__device__ void ProcessOutput_share(T* smem_cross_mean_sum, T* smem_cross_square_sum,
                                    T* mem_fw_cross_mean_sum, T* mem_fw_cross_square_sum,
                                    int32_t* smem_fw_field_map, int32_t* smem_fw_map_idx,
                                    const T* fw_weight_tensor, T* output_gmem, int32_t batch_id,
                                    int32_t embedding_size, int32_t field_num,
                                    int32_t fw_field_num) {

    int32_t warp_id = threadIdx.y;
    int32_t lane_id = threadIdx.x;

    int32_t weight_size_pad = (embedding_size + 31) / 32 * 32;
    int32_t common_fw_field_map_offset_for_ad = blockIdx.x > 0 ? fw_field_num : 0;

    // T output_accu[(embedding_size + 31) / 32] = {0};
    T output_accu[4] = {0};
    int32_t total_vaild_fw_field = smem_fw_map_idx[fw_field_num];

    for (int32_t fw_field_1_idx = warp_id; fw_field_1_idx < total_vaild_fw_field;
    fw_field_1_idx += warp_num) {
        int32_t fw_field_1 = smem_fw_map_idx[fw_field_1_idx];
        int32_t fw_iter = (2 + fw_field_1) * (fw_field_1 + 1) / 2 - (fw_field_1 + 1);

        int32_t field_1 = smem_fw_field_map[fw_field_1];

        for (int32_t fw_field_2_idx = 0; fw_field_2_idx < fw_field_1_idx; fw_field_2_idx++) {
            int32_t fw_field_2 = smem_fw_map_idx[fw_field_2_idx];
            int32_t field_2 = smem_fw_field_map[fw_field_2];

            T fw_weight_reg = fw_weight_tensor[fw_iter + fw_field_2] + T(1);
            int32_t index_1 = (field_1 * fw_field_num + fw_field_2) * embedding_size;
            int32_t index_2 = (field_2 * fw_field_num + fw_field_1) * embedding_size;

            for (int32_t n = 0; n < weight_size_pad; n += 32) {

                T mean_index_1_sum = T(0);
                T mean_index_2_sum = T(0);

                if (n + lane_id < embedding_size) {
                    mean_index_1_sum = smem_cross_mean_sum[index_1 + n + lane_id];
                    mean_index_2_sum = smem_cross_mean_sum[index_2 + n + lane_id];
                }
                output_accu[n / 32] += mean_index_1_sum * mean_index_2_sum * fw_weight_reg;
            }
        }
        T fw_weight_reg = fw_weight_tensor[fw_iter + fw_field_1] + T(1);
        int32_t index_1 = (field_1 * fw_field_num + fw_field_1) * embedding_size;
        for (int32_t n = 0; n < weight_size_pad; n += 32) {

            T cross_mean_sum = T(0);
            T cross_square_sum = T(0);
            if (n + lane_id < embedding_size) {
                cross_mean_sum = smem_cross_mean_sum[index_1 + n + lane_id];
                cross_square_sum = smem_cross_square_sum[fw_field_1 * embedding_size + n + lane_id];
            }
            output_accu[n / 32] +=
            T(0.5) * (cross_mean_sum * cross_mean_sum - cross_square_sum) * fw_weight_reg;
        }
    }

    for (int32_t n = 0; n < weight_size_pad; n += 32) {
        if (n + lane_id < embedding_size) {
            T* Outptr = (output_gmem + batch_id * embedding_size + n + lane_id);
            atomicAdd(Outptr, output_accu[n / 32]);
        }
    }
}

template <typename T = float, int32_t warp_num = 32>
__device__ void ProcessOutput(T* mem_cross_mean_sum, T* mem_cross_square_sum,
                              T* mem_fw_cross_mean_sum, T* mem_fw_cross_square_sum,
                              int32_t* smem_fw_field_map, int32_t* smem_fw_map_idx,
                              const T* fw_weight_tensor, T* output_gmem, int32_t batch_id,
                              int32_t embedding_size, int32_t field_num, int32_t fw_field_num) {

    int32_t warp_id = threadIdx.y;
    int32_t lane_id = threadIdx.x;

    int32_t weight_size_pad = (embedding_size + 31) / 32 * 32;
    int32_t common_fw_field_map_offset_for_ad = blockIdx.x > 0 ? fw_field_num : 0;

    // T output_accu[(embedding_size + 31) / 32] = {0};
    T output_accu[6] = { 0 };
    int32_t total_vaild_fw_field = smem_fw_map_idx[fw_field_num];

    for (int32_t fw_field_1_idx = warp_id; fw_field_1_idx < total_vaild_fw_field;
    fw_field_1_idx += warp_num) {
        int32_t fw_field_1 = smem_fw_map_idx[fw_field_1_idx];
        int32_t fw_iter = (2 + fw_field_1) * (fw_field_1 + 1) / 2 - (fw_field_1 + 1);

        int32_t field_1 = smem_fw_field_map[fw_field_1];

        for (int32_t fw_field_2_idx = 0; fw_field_2_idx < fw_field_1_idx; fw_field_2_idx++) {
            int32_t fw_field_2 = smem_fw_map_idx[fw_field_2_idx];
            int32_t field_2 = smem_fw_field_map[fw_field_2];

            T fw_weight_reg = fw_weight_tensor[fw_iter + fw_field_2] + T(1);
            int32_t index_1 = (field_1 * fw_field_num + fw_field_2) * embedding_size;
            int32_t index_2 = (field_2 * fw_field_num + fw_field_1) * embedding_size;

            for (int32_t n = 0; n < weight_size_pad; n += 32) {

                T mean_index_1_sum = T(0);
                T mean_index_2_sum = T(0);

                if (n + lane_id < embedding_size) {
                    mean_index_1_sum = mem_cross_mean_sum[index_1 + n + lane_id];
                    mean_index_2_sum = mem_cross_mean_sum[index_2 + n + lane_id];
                }
                output_accu[n / 32] += mean_index_1_sum * mean_index_2_sum * fw_weight_reg;
            }
        }
        T fw_weight_reg = fw_weight_tensor[fw_iter + fw_field_1] + T(1);
        int32_t index_1 = (field_1 * fw_field_num + fw_field_1) * embedding_size;
        for (int32_t n = 0; n < weight_size_pad; n += 32) {

            T cross_mean_sum = T(0);
            T cross_square_sum = T(0);
            if (n + lane_id < embedding_size) {
                cross_mean_sum = mem_cross_mean_sum[index_1 + n + lane_id];
                cross_square_sum = mem_cross_square_sum[fw_field_1 * embedding_size + n + lane_id];
            }
            output_accu[n / 32] +=
            T(0.5) * (cross_mean_sum * cross_mean_sum - cross_square_sum) * fw_weight_reg;
        }
    }

    for (int32_t n = 0; n < weight_size_pad; n += 32) {
        if (n + lane_id < embedding_size) {
            T* Outptr = (output_gmem + batch_id * embedding_size + n + lane_id);
            atomicAdd(Outptr, output_accu[n / 32]);
        }
    }
}

template <typename T = float, int32_t warp_num = 32>
__global__ void ProcessFwffmOutput(int32_t embedding_size, int32_t field_num, int32_t fw_field_num,
                                   bool fw_weight_multil_flag, int32_t* sample_feature_start_addr,
                                   int32_t* sample_feature_end_addr, const T* weight_tensor,
                                   const int32_t* field_tensor, const T* fw_weight_tensor,
                                   T* output_tensor, T* workspace)

                                   {
    int32_t batch_size = gridDim.x;
    int32_t warp_id = threadIdx.y;

    extern __shared__ float smem_pool[];
    int32_t batch_id = blockIdx.x;
    int32_t fw_weight_size = (fw_field_num + 1) * fw_field_num / 2;
    int32_t* smem_fw_field_map = reinterpret_cast<int32_t*>(smem_pool);
    int32_t* smem_fw_map_idx = smem_fw_field_map + 2 * fw_field_num;
    // Use global memory in case of lacking atomicAdd float in shared mem

    T* mem_cross_mean_sum =
    workspace + batch_id * (embedding_size * (field_num + 1) * fw_field_num);
    T* mem_cross_square_sum = mem_cross_mean_sum + embedding_size * field_num * fw_field_num;
    T* mem_fw_cross_mean_sum =
    workspace + batch_size * (embedding_size * (field_num + 1) * fw_field_num);
    T* mem_fw_cross_square_sum = mem_fw_cross_mean_sum + embedding_size * field_num * fw_field_num;

    int32_t* mem_fw_field_map =
    reinterpret_cast<int*>(mem_fw_cross_square_sum + fw_field_num * embedding_size);

    const T* local_fw_weight_data = fw_weight_tensor;
    if (fw_weight_multil_flag) {
        local_fw_weight_data = fw_weight_tensor + batch_id * fw_weight_size;
    }

    int32_t this_sample_feature_start_addr = sample_feature_start_addr[batch_id];
    int32_t this_sample_feature_end_addr = sample_feature_start_addr[batch_id + 1];
    int32_t this_sample_feature_num = this_sample_feature_end_addr - this_sample_feature_start_addr;

    int32_t sample_0_feature_start_addr = sample_feature_start_addr[0];
    int32_t sample_0_feature_end_addr = sample_feature_start_addr[1];
    int32_t sample_0_feature_num = sample_0_feature_end_addr - sample_0_feature_start_addr;

    ProcessSamplePart<T, warp_num>(
    smem_fw_field_map, smem_fw_map_idx, mem_cross_mean_sum, mem_cross_square_sum,
    mem_fw_cross_mean_sum, mem_fw_cross_square_sum, mem_fw_field_map, embedding_size, field_num,
    fw_field_num, this_sample_feature_num, this_sample_feature_start_addr, sample_0_feature_num,
    sample_0_feature_start_addr, weight_tensor, field_tensor, warp_num * embedding_size);

    ProcessOutput<T, warp_num>(mem_cross_mean_sum, mem_cross_square_sum, mem_fw_cross_mean_sum,
                               mem_fw_cross_square_sum, smem_fw_field_map, smem_fw_map_idx,
                               local_fw_weight_data, output_tensor, batch_id, embedding_size, field_num,
                               fw_field_num);
                                   }

                                   template <typename T = float, int32_t warp_num = 32>
                                   __global__ void ProcessFwffmOutput_share(int32_t embedding_size, int32_t field_num,
                                                                            int32_t fw_field_num, bool fw_weight_multil_flag,
                                                                            int32_t* sample_feature_start_addr,
                                                                            int32_t* sample_feature_end_addr, const T* weight_tensor,
                                                                            const int32_t* field_tensor, const T* fw_weight_tensor,
                                                                            T* output_tensor, T* workspace) {
    int32_t batch_size = gridDim.x;
    int32_t warp_id = threadIdx.y;

    extern __shared__ float smem_pool[];
    int32_t batch_id = blockIdx.x;
    int32_t fw_weight_size = (fw_field_num + 1) * fw_field_num / 2;
    int32_t* smem_fw_field_map = reinterpret_cast<int32_t*>(smem_pool);
    int32_t* smem_fw_map_idx = smem_fw_field_map + 2 * fw_field_num;
    // Use global memory in case of lacking atomicAdd float in shared mem

    T* smem_cross_mean_sum = reinterpret_cast<T*>(smem_fw_map_idx + fw_field_num + 1);
    T* smem_cross_square_sum = smem_cross_mean_sum + embedding_size * field_num * fw_field_num;
    T* mem_fw_cross_mean_sum =
    workspace + batch_size * (embedding_size * (field_num + 1) * fw_field_num);
    T* mem_fw_cross_square_sum = mem_fw_cross_mean_sum + embedding_size * field_num * fw_field_num;

    int32_t* mem_fw_field_map =
    reinterpret_cast<int*>(mem_fw_cross_square_sum + fw_field_num * embedding_size);

    const T* local_fw_weight_data = fw_weight_tensor;
    if (fw_weight_multil_flag) {
        local_fw_weight_data = fw_weight_tensor + batch_id * fw_weight_size;
    }

    int32_t this_sample_feature_start_addr = sample_feature_start_addr[batch_id];
    int32_t this_sample_feature_end_addr = sample_feature_start_addr[batch_id + 1];
    int32_t this_sample_feature_num = this_sample_feature_end_addr - this_sample_feature_start_addr;

    int32_t sample_0_feature_start_addr = sample_feature_start_addr[0];
    int32_t sample_0_feature_end_addr = sample_feature_start_addr[1];
    int32_t sample_0_feature_num = sample_0_feature_end_addr - sample_0_feature_start_addr;

    ProcessSamplePart_share<T, warp_num>(
    smem_fw_field_map, smem_fw_map_idx, smem_cross_mean_sum, smem_cross_square_sum,
    mem_fw_cross_mean_sum, mem_fw_cross_square_sum, mem_fw_field_map, embedding_size, field_num,
    fw_field_num, this_sample_feature_num, this_sample_feature_start_addr, sample_0_feature_num,
    sample_0_feature_start_addr, weight_tensor, field_tensor, warp_num * embedding_size);

    ProcessOutput_share<T, warp_num>(
    smem_cross_mean_sum, smem_cross_square_sum, mem_fw_cross_mean_sum, mem_fw_cross_square_sum,
    smem_fw_field_map, smem_fw_map_idx, fw_weight_tensor, output_tensor, batch_id,
    embedding_size, field_num, fw_field_num);
}

namespace functor {
template <typename T>
int32_t ComputeSparseFwffm(hipStream_t stream, const void* const* input, T* output,
                           void* worksapce, const int32_t fw_field_num,
                           const int32_t fw_weight_size, const bool fw_weight_multil_flag,
                           const int32_t sample_feature_size, const int32_t field_num,
                           const int32_t embedding_size, const int32_t batch_size) {
    const T* weight_data = static_cast<const T*>(input[0]);
    const T* fw_weight_data = static_cast<const T*>(input[1]);
    const int32_t* field_data = static_cast<const int32_t*>(input[2]);
    const int32_t* index_data = static_cast<const int32_t*>(input[3]);

    const int32_t kThreadsPerBlock = 1024;
    const size_t kBufferSize = field_num * fw_field_num * embedding_size;

    int32_t* sample_feature_start_addr = reinterpret_cast<int32_t*>(worksapce);
    int32_t* sample_feature_end_addr = sample_feature_start_addr + batch_size + 1;
    T* gmem_cross_sum = reinterpret_cast<T*>(sample_feature_end_addr + batch_size + 1);
    T* gmem_fw_cross_mean_sum = gmem_cross_sum + batch_size * (field_num + 1) * embedding_size * fw_field_num;
    T* gmem_fw_cross_square_sum = gmem_fw_cross_mean_sum + embedding_size * field_num * fw_field_num;

    int32_t* gmem_fw_field_map =
    reinterpret_cast<int*>(gmem_fw_cross_square_sum + embedding_size * fw_field_num);

    CUDA_CHECK(hipMemsetAsync(gmem_fw_cross_mean_sum, 0,
                               sizeof(float) * (embedding_size * field_num * fw_field_num +
                               embedding_size * fw_field_num), stream));
    CUDA_CHECK(hipMemsetAsync(gmem_fw_field_map, -1, sizeof(int) * (fw_field_num), stream));
    CUDA_CHECK(hipMemsetAsync(output, 0, sizeof(float) * (batch_size * embedding_size), stream));

    CUDA_CHECK(hipMemsetAsync(sample_feature_start_addr, 0, sizeof(int32_t) * (batch_size + 1), stream));
    CUDA_CHECK(hipMemsetAsync(sample_feature_end_addr, 0, sizeof(int32_t) * (batch_size + 1), stream));

    ComputeBatchBoundary<<<DIVUP(sample_feature_size, 1024), 1024, 0, stream>>>(
    index_data, sample_feature_size, batch_size, sample_feature_start_addr,
    sample_feature_end_addr);
    constexpr int32_t warp_num = 32;
    dim3 block(32, 32);
    dim3 grid0(1);

    ProcessCommonPart<T, 32><<<grid0, block, 0, stream>>>(
    embedding_size, field_num, fw_field_num, sample_feature_start_addr, weight_data, field_data,
    gmem_fw_cross_mean_sum, gmem_fw_cross_square_sum, gmem_fw_field_map);
    int32_t share_mem_size = (fw_field_num * 3 + 1) * sizeof(int32_t) +
    embedding_size * (field_num + 1) * fw_field_num * sizeof(T);

    if (share_mem_size < 65536) {
        hipFuncSetAttribute(reinterpret_cast<const void*>(ProcessFwffmOutput_share<T), warp_num>,
                             hipFuncAttributeMaxDynamicSharedMemorySize, 65536);
        dim3 grid(batch_size);

        ProcessFwffmOutput_share<T, warp_num><<<grid, block, share_mem_size, stream>>>(
        embedding_size, field_num, fw_field_num, fw_weight_multil_flag,
        sample_feature_start_addr, sample_feature_end_addr, weight_data, field_data,
        fw_weight_data, output, gmem_cross_sum);
    } else {
        //                    printf("Do not use shared memory.\n");
        dim3 block_set(embedding_size);
        dim3 grid_set(batch_size * fw_field_num);
        BroadcastCommonPart<T><<<grid_set, block_set, 0, stream>>>(
        batch_size, embedding_size, field_num, fw_field_num, gmem_fw_cross_mean_sum,
        gmem_fw_cross_square_sum, gmem_cross_sum);

        int32_t shared_mem_required_bytes = (fw_field_num * (field_num + 1) + 1)* sizeof(int32_t);
        dim3 grid(batch_size);
        hipFuncSetAttribute(reinterpret_cast<const void*>(ProcessFwffmOutput<T), warp_num>,
                             hipFuncAttributeMaxDynamicSharedMemorySize, 65536);

        ProcessFwffmOutput<T, warp_num><<<grid, block, shared_mem_required_bytes, stream>>>(
        embedding_size, field_num, fw_field_num, fw_weight_multil_flag,
        sample_feature_start_addr, sample_feature_end_addr, weight_data, field_data,
        fw_weight_data, output, gmem_cross_sum);
    }
    return 1;
}

template int32_t ComputeSparseFwffm(hipStream_t stream, const void* const* input, float* output,
void* worksapce, const int32_t fw_field_num,
const int32_t fw_weight_size, const bool fw_weight_multil_flag,
const int32_t sample_feature_size, const int32_t field_num,
const int32_t embedding_size, const int32_t batch_size);

template int32_t ComputeSparseFwffm(hipStream_t stream, const void* const* input, half* output,
void* worksapce, const int32_t fw_field_num,
const int32_t fw_weight_size, const bool fw_weight_multil_flag,
const int32_t sample_feature_size, const int32_t field_num,
const int32_t embedding_size, const int32_t batch_size);
}  // namespace functor
}  // namespace sparse_fwffm
}  // namespace nvinfer1