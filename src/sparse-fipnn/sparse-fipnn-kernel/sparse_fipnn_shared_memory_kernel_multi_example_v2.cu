#include "hip/hip_runtime.h"
// Copyright 2020, Tencent Inc.
// All rights reserved.
//
// @author shaorunwang <shaorunwang@tencent.com>
// @author hungryhe<hungryhe@tencent.com>

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <thrust/extrema.h>

#include <iostream>

#define CUDA_CHECK(condition)                                    \
/* Code block avoids redefinition of hipError_t error */    \
do {                                                         \
hipError_t error = condition;                           \
if (error != hipSuccess) {                              \
std::cout << hipGetErrorString(error) << std::endl; \
}                                                        \
} while (0)

#define DIVUP(m, n) (((m) / (n)) + ((m) % (n) > 0))

#define CUDA_1D_KERNEL_LOOP(i, n) \
for (int32_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x)

#define ALIGN_UP(x, align_count)    (((x) + ((align_count) - 1)) / (align_count) * (align_count))
#define DATA_ALIGN_BYTE_COUNT       128
#define DATA_ALIGN_INT32_COUNT      32

namespace nvinfer1 {

namespace sparse_fipnn_shared_multi {

template <typename T = float>
__device__ T __reduce_sum_across_warp(T val) {
    T rtn = val;
    //    __syncwarp(0xFFFFFFFF);
    for (int32_t i = 1; i < 32; i *= 2) {
        rtn += __shfl_xor_sync(0xFFFFFFFF, rtn, i);
    }
    return rtn;
}

__global__ void ComputeBatchBoundary(
    const int32_t* index_tensor, void* workspace,
    const int32_t* sample_count_prefix_sum_vec, const int32_t* sample_count_vec,
    const int32_t* batch_size_prefix_sum_vec, const int32_t* batch_size_vec) {
    int32_t total_thread = gridDim.x * blockDim.x * blockDim.y;
    int32_t tid = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
    int32_t batch_size = batch_size_vec[blockIdx.y];
    int32_t total_feature_num = sample_count_vec[blockIdx.y];

    uint32_t bound_data_offset = uint32_t(batch_size_prefix_sum_vec[blockIdx.y]) + blockIdx.y * DATA_ALIGN_INT32_COUNT;
    bound_data_offset = ALIGN_UP(bound_data_offset, DATA_ALIGN_INT32_COUNT);

    int32_t* sample_feature_start_addr = static_cast<int32_t*>(workspace) + bound_data_offset;
    const int32_t* index_ptr = index_tensor + sample_count_prefix_sum_vec[blockIdx.y];

    if (tid < batch_size + 1) {
        sample_feature_start_addr[tid] = 0;
    }
    __syncthreads();

    if (tid < total_feature_num) {
        int32_t idx = index_ptr[tid];
        if (tid > 0) {
            int32_t pre_idx = index_ptr[tid - 1];
            for (int32_t i = idx; i > pre_idx; --i) {
                sample_feature_start_addr[i] = tid;
            }
        } else {
            int32_t first_idx = index_ptr[0];
            for (int32_t i = 0; i <= first_idx; ++i) {
                sample_feature_start_addr[i] = 0;
            }
            int32_t last_idx = index_ptr[total_feature_num - 1];
            for (int32_t i = batch_size - 1; i > last_idx; --i) {
                sample_feature_start_addr[i] = total_feature_num;
            }
            sample_feature_start_addr[batch_size] = total_feature_num;
        }
    }
}

template <typename T>
__global__ void ComputeCommonPartOutput(
    const T* multi_weight_tensor, const int32_t* multi_field_tensor, void* workspace,
    const int32_t* sample_count_prefix_sum_vec, const int32_t* batch_size_prefix_sum_vec,
    const uint32_t common_cross_start_offset, const uint32_t fw_field_map_start_offset,
    const uint32_t common_output_start_offset, const uint32_t embedding_size,
    const uint32_t embedding_segment_size, const int32_t field_num, const uint32_t fw_field_num,
    const uint32_t segment_common_fw_cross_size, const uint32_t fw_weight_size) {

    const uint32_t data_align_count = DATA_ALIGN_BYTE_COUNT / sizeof(T);
    uint32_t common_cross_offset = common_cross_start_offset + blockIdx.y * ((field_num + 1) * fw_field_num * embedding_size + data_align_count);
    common_cross_offset = ALIGN_UP(common_cross_offset, data_align_count);
    uint32_t common_output_offset = common_output_start_offset + blockIdx.y * (fw_weight_size + data_align_count);
    common_output_offset = ALIGN_UP(common_output_offset, data_align_count);
    uint32_t bound_data_offset = uint32_t(batch_size_prefix_sum_vec[blockIdx.y]) + blockIdx.y * DATA_ALIGN_INT32_COUNT;
    bound_data_offset = ALIGN_UP(bound_data_offset, DATA_ALIGN_INT32_COUNT);
    uint32_t fw_field_map_offset = fw_field_map_start_offset + blockIdx.y * (2 * (fw_field_num + 1) + DATA_ALIGN_INT32_COUNT);
    fw_field_map_offset = ALIGN_UP(fw_field_map_offset, DATA_ALIGN_INT32_COUNT);

    const T* weight_tensor = multi_weight_tensor + sample_count_prefix_sum_vec[blockIdx.y] * field_num * embedding_size;
    const int32_t* field_tensor = multi_field_tensor + sample_count_prefix_sum_vec[blockIdx.y] * field_num;
    T* gmem_fw_cross_mean_sum = static_cast<T*>(workspace) + common_cross_offset;
    T* common_output = static_cast<T*>(workspace) + common_output_offset;
    int32_t* sample_feature_start_addr = static_cast<int32_t*>(workspace) + bound_data_offset;
    int32_t* gmem_fw_field_map = static_cast<int32_t*>(workspace) + fw_field_map_offset;

    extern __shared__ float smem_pool[];
    int32_t* smem_fw_field_map = reinterpret_cast<int32_t*>(smem_pool);
    int32_t* smem_fw_map_idx = smem_fw_field_map + fw_field_num + 1;
    T* smem_cross_mean_sum = reinterpret_cast<T*>(smem_fw_map_idx + fw_field_num + 1);
    T* smem_cross_square_sum = smem_cross_mean_sum + embedding_segment_size * field_num * fw_field_num;
    T* smem_output = smem_cross_square_sum + embedding_segment_size * fw_field_num;

    int32_t warp_id = threadIdx.y;
    int32_t lane_id = threadIdx.x;
    int32_t total_global_warp_num = blockDim.y;
    int32_t tid = threadIdx.x + threadIdx.y * blockDim.x;
    int32_t total_thread = blockDim.x * blockDim.y;
    int32_t embedding_segment_start = blockIdx.x * embedding_segment_size;
    int32_t segment_size_div_blockdimx = embedding_segment_size / blockDim.x;
    int32_t common_feature_num = sample_feature_start_addr[0];

    for (int32_t i = tid; i < segment_common_fw_cross_size; i += total_thread) {
        smem_cross_mean_sum[i] = T(0);
        if (i < fw_field_num) {
            smem_fw_field_map[i] = -1;
        }
    }
    for (int32_t i = tid; i < fw_weight_size; i += total_thread) {
        smem_output[i] = T(0);
    }
    if (blockIdx.x == 0) {
        for (int32_t i = tid; i < fw_weight_size; i += total_thread) {
            common_output[i] = T(0);
        }
    }
    __syncthreads();

    for (int32_t wid = warp_id; wid < common_feature_num; wid += total_global_warp_num) {
        int32_t field_1 = field_tensor[wid * 2] - 1;
        int32_t fw_field_1 = field_tensor[wid * 2 + 1] - 1;

        if (fw_field_1 < 0 || fw_field_1 >= fw_field_num || field_1 < 0 || field_1 >= field_num)
            continue;

        if (lane_id == 0) smem_fw_field_map[fw_field_1] = field_1;

#pragma unroll
        for (int32_t field_2 = 0; field_2 < field_num; field_2++) {
            int32_t mem_field_offset = (field_2 * fw_field_num + fw_field_1) * embedding_segment_size;
#pragma unroll
            for (int n = 0; n < segment_size_div_blockdimx; ++n) {
                T reg = T(0);
                int32_t rd_offset = wid * embedding_size * field_num + field_2 * embedding_size
                + embedding_segment_start + n * blockDim.x + lane_id;
                T *wr_ptr = smem_cross_mean_sum + mem_field_offset + n * blockDim.x + lane_id;

                if (embedding_segment_start + n * blockDim.x + lane_id < embedding_size) {
                    reg = weight_tensor[rd_offset];
                    atomicAdd(wr_ptr, reg);
                }
            }
        }

        int32_t mem_field_offset = fw_field_1 * embedding_segment_size;
#pragma unroll
        for (int n = 0; n < segment_size_div_blockdimx; ++n) {
            T reg = T(0);
            T square = T(0);
            int32_t rd_offset = wid * embedding_size * field_num + field_1 * embedding_size + embedding_segment_start
            + n * blockDim.x + lane_id;
            T *wr_ptr = smem_cross_square_sum + mem_field_offset + n * blockDim.x + lane_id;

            if (embedding_segment_start + n * blockDim.x + lane_id < embedding_size) {
                reg = weight_tensor[rd_offset];
                square = reg * reg;
                atomicAdd(wr_ptr, square);
            }
        }
    }
    __syncthreads();

    if (warp_id == 0 && lane_id == 0) {
        int32_t cnt = 0;
        for (int32_t i = 0; i < fw_field_num; i++) {
            if (smem_fw_field_map[i] >= 0) {
                smem_fw_map_idx[cnt++] = i;
            }
        }
        smem_fw_map_idx[fw_field_num] = cnt;
    }
    __syncthreads();

    int32_t total_vaild_fw_field = smem_fw_map_idx[fw_field_num];
    for (int32_t fw_field_1_idx = warp_id; fw_field_1_idx < total_vaild_fw_field;
    fw_field_1_idx += blockDim.y) {
        int32_t fw_field_1 = smem_fw_map_idx[fw_field_1_idx];
        int32_t field_1 = smem_fw_field_map[fw_field_1];
        int32_t fw_iter = fw_field_1 * (fw_field_1 + 1) / 2;

        int32_t index_1 = (field_1 * fw_field_num + fw_field_1) * embedding_segment_size;
        T output_value = T(0);
#pragma unroll
        for (int n = 0; n < segment_size_div_blockdimx; ++n) {
            T cross_mean_sum = T(0);
            T cross_square_sum = T(0);
            if (embedding_segment_start + n * blockDim.x + lane_id < embedding_size) {
                cross_mean_sum = smem_cross_mean_sum[index_1 + n * blockDim.x + lane_id];
                cross_square_sum = smem_cross_square_sum[fw_field_1 * embedding_segment_size + n * blockDim.x + lane_id];
            }
            output_value += T(0.5) * (cross_mean_sum * cross_mean_sum - cross_square_sum);
        }
        output_value = __reduce_sum_across_warp(output_value);
        // store here
        if (lane_id == 0) {
            smem_output[fw_iter + fw_field_1] = output_value;
        }
        //        __syncwarp(0xFFFFFFFF);

        for (int32_t fw_field_2_idx = 0; fw_field_2_idx < fw_field_1_idx; fw_field_2_idx++) {
            int32_t fw_field_2 = smem_fw_map_idx[fw_field_2_idx];
            int32_t field_2 = smem_fw_field_map[fw_field_2];
            int32_t index_1 = (field_1 * fw_field_num + fw_field_2) * embedding_segment_size;
            int32_t index_2 = (field_2 * fw_field_num + fw_field_1) * embedding_segment_size;

            T output_value = T(0);
#pragma unroll
            for (int n = 0; n < segment_size_div_blockdimx; ++n) {
                T mean_index_1_sum = T(0);
                T mean_index_2_sum = T(0);
                if (embedding_segment_start + n * blockDim.x + lane_id < embedding_size) {
                    mean_index_1_sum = smem_cross_mean_sum[index_1 + n * blockDim.x + lane_id];
                    mean_index_2_sum = smem_cross_mean_sum[index_2 + n * blockDim.x + lane_id];
                }
                output_value += mean_index_1_sum * mean_index_2_sum;
            }
            output_value = __reduce_sum_across_warp(output_value);
            // store here
            if (lane_id == 0) {
                smem_output[fw_iter + fw_field_2] = output_value;
            }
            //            __syncwarp(0xFFFFFFFF);
        }

    }
    __syncthreads();

    for (int32_t i = tid; i < fw_field_num + 1; i += total_thread) {
        gmem_fw_field_map[i] = smem_fw_field_map[i];
        gmem_fw_field_map[fw_field_num + 1 + i] = smem_fw_field_map[fw_field_num + 1 + i];
    }

    for (int32_t i = warp_id; i < (field_num + 1) * fw_field_num; i += blockDim.y) {
#pragma unroll
        for (int n = 0; n < segment_size_div_blockdimx; ++n) {
            int32_t embedding_offset = embedding_segment_start + n * blockDim.x + lane_id;
            if (embedding_offset < embedding_size) {
                gmem_fw_cross_mean_sum[i * embedding_size + embedding_offset]
                = smem_cross_mean_sum[i * embedding_segment_size + n * blockDim.x + lane_id];
            }
        }
    }

    for (int32_t i = tid; i < fw_weight_size; i += total_thread) {
        atomicAdd(common_output + i, smem_output[i]);
//        common_output[i] += smem_output[i];
//        atomicAdd(common_output + i, smem_output[i] / T(gridDim.x));
    }
}

template <typename T>
__device__ void AccumulateSamplePart(
int32_t* smem_fw_field_map, int32_t* smem_fw_map_idx, T* smem_cross_mean_sum,
T* smem_cross_square_sum, T* gmem_common_cross_mean_sum, T* gmem_common_cross_square_sum,
int32_t* gmem_common_field_map, int32_t embedding_size, int32_t embedding_segment_size,
int32_t field_num, int32_t fw_field_num, int32_t this_sample_feature_num, int32_t this_sample_feature_start_addr,
const T* weight_tensor, const int32_t* field_tensor) {
    int32_t warp_id = threadIdx.y;
    int32_t lane_id = threadIdx.x;
    int32_t tid = threadIdx.x + threadIdx.y * blockDim.x;
    int32_t total_thread = blockDim.x * blockDim.y;
    int32_t segment_size_div_blockdimx = embedding_segment_size / blockDim.x;
    int32_t embedding_segment_start = blockIdx.y * embedding_segment_size;

    for (int32_t i = tid; i < fw_field_num + 1; i += total_thread) {
        smem_fw_field_map[i] = -1;
        smem_fw_field_map[fw_field_num + 1 + i] = gmem_common_field_map[i];
        smem_fw_map_idx[fw_field_num + 1 + i] = gmem_common_field_map[fw_field_num + 1 + i];
    }

    for (int32_t i = warp_id; i < (field_num + 1) * fw_field_num; i += blockDim.y) {
#pragma unroll
        for (int n = 0; n < segment_size_div_blockdimx; ++n) {
            int32_t embedding_offset = embedding_segment_start + n * blockDim.x + lane_id;
            if (embedding_offset < embedding_size) {
                smem_cross_mean_sum[i * embedding_segment_size + n * blockDim.x + lane_id] =
                gmem_common_cross_mean_sum[i * embedding_size + embedding_offset];
            }
        }
    }
    __syncthreads();

    // sample feature phase
    int32_t sample_start_row = warp_id + this_sample_feature_start_addr;
    int32_t sample_end_row = this_sample_feature_num + this_sample_feature_start_addr;
    for (int32_t wid = sample_start_row; wid < sample_end_row; wid += blockDim.y) {
        int32_t field_1 = field_tensor[wid * 2] - 1;
        int32_t fw_field_1 = field_tensor[wid * 2 + 1] - 1;
        if (fw_field_1 < 0 || fw_field_1 >= fw_field_num || field_1 < 0 || field_1 >= field_num)
            continue;
        if (lane_id == 0) smem_fw_field_map[fw_field_1] = field_1;

#pragma unroll
        for (int32_t field_2 = 0; field_2 < field_num; field_2++) {
            int32_t mem_field_offset = (field_2 * fw_field_num + fw_field_1) * embedding_segment_size;
#pragma unroll
            for (int n = 0; n < segment_size_div_blockdimx; ++n) {
                T reg = T(0);
                int32_t rd_offset = wid * embedding_size * field_num + field_2 * embedding_size
                + embedding_segment_start + n * blockDim.x + lane_id;
                T *wr_ptr = smem_cross_mean_sum + mem_field_offset + n * blockDim.x + lane_id;

                if (embedding_segment_start + n * blockDim.x + lane_id < embedding_size) {
                    reg = weight_tensor[rd_offset];
                    atomicAdd(wr_ptr, reg);
                }
            }
        }

        int32_t mem_field_offset = fw_field_1 * embedding_segment_size;
#pragma unroll
        for (int n = 0; n < segment_size_div_blockdimx; ++n) {
            T reg = T(0);
            T square = T(0);
            int32_t rd_offset = wid * embedding_size * field_num + field_1 * embedding_size + embedding_segment_start
            + n * blockDim.x + lane_id;
            T *wr_ptr = smem_cross_square_sum + mem_field_offset + n * blockDim.x + lane_id;

            if (embedding_segment_start + n * blockDim.x + lane_id < embedding_size) {
                reg = weight_tensor[rd_offset];
                square = reg * reg;
                atomicAdd(wr_ptr, square);
            }
        }
    }
    __syncthreads();

    if (warp_id == 0 && lane_id == 0) {
        int32_t cnt = 0;
        for (int32_t i = 0; i < fw_field_num; i++) {
            if (smem_fw_field_map[i] >= 0) {
                smem_fw_map_idx[cnt++] = i;
            }
        }
        smem_fw_map_idx[fw_field_num] = cnt;
    }
    __syncthreads();
}

template <typename T>
__device__ void ComputeCommonSamplePartOutput(
int32_t embedding_size, int32_t embedding_segment_size, int32_t field_num, int32_t fw_field_num,
T* smem_cross_mean_sum, T* smem_cross_square_sum, int32_t* smem_fw_field_map, int32_t* smem_fw_map_idx,
T* output) {
    int32_t warp_id = threadIdx.y;
    int32_t lane_id = threadIdx.x;
    int32_t segment_size_div_blockdimx = embedding_segment_size / blockDim.x;
    int32_t embedding_segment_start = blockIdx.y * embedding_segment_size;

    int32_t* smem_sample_fw_field_map = smem_fw_field_map;
    int32_t* smem_common_fw_field_map = smem_fw_field_map + fw_field_num + 1;
    int32_t* smem_sample_fw_map_idx = smem_fw_map_idx;
    int32_t* smem_common_fw_map_idx = smem_fw_map_idx + fw_field_num + 1;
    int32_t common_vaild_fw_field = smem_common_fw_map_idx[fw_field_num];
    int32_t sample_vaild_fw_field = smem_sample_fw_map_idx[fw_field_num];
    //    if (blockIdx.x == 0 && blockIdx.y == 0 && warp_id == 0 && lane_id == 0) {
    //        printf("[FIPNN] common_vaild_fw_field=%d\n", common_vaild_fw_field);
    //        printf("[FIPNN] sample_vaild_fw_field=%d\n", sample_vaild_fw_field);
    //    }

    for (int32_t fw_field_1_idx = warp_id; fw_field_1_idx < sample_vaild_fw_field; fw_field_1_idx += blockDim.y) {
        int32_t fw_field_1 = smem_sample_fw_map_idx[fw_field_1_idx];
        int32_t field_1 = smem_sample_fw_field_map[fw_field_1];
        int32_t fw_iter = fw_field_1 * (fw_field_1 + 1) / 2;

        int32_t index_1 = (field_1 * fw_field_num + fw_field_1) * embedding_segment_size;
        T output_value = T(0);
#pragma unroll
        for (int n = 0; n < segment_size_div_blockdimx; ++n) {
            T cross_mean_sum = T(0);
            T cross_square_sum = T(0);
            if (embedding_segment_start + n * blockDim.x + lane_id < embedding_size) {
                cross_mean_sum = smem_cross_mean_sum[index_1 + n * blockDim.x + lane_id];
                cross_square_sum = smem_cross_square_sum[fw_field_1 * embedding_segment_size + n * blockDim.x + lane_id];
            }
            output_value += T(0.5) * (cross_mean_sum * cross_mean_sum - cross_square_sum);
        }
        output_value = __reduce_sum_across_warp(output_value);
        // store here
        if (lane_id == 0) {
            atomicAdd(output + fw_iter + fw_field_1, output_value);
        }
        __syncwarp(0xFFFFFFFF);

        for (int32_t fw_field_2_idx = 0; fw_field_2_idx < fw_field_1_idx; fw_field_2_idx++) {
            int32_t fw_field_2 = smem_fw_map_idx[fw_field_2_idx];
            int32_t field_2 = smem_fw_field_map[fw_field_2];
            int32_t index_1 = (field_1 * fw_field_num + fw_field_2) * embedding_segment_size;
            int32_t index_2 = (field_2 * fw_field_num + fw_field_1) * embedding_segment_size;

            T output_value = T(0);
#pragma unroll
            for (int n = 0; n < segment_size_div_blockdimx; ++n) {
                T mean_index_1_sum = T(0);
                T mean_index_2_sum = T(0);
                if (embedding_segment_start + n * blockDim.x + lane_id < embedding_size) {
                    mean_index_1_sum = smem_cross_mean_sum[index_1 + n * blockDim.x + lane_id];
                    mean_index_2_sum = smem_cross_mean_sum[index_2 + n * blockDim.x + lane_id];
                }
                output_value += mean_index_1_sum * mean_index_2_sum;
            }
            output_value = __reduce_sum_across_warp(output_value);
            // store here
            if (lane_id == 0) {
                atomicAdd(output + fw_iter + fw_field_2, output_value);
            }
            __syncwarp(0xFFFFFFFF);
        }

    }

    for (int32_t fw_field_1_idx = warp_id; fw_field_1_idx < common_vaild_fw_field; fw_field_1_idx += blockDim.y) {
        int32_t fw_field_1 = smem_common_fw_map_idx[fw_field_1_idx];
        int32_t field_1 = smem_common_fw_field_map[fw_field_1];

        for (int32_t fw_field_2_idx = 0; fw_field_2_idx < sample_vaild_fw_field; fw_field_2_idx++) {
            int32_t fw_field_2 = smem_sample_fw_map_idx[fw_field_2_idx];
            int32_t field_2 = smem_sample_fw_field_map[fw_field_2];
            int32_t index_1 = (field_1 * fw_field_num + fw_field_2) * embedding_segment_size;
            int32_t index_2 = (field_2 * fw_field_num + fw_field_1) * embedding_segment_size;
            int32_t fw_iter = fw_field_2 * (fw_field_2 + 1) / 2;
            if (fw_field_2 < fw_field_1) {
                fw_iter = fw_field_1 * (fw_field_1 + 1) / 2;
                fw_field_1 = fw_field_2;
            }

            T output_value = T(0);
#pragma unroll
            for (int n = 0; n < segment_size_div_blockdimx; ++n) {
                T mean_index_1_sum = T(0);
                T mean_index_2_sum = T(0);
                if (embedding_segment_start + n * blockDim.x + lane_id < embedding_size) {
                    mean_index_1_sum = smem_cross_mean_sum[index_1 + n * blockDim.x + lane_id];
                    mean_index_2_sum = smem_cross_mean_sum[index_2 + n * blockDim.x + lane_id];
                }
                output_value += mean_index_1_sum * mean_index_2_sum;
            }
            output_value = __reduce_sum_across_warp(output_value);
            // store here
            if (lane_id == 0) {
                atomicAdd(output + fw_iter + fw_field_1, output_value);
            }
            __syncwarp(0xFFFFFFFF);
        }

    }
}

template <typename T>
__device__ void ProcessSamplePartShare(
int32_t* smem_fw_field_map, int32_t* smem_fw_map_idx, T* smem_cross_mean_sum,
T* smem_cross_square_sum, T* gmem_common_cross_mean_sum, T* gmem_common_cross_square_sum,
int32_t* gmem_common_field_map, int32_t embedding_size, int32_t embedding_segment_size,
int32_t field_num, int32_t fw_field_num, int32_t this_sample_feature_num,
int32_t this_sample_feature_start_addr, const T* weight_tensor, const int32_t* field_tensor) {
    int32_t warp_id = threadIdx.y;
    int32_t lane_id = threadIdx.x;
    int32_t tid = threadIdx.x + threadIdx.y * blockDim.x;
    int32_t total_thread = blockDim.x * blockDim.y;
    int32_t segment_size_div_blockdimx = embedding_segment_size / blockDim.x;
    int32_t embedding_segment_start = blockIdx.y * embedding_segment_size;

    for (int32_t i = tid; i < fw_field_num; i += total_thread) {
        smem_fw_field_map[i] = gmem_common_field_map[i];
    }

    for (int32_t i = warp_id; i < (field_num + 1) * fw_field_num; i += blockDim.y) {
#pragma unroll
        for (int n = 0; n < segment_size_div_blockdimx; ++n) {
            int32_t embedding_offset = embedding_segment_start + n * blockDim.x + lane_id;
            if (embedding_offset < embedding_size) {
                smem_cross_mean_sum[i * embedding_segment_size + n * blockDim.x + lane_id] =
                gmem_common_cross_mean_sum[i * embedding_size + embedding_offset];
            }
        }
    }
    __syncthreads();

    // sample feature phase
    int32_t sample_start_row = warp_id + this_sample_feature_start_addr;
    int32_t sample_end_row = this_sample_feature_num + this_sample_feature_start_addr;
    for (int32_t wid = sample_start_row; wid < sample_end_row; wid += blockDim.y) {
        int32_t field_1 = field_tensor[wid * 2] - 1;
        int32_t fw_field_1 = field_tensor[wid * 2 + 1] - 1;
        if (fw_field_1 < 0 || fw_field_1 >= fw_field_num || field_1 < 0 || field_1 >= field_num)
            continue;
        if (lane_id == 0) smem_fw_field_map[fw_field_1] = field_1;

#pragma unroll
        for (int32_t field_2 = 0; field_2 < field_num; field_2++) {
            int32_t mem_field_offset = (field_2 * fw_field_num + fw_field_1) * embedding_segment_size;
#pragma unroll
            for (int n = 0; n < segment_size_div_blockdimx; ++n) {
                T reg = T(0);
                int32_t rd_offset = wid * embedding_size * field_num + field_2 * embedding_size
                + embedding_segment_start + n * blockDim.x + lane_id;
                T *wr_ptr = smem_cross_mean_sum + mem_field_offset + n * blockDim.x + lane_id;

                if (embedding_segment_start + n * blockDim.x + lane_id < embedding_size) {
                    reg = weight_tensor[rd_offset];
                    atomicAdd(wr_ptr, reg);
                }
            }
        }

        int32_t mem_field_offset = fw_field_1 * embedding_segment_size;
#pragma unroll
        for (int n = 0; n < segment_size_div_blockdimx; ++n) {
            T reg = T(0);
            T square = T(0);
            int32_t rd_offset = wid * embedding_size * field_num + field_1 * embedding_size + embedding_segment_start
            + n * blockDim.x + lane_id;
            T *wr_ptr = smem_cross_square_sum + mem_field_offset + n * blockDim.x + lane_id;

            if (embedding_segment_start + n * blockDim.x + lane_id < embedding_size) {
                reg = weight_tensor[rd_offset];
                square = reg * reg;
                atomicAdd(wr_ptr, square);
            }
        }
    }
    __syncthreads();

    if (warp_id == 0 && lane_id == 0) {
        int32_t cnt = 0;
        for (int32_t i = 0; i < fw_field_num; i++) {
            if (smem_fw_field_map[i] >= 0) {
                smem_fw_map_idx[cnt++] = i;
            }
        }
        smem_fw_map_idx[fw_field_num] = cnt;
    }
    __syncthreads();
}

template <typename T = float>
__device__ void ProcessOutputShare(
int32_t embedding_size, int32_t embedding_segment_size, int32_t field_num, int32_t fw_field_num,
T* smem_cross_mean_sum, T* smem_cross_square_sum, int32_t* smem_fw_field_map, int32_t* smem_fw_map_idx,
T* output) {
    int32_t warp_id = threadIdx.y;
    int32_t lane_id = threadIdx.x;
    int32_t segment_size_div_blockdimx = embedding_segment_size / blockDim.x;
    int32_t embedding_segment_start = blockIdx.y * embedding_segment_size;
    int32_t total_vaild_fw_field = smem_fw_map_idx[fw_field_num];

    for (int32_t fw_field_1_idx = warp_id; fw_field_1_idx < total_vaild_fw_field;
    fw_field_1_idx += blockDim.y) {
        int32_t fw_field_1 = smem_fw_map_idx[fw_field_1_idx];
        int32_t field_1 = smem_fw_field_map[fw_field_1];
        int32_t fw_iter = fw_field_1 * (fw_field_1 + 1) / 2;

        for (int32_t fw_field_2_idx = 0; fw_field_2_idx < fw_field_1_idx; fw_field_2_idx++) {
            int32_t fw_field_2 = smem_fw_map_idx[fw_field_2_idx];
            int32_t field_2 = smem_fw_field_map[fw_field_2];
            int32_t index_1 = (field_1 * fw_field_num + fw_field_2) * embedding_segment_size;
            int32_t index_2 = (field_2 * fw_field_num + fw_field_1) * embedding_segment_size;

            T output_value = T(0);
#pragma unroll
            for (int n = 0; n < segment_size_div_blockdimx; ++n) {
                T mean_index_1_sum = T(0);
                T mean_index_2_sum = T(0);
                if (embedding_segment_start + n * blockDim.x + lane_id < embedding_size) {
                    mean_index_1_sum = smem_cross_mean_sum[index_1 + n * blockDim.x + lane_id];
                    mean_index_2_sum = smem_cross_mean_sum[index_2 + n * blockDim.x + lane_id];
                }
                output_value += mean_index_1_sum * mean_index_2_sum;
            }
            output_value = __reduce_sum_across_warp(output_value);
            // store here
            if (lane_id == 0) {
                atomicAdd(output + fw_iter + fw_field_2, output_value);
            }
            __syncwarp(0xFFFFFFFF);
        }

        int32_t index_1 = (field_1 * fw_field_num + fw_field_1) * embedding_segment_size;
        T output_value = T(0);
#pragma unroll
        for (int n = 0; n < segment_size_div_blockdimx; ++n) {
            T cross_mean_sum = T(0);
            T cross_square_sum = T(0);
            if (embedding_segment_start + n * blockDim.x + lane_id < embedding_size) {
                cross_mean_sum = smem_cross_mean_sum[index_1 + n * blockDim.x + lane_id];
                cross_square_sum = smem_cross_square_sum[fw_field_1 * embedding_segment_size + n * blockDim.x + lane_id];
            }
            output_value += T(0.5) * (cross_mean_sum * cross_mean_sum - cross_square_sum);
        }
        output_value = __reduce_sum_across_warp(output_value);
        // store here
        if (lane_id == 0) {
            atomicAdd(output + fw_iter + fw_field_1, output_value);
        }
        __syncwarp(0xFFFFFFFF);
    }
}

template <typename T = float>
__global__ void SparseFIPNNGpuShare(
int32_t embedding_size, int32_t embedding_segment_size, int32_t field_num,
int32_t fw_field_num, int32_t* sample_feature_start_addr,
T* gmem_common_cross_mean_sum, T* gmem_common_cross_square_sum,
int32_t* gmem_common_field_map, const T* weight_tensor,
const int32_t* field_tensor, T* output_tensor,
T* workspace) {
    extern __shared__ float smem_pool[];
    int32_t batch_id = blockIdx.x;
    int32_t fw_weight_size = (fw_field_num + 1) * fw_field_num / 2;
    int32_t* smem_fw_field_map = reinterpret_cast<int32_t*>(smem_pool);
    int32_t* smem_fw_map_idx = smem_fw_field_map + 2 * (fw_field_num + 1);

    T* smem_cross_mean_sum = reinterpret_cast<T*>(smem_fw_map_idx + 2 * (fw_field_num + 1));
    T* smem_cross_square_sum = smem_cross_mean_sum + embedding_segment_size * field_num * fw_field_num;

    T* gmem_output = output_tensor + batch_id * fw_weight_size;
    int32_t this_sample_feature_start_addr = sample_feature_start_addr[batch_id];
    int32_t this_sample_feature_end_addr = sample_feature_start_addr[batch_id + 1];
    int32_t this_sample_feature_num = this_sample_feature_end_addr - this_sample_feature_start_addr;

    ProcessSamplePartShare<T>(
    smem_fw_field_map, smem_fw_map_idx, smem_cross_mean_sum, smem_cross_square_sum,
    gmem_common_cross_mean_sum, gmem_common_cross_square_sum, gmem_common_field_map,
    embedding_size, embedding_segment_size, field_num, fw_field_num, this_sample_feature_num,
    this_sample_feature_start_addr, weight_tensor, field_tensor);
    ProcessOutputShare<T>(
    embedding_size, embedding_segment_size, field_num, fw_field_num, smem_cross_mean_sum,
    smem_cross_square_sum, smem_fw_field_map, smem_fw_map_idx, gmem_output);
}

template <typename T = float>
__global__ void SparseFIPNNGpuShareV2(
    const T* multi_weight_tensor, const int32_t* multi_field_tensor, void* workspace,
    const int32_t* sample_count_prefix_sum_vec, const int32_t* batch_size_prefix_sum_vec, const int32_t* batch_size_vec,
    const uint32_t common_cross_start_offset, const uint32_t fw_field_map_start_offset,
    const uint32_t common_output_start_offset, const int32_t embedding_size,
    const int32_t embedding_segment_size, const int32_t field_num,
    const int32_t fw_field_num, const int32_t fw_weight_size,
    T* multi_output_tensor) {

    const int32_t batch_size = batch_size_vec[blockIdx.z];
    if (blockIdx.x < batch_size) {
        const uint32_t data_align_count = DATA_ALIGN_BYTE_COUNT / sizeof(T);
        uint32_t common_cross_offset = common_cross_start_offset + blockIdx.z * ((field_num + 1) * fw_field_num * embedding_size + data_align_count);
        common_cross_offset = ALIGN_UP(common_cross_offset, data_align_count);
        uint32_t common_output_offset = common_output_start_offset + blockIdx.z * (fw_weight_size + data_align_count);
        common_output_offset = ALIGN_UP(common_output_offset, data_align_count);
        uint32_t bound_data_offset = uint32_t(batch_size_prefix_sum_vec[blockIdx.z]) + blockIdx.z * DATA_ALIGN_INT32_COUNT;
        bound_data_offset = ALIGN_UP(bound_data_offset, DATA_ALIGN_INT32_COUNT);
        uint32_t fw_field_map_offset = fw_field_map_start_offset + blockIdx.z * (2 * (fw_field_num + 1) + DATA_ALIGN_INT32_COUNT);
        fw_field_map_offset = ALIGN_UP(fw_field_map_offset, DATA_ALIGN_INT32_COUNT);

        const T* weight_tensor = multi_weight_tensor + sample_count_prefix_sum_vec[blockIdx.z] * field_num * embedding_size;
        const int32_t* field_tensor = multi_field_tensor + sample_count_prefix_sum_vec[blockIdx.z] * field_num;
        T* gmem_common_cross_mean_sum = static_cast<T*>(workspace) + common_cross_offset;
        T* gmem_common_cross_square_sum = gmem_common_cross_mean_sum + field_num * fw_field_num + embedding_size;
        T* common_output = static_cast<T*>(workspace) + common_output_offset;
        T* output_tensor = multi_output_tensor + batch_size_prefix_sum_vec[blockIdx.z] * fw_weight_size;
        int32_t* sample_feature_start_addr = static_cast<int32_t*>(workspace) + bound_data_offset;
        int32_t* gmem_common_field_map = static_cast<int32_t*>(workspace) + fw_field_map_offset;

        extern __shared__ float smem_pool[];
        int32_t* smem_fw_field_map = reinterpret_cast<int32_t*>(smem_pool);
        int32_t* smem_fw_map_idx = smem_fw_field_map + 2 * (fw_field_num + 1);
        T* smem_cross_mean_sum = reinterpret_cast<T*>(smem_fw_map_idx + 2 * (fw_field_num + 1));
        T* smem_cross_square_sum = smem_cross_mean_sum + embedding_segment_size * field_num * fw_field_num;

        int32_t batch_id = blockIdx.x;
        T* gmem_output = output_tensor + batch_id * fw_weight_size;
        int32_t this_sample_feature_start_addr = sample_feature_start_addr[batch_id];
        int32_t this_sample_feature_end_addr = sample_feature_start_addr[batch_id + 1];
        int32_t this_sample_feature_num = this_sample_feature_end_addr - this_sample_feature_start_addr;

        int32_t tid = threadIdx.x + threadIdx.y * blockDim.x;
        int32_t total_thread = blockDim.x * blockDim.y;
        if (blockIdx.y == 0) {
            for (int32_t i = tid; i < fw_weight_size; i += total_thread) {
//                gmem_output[i] += common_output[i];
                atomicAdd(gmem_output + i, common_output[i]);
            }
        }

        AccumulateSamplePart<T>(
            smem_fw_field_map, smem_fw_map_idx, smem_cross_mean_sum, smem_cross_square_sum,
            gmem_common_cross_mean_sum, gmem_common_cross_square_sum, gmem_common_field_map,
            embedding_size, embedding_segment_size, field_num, fw_field_num, this_sample_feature_num,
            this_sample_feature_start_addr, weight_tensor, field_tensor);
        ComputeCommonSamplePartOutput<T>(
            embedding_size, embedding_segment_size, field_num, fw_field_num,
            smem_cross_mean_sum, smem_cross_square_sum, smem_fw_field_map, smem_fw_map_idx, gmem_output);
    }
}

template <typename T = float, int32_t warp_num = 32>
__global__ void ProcessCommonPart(int32_t embedding_size, int32_t field_num, int32_t fw_field_num,
                                  int32_t common_fw_cross_size, int32_t output_size,
                                  int32_t* sample_feature_start_addr, const T* weight_tensor,
                                  const int32_t* field_tensor, T* gmem_fw_cross_mean_sum,
                                  T* gmem_fw_cross_square_sum, int32_t* gmem_fw_field_map,
                                  T* output) {
    int32_t warp_id = threadIdx.y;
    int32_t lane_id = threadIdx.x;

    int32_t global_warp_id = blockIdx.x * warp_num + warp_id;
    int32_t total_global_warp_num = gridDim.x * warp_num;

    int32_t tid = threadIdx.x + threadIdx.y * blockDim.x;
    int32_t total_thread = warp_num * 32;
    int32_t common_feature_num = sample_feature_start_addr[0];

    for (int32_t i = tid; i < common_fw_cross_size; i += total_thread) {
        gmem_fw_cross_mean_sum[i] = 0;
        if (i < fw_field_num) {
            gmem_fw_field_map[i] = -1;
        }
    }
    for (int32_t i = tid; i < output_size; i += total_thread) {
        output[i] = 0;
    }
    //    __syncthreads();

    for (int32_t wid = global_warp_id; wid < common_feature_num; wid += total_global_warp_num) {
        int32_t field_1 = field_tensor[wid * 2] - 1;
        int32_t fw_field_1 = field_tensor[wid * 2 + 1] - 1;

        if (fw_field_1 < 0 || fw_field_1 >= fw_field_num || field_1 < 0 || field_1 >= field_num)
            continue;

        if (lane_id == 0) gmem_fw_field_map[fw_field_1] = field_1;

#pragma unroll
        for (int32_t field_2 = 0; field_2 < field_num; field_2++) {
            int32_t mem_field_offset = (field_2 * fw_field_num + fw_field_1) * embedding_size;
#pragma unroll
            for (int32_t n = 0; n < (embedding_size + 31) / 32 * 32; n += 32) {
                if (n + lane_id < embedding_size) {
                    T reg = weight_tensor[wid * embedding_size * field_num +
                                          field_2 * embedding_size + n + lane_id];
                    atomicAdd(gmem_fw_cross_mean_sum + mem_field_offset + n + lane_id, reg);
                }
            }
        }

        int32_t mem_field_offset = fw_field_1 * embedding_size;
#pragma unroll
        for (int32_t n = 0; n < (embedding_size + 31) / 32 * 32; n += 32) {
            if (n + lane_id < embedding_size) {
                T reg = weight_tensor[wid * embedding_size * field_num + field_1 * embedding_size +
                                      n + lane_id];
                T square = reg * reg;
                atomicAdd(gmem_fw_cross_square_sum + mem_field_offset + n + lane_id, square);
            }
        }
    }
}

template <typename T = float>
__global__ void BroadcastCommonPart(int32_t batch, int32_t embedding_size, int32_t field_num,
                                    int32_t fw_field_num, T* gmem_fw_cross_mean_sum,
                                    T* gmem_fw_cross_square_sum, T* output) {

    int32_t lane_id = threadIdx.x;
    int32_t fw_field_id = blockIdx.x % fw_field_num;
    int32_t tid = lane_id + fw_field_id * embedding_size;
    int32_t bid = blockIdx.x / fw_field_num;
    T Reg_square = gmem_fw_cross_square_sum[tid];
    output[bid * (embedding_size * (field_num + 1) * fw_field_num) +
    embedding_size * field_num * fw_field_num + tid] = Reg_square;

    T Reg_mean_0 = gmem_fw_cross_mean_sum[tid];
    T Reg_mean_1 = gmem_fw_cross_mean_sum[embedding_size * fw_field_num + tid];
    output[bid * (embedding_size * (field_num + 1) * fw_field_num) + tid] = Reg_mean_0;
    output[bid * (embedding_size * (field_num + 1) * fw_field_num) + embedding_size * fw_field_num +
    tid] = Reg_mean_1;
}

template <typename T = float, int32_t warp_num = 32>
__device__ void ProcessSamplePart(
int32_t* smem_fw_field_map, int32_t* smem_fw_map_idx, T* gmem_cross_mean_sum,
T* gmem_cross_square_sum, T* gmem_common_cross_mean_sum, T* gmem_common_cross_square_sum,
int32_t* gmem_common_field_map, int32_t weight_size, int32_t field_num, int32_t fw_field_num,
int32_t this_sample_feature_num, int32_t this_sample_feature_start_addr, const T* weight_tensor,
const int32_t* field_tensor, T* smem_output, int32_t shared_mem_elements) {

    int32_t warp_id = threadIdx.y;
    int32_t lane_id = threadIdx.x;

    int32_t tid = threadIdx.x + threadIdx.y * blockDim.x;
    int32_t total_thread = warp_num * 32;

    for (int32_t i = 0; i < fw_field_num; i += warp_num * 32) {
        if (i + tid < fw_field_num) {
            smem_fw_field_map[i + tid] = gmem_common_field_map[i + tid];
        }
    }

    __syncthreads();

    // sample feature phase
    int32_t sample_start_row = warp_id + this_sample_feature_start_addr;
    int32_t sample_end_row = this_sample_feature_num + this_sample_feature_start_addr;
    for (int32_t wid = sample_start_row; wid < sample_end_row; wid += warp_num) {
        int32_t field_1 = field_tensor[wid * 2] - 1;
        int32_t fw_field_1 = field_tensor[wid * 2 + 1] - 1;
        if (fw_field_1 < 0 || fw_field_1 >= fw_field_num || field_1 < 0 || field_1 >= field_num)
            continue;

        if (lane_id == 0) smem_fw_field_map[fw_field_1] = field_1;

#pragma unroll
        for (int32_t field_2 = 0; field_2 < field_num; field_2++) {
            int32_t mem_field_offset = (field_2 * fw_field_num + fw_field_1) * weight_size;
#pragma unroll
            for (int32_t n = 0; n < (weight_size + 31) / 32 * 32; n += 32) {
                if (n + lane_id < weight_size) {
                    T reg = weight_tensor[wid * weight_size * field_num + field_2 * weight_size +
                                          n + lane_id];
                    atomicAdd(gmem_cross_mean_sum + mem_field_offset + n + lane_id, reg);
                }
            }
        }
        int32_t mem_field_offset = fw_field_1 * weight_size;
#pragma unroll
        for (int32_t n = 0; n < (weight_size + 31) / 32 * 32; n += 32) {
            if (n + lane_id < weight_size) {
                T reg = weight_tensor[wid * weight_size * field_num + field_1 * weight_size + n +
                                      lane_id];
                T square = reg * reg;
                atomicAdd(gmem_cross_square_sum + mem_field_offset + n + lane_id, square);
            }
        }
    }

    __syncthreads();
    if (warp_id == 0 && lane_id == 0) {
        int32_t cnt = 0;
        for (int32_t i = 0; i < fw_field_num; i++) {
            if (smem_fw_field_map[i] >= 0) {
                smem_fw_map_idx[cnt++] = i;
            }
        }
        smem_fw_map_idx[fw_field_num] = cnt;
    }
    __syncthreads();
}

template <typename T = float, int32_t warp_num = 32>
__device__ void ProcessOutput(int32_t weight_size, int32_t field_num, int32_t fw_field_num,
                              T* mem_cross_mean_sum, T* mem_cross_square_sum,
                              int32_t* smem_fw_field_map, int32_t* smem_fw_map_idx,
                              T* output_smem) {

    int32_t warp_id = threadIdx.y;
    int32_t lane_id = threadIdx.x;
    int32_t weight_size_pad = (weight_size + 31) / 32 * 32;

    int32_t total_vaild_fw_field = smem_fw_map_idx[fw_field_num];

    for (int32_t fw_field_1_idx = warp_id; fw_field_1_idx < total_vaild_fw_field;
    fw_field_1_idx += warp_num) {
        int32_t fw_field_1 = smem_fw_map_idx[fw_field_1_idx];
        int32_t field_1 = smem_fw_field_map[fw_field_1];

        int32_t fw_iter = (2 + fw_field_1) * (fw_field_1 + 1) / 2 - (fw_field_1 + 1);

        T reg_cross_mean_sum_tmp[2][6] = { 0 };  // weight_size <= 192

        for (int32_t n = 0; n < weight_size_pad; n += 32) {
            if (n + lane_id < weight_size) {
                reg_cross_mean_sum_tmp[0][n / 32] =
                mem_cross_mean_sum[(0 * fw_field_num + fw_field_1) * weight_size + n + lane_id];
                reg_cross_mean_sum_tmp[1][n / 32] =
                mem_cross_mean_sum[(1 * fw_field_num + fw_field_1) * weight_size + n + lane_id];
            }
        }

        for (int32_t fw_field_2_idx = 0; fw_field_2_idx < fw_field_1_idx; fw_field_2_idx++) {
            int32_t fw_field_2 = smem_fw_map_idx[fw_field_2_idx];
            int32_t field_2 = smem_fw_field_map[fw_field_2];

            int32_t index_1 = (field_1 * fw_field_num + fw_field_2) * weight_size;
            int32_t index_2 = field_2 * weight_size;

            T output_value = T(0);
            for (int32_t n = 0; n < (weight_size + 31) / 32 * 32; n += 32) {
                T reg_index_1 = T(0);
                T reg_index_2 = T(0);

                if (n + lane_id < weight_size) {
                    reg_index_1 = mem_cross_mean_sum[index_1 + n + lane_id];
                    if (field_2 == 0)
                        reg_index_2 = reg_cross_mean_sum_tmp[0][n / 32];
                    else
                        reg_index_2 = reg_cross_mean_sum_tmp[1][n / 32];

                    // reg_index_2 = mem_cross_mean_sum[index_2 + n + lane_id];
                }
                output_value += reg_index_1 * reg_index_2;
            }
            output_value = __reduce_sum_across_warp(output_value);
            // store here
            if (lane_id == 0) {
                output_smem[fw_iter + fw_field_2] = output_value;
            }
            __syncwarp(0xFFFFFFFF);
        }
        T output_value = T(0);
        for (int32_t n = 0; n < (weight_size + 31) / 32 * 32; n += 32) {
            int32_t index_1 = (field_1 * fw_field_num + fw_field_1) * weight_size;
            T reg_mean = T(0);
            T reg_square = T(0);

            if (n + lane_id < weight_size) {
                reg_mean = mem_cross_mean_sum[index_1 + n + lane_id];
                reg_square = mem_cross_square_sum[fw_field_1 * weight_size + n + lane_id];
            }
            output_value += T(0.5) * (reg_mean * reg_mean - reg_square);
        }
        output_value = __reduce_sum_across_warp(output_value);
        // store here
        if (lane_id == 0) output_smem[fw_iter + fw_field_1] = output_value;
        __syncwarp(0xFFFFFFFF);
    }
}

template <typename T = float, int32_t warp_num = 32>
__global__ void SparseFIPNNGpu(int32_t weight_size, int32_t field_num, int32_t fw_field_num,
                               int32_t* sample_feature_start_addr,
                               T* gmem_common_cross_mean_sum, T* gmem_common_cross_square_sum,
                               int32_t* gmem_common_field_map, const T* weight_tensor,
                               const int32_t* field_tensor, T* output_tensor,
                               T* workspace  // for mean_sum and square_sum
                               ) {
    int32_t batch_id = blockIdx.x;
    int32_t embedding_size = fw_field_num * (fw_field_num + 1) / 2;

    extern __shared__ float smem_pool[];

    int32_t* smem_fw_field_map = reinterpret_cast<int*>(smem_pool);
    int32_t* smem_fw_map_idx = smem_fw_field_map + fw_field_num;
    T* gmem_output = output_tensor + batch_id * embedding_size;

    // Use global memory in case of lacking atomicAdd float in shared mem
    T* mem_cross_mean_sum = workspace + batch_id * (weight_size * (field_num + 1) * fw_field_num);
    T* mem_cross_square_sum = mem_cross_mean_sum + weight_size * field_num * fw_field_num;

    int this_sample_feature_start_addr = sample_feature_start_addr[batch_id];
    int this_sample_feature_end_addr = sample_feature_start_addr[batch_id + 1];
    int this_sample_feature_num = this_sample_feature_end_addr - this_sample_feature_start_addr;

    ProcessSamplePart<T, warp_num>(
    smem_fw_field_map, smem_fw_map_idx, mem_cross_mean_sum, mem_cross_square_sum,
    gmem_common_cross_mean_sum, gmem_common_cross_square_sum, gmem_common_field_map,
    weight_size, field_num, fw_field_num, this_sample_feature_num,
    this_sample_feature_start_addr, weight_tensor, field_tensor, gmem_output, embedding_size);

    ProcessOutput<T, warp_num>(weight_size, field_num, fw_field_num, mem_cross_mean_sum,
                               mem_cross_square_sum, smem_fw_field_map, smem_fw_map_idx,
                               gmem_output);
}

namespace functor {
    template <typename T>
    int32_t ComputeSparseFipnnSharedMemoryMultiExample(
    hipStream_t stream, const void* const* input, T* output, void* workspace,
    const int32_t total_batch_size, const int32_t example_count,
    const int32_t max_sample_count, const int32_t max_batch_size,
    const int32_t fw_field_num, const int32_t field_num, const int32_t embedding_size) {
        int32_t fw_weight_size = fw_field_num * (fw_field_num + 1) / 2;
        int32_t output_size = total_batch_size * fw_weight_size;

        int32_t max_share_mem_size = 65536;
        int32_t max_embedding_segment_size =
        (max_share_mem_size - 4 * (fw_field_num + 1) * sizeof(int32_t) - fw_weight_size * sizeof(T))
        / ((field_num + 1) * fw_field_num * sizeof(T));
        max_embedding_segment_size = max_embedding_segment_size / 32 * 32;
        int32_t embedding_segment_count = (embedding_size + max_embedding_segment_size - 1) / max_embedding_segment_size;
        int32_t embedding_segment_size = embedding_size / embedding_segment_count;
        embedding_segment_size = (embedding_segment_size + 31) / 32 * 32;
        int32_t share_mem_size = 4 * (fw_field_num + 1) * sizeof(int32_t) + fw_weight_size * sizeof(T)
        + embedding_segment_size * (field_num + 1) * fw_field_num * sizeof(T);

        const T* weight_tensor = static_cast<const T*>(input[0]);
        const int32_t* field_tensor = static_cast<const int32_t*>(input[1]);
        const int32_t* index_tensor = static_cast<const int32_t*>(input[2]);
        const int32_t* sample_count_prefix_sum_vec = static_cast<const int32_t*>(input[3]);
        const int32_t* sample_count_vec = static_cast<const int32_t*>(input[4]);
        const int32_t* batch_size_prefix_sum_vec = static_cast<const int32_t*>(input[5]);
        const int32_t* batch_size_vec = static_cast<const int32_t*>(input[6]);

        uint32_t data_size = (total_batch_size + example_count) * sizeof(int32_t) + example_count * DATA_ALIGN_BYTE_COUNT;
        uint32_t fw_field_map_start_offset = ALIGN_UP(data_size, DATA_ALIGN_BYTE_COUNT) / sizeof(int32_t);

        data_size += example_count * (2 * (fw_field_num + 1) * sizeof(int32_t) + DATA_ALIGN_BYTE_COUNT);
        uint32_t common_output_start_offset = ALIGN_UP(data_size, DATA_ALIGN_BYTE_COUNT) / sizeof(T);

        data_size += example_count * (fw_weight_size * sizeof(T) + DATA_ALIGN_BYTE_COUNT);
        uint32_t sample_cross_start_offset = ALIGN_UP(data_size, DATA_ALIGN_BYTE_COUNT) / sizeof(T);

        data_size += (total_batch_size * (field_num + 1) * fw_field_num * embedding_size * sizeof(T)
                  + example_count * DATA_ALIGN_BYTE_COUNT);
        uint32_t common_cross_start_offset = ALIGN_UP(data_size, DATA_ALIGN_BYTE_COUNT) / sizeof(T);

        hipMemsetAsync(output, 0, output_size * sizeof(T), stream);
        dim3 block_boundary(32, 32);
        dim3 grid_boundary(DIVUP(max_sample_count, 1024), example_count);
        if ((max_embedding_segment_size > 0) && (embedding_size >= 32)) {
            ComputeBatchBoundary<<<grid_boundary, block_boundary, 0, stream>>>(
                index_tensor, workspace, sample_count_prefix_sum_vec, sample_count_vec,
                batch_size_prefix_sum_vec, batch_size_vec);

            hipFuncSetAttribute(reinterpret_cast<const void*>(ComputeCommonPartOutput<T>), hipFuncAttributeMaxDynamicSharedMemorySize, share_mem_size);
            dim3 grid_common(embedding_segment_count, example_count);
            dim3 block_common(32, 32);
            int32_t common_fw_cross_size = (field_num + 1) * fw_field_num * embedding_segment_size;
            ComputeCommonPartOutput<T><<<grid_common, block_common, share_mem_size, stream>>>(
                weight_tensor, field_tensor, workspace, sample_count_prefix_sum_vec, batch_size_prefix_sum_vec,
                common_cross_start_offset, fw_field_map_start_offset, common_output_start_offset,
                embedding_size, embedding_segment_size, field_num, fw_field_num,
                common_fw_cross_size, fw_weight_size);

            hipFuncSetAttribute(reinterpret_cast<const void*>(SparseFIPNNGpuShareV2<T>), hipFuncAttributeMaxDynamicSharedMemorySize, share_mem_size);
            dim3 grid(max_batch_size, embedding_segment_count, example_count);
            dim3 block(32, 32);
            SparseFIPNNGpuShareV2<T><<<grid, block, share_mem_size, stream>>>(
                weight_tensor, field_tensor, workspace,
                sample_count_prefix_sum_vec, batch_size_prefix_sum_vec, batch_size_vec,
                common_cross_start_offset, fw_field_map_start_offset, common_output_start_offset,
                embedding_size, embedding_segment_size, field_num, fw_field_num, fw_weight_size,
                output);

        } else if (embedding_size <= 192) {
            printf("This Fipnn op will support such input parameters soon!\n");
        } else {
            printf("This Fipnn op is not support such input parameters!\n");
        }

        return 1;
    }

    template int32_t ComputeSparseFipnnSharedMemoryMultiExample(
    hipStream_t stream, const void* const* input, float* output, void* workspace,
    const int32_t total_batch_size, const int32_t example_count,
    const int32_t max_sample_count, const int32_t max_batch_size,
    int32_t fw_field_num, const int32_t field_num, const int32_t embedding_size);
    template int32_t ComputeSparseFipnnSharedMemoryMultiExample(
    hipStream_t stream, const void* const* input, half* output, void* workspace,
    const int32_t total_batch_size, const int32_t example_count,
    const int32_t max_sample_count, const int32_t max_batch_size,
    int32_t fw_field_num, const int32_t field_num, const int32_t embedding_size);

}  // namespace functor
}  // namespace sparse_fipnn_shared_multi
}  // namespace nvinfer1
