#include "hip/hip_runtime.h"
// Copyright 2020, Tencent Inc.
// All rights reserved.
//
// @author shaorunwang <shaorunwang@tencent.com>

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <thrust/extrema.h>

#include <iostream>

//#include "NvInfer.h"
//#include "sparse_fipnn_plugin.h"

#define CUDA_CHECK(condition)                                    \
    /* Code block avoids redefinition of hipError_t error */    \
    do {                                                         \
        hipError_t error = condition;                           \
        if (error != hipSuccess) {                              \
            std::cout << hipGetErrorString(error) << std::endl; \
        }                                                        \
    } while (0)

#define DIVUP(m, n) (((m) / (n)) + ((m) % (n) > 0))

#define CUDA_1D_KERNEL_LOOP(i, n) \
    for (int32_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x)

namespace nvinfer1 {

namespace sparse_fipnn {

template <typename T = float>
__device__ T __reduce_sum_across_warp(T val) {
    T rtn = val;
    __syncwarp(0xFFFFFFFF);
    for (int32_t i = 1; i < 32; i *= 2) {
        rtn += __shfl_xor_sync(0xFFFFFFFF, rtn, i);
    }
    return rtn;
}

template <typename T = float, int32_t warp_num = 32>
__global__ void ProcessCommonPart(int32_t embedding_size, int32_t field_num, int32_t fw_field_num,
                                  int32_t* sample_feature_start_addr, const T* weight_tensor,
                                  const int32_t* field_tensor, T* gmem_fw_cross_mean_sum,
                                  T* gmem_fw_cross_square_sum, int32_t* gmem_fw_field_map) {
    int32_t warp_id = threadIdx.y;
    int32_t lane_id = threadIdx.x;

    int32_t global_warp_id = blockIdx.x * warp_num + warp_id;
    int32_t total_global_warp_num = gridDim.x * warp_num;

    int32_t tid = threadIdx.x + threadIdx.y * blockDim.x;
    int32_t total_thread = warp_num * 32;
    int32_t common_feature_num = sample_feature_start_addr[0];

    for (int32_t wid = global_warp_id; wid < common_feature_num; wid += total_global_warp_num) {
        int32_t field_1 = field_tensor[wid * 2] - 1;
        int32_t fw_field_1 = field_tensor[wid * 2 + 1] - 1;

        if (fw_field_1 < 0 || fw_field_1 >= fw_field_num || field_1 < 0 || field_1 >= field_num)
            continue;

        if (lane_id == 0) gmem_fw_field_map[fw_field_1] = field_1;

#pragma unroll
        for (int32_t field_2 = 0; field_2 < field_num; field_2++) {
            int32_t mem_field_offset = (field_2 * fw_field_num + fw_field_1) * embedding_size;
#pragma unroll
            for (int32_t n = 0; n < (embedding_size + 31) / 32 * 32; n += 32) {
                if (n + lane_id < embedding_size) {
                    T reg = weight_tensor[wid * embedding_size * field_num +
                                          field_2 * embedding_size + n + lane_id];
                    atomicAdd(gmem_fw_cross_mean_sum + mem_field_offset + n + lane_id, reg);
                }
            }
        }

        int32_t mem_field_offset = fw_field_1 * embedding_size;
#pragma unroll
        for (int32_t n = 0; n < (embedding_size + 31) / 32 * 32; n += 32) {
            if (n + lane_id < embedding_size) {
                T reg = weight_tensor[wid * embedding_size * field_num + field_1 * embedding_size +
                                      n + lane_id];
                float square = reg * reg;
                atomicAdd(gmem_fw_cross_square_sum + mem_field_offset + n + lane_id, square);
            }
        }
    }
}

template <typename T = float>
__global__ void BroadcastCommonPart(int32_t batch, int32_t embedding_size, int32_t field_num,
                                    int32_t fw_field_num, T* gmem_fw_cross_mean_sum,
                                    T* gmem_fw_cross_square_sum, T* output) {

    int32_t lane_id = threadIdx.x;
    int32_t fw_field_id = blockIdx.x % fw_field_num;
    int32_t tid = lane_id + fw_field_id * embedding_size;
    int32_t bid = blockIdx.x / fw_field_num;
    T Reg_square = gmem_fw_cross_square_sum[tid];
    output[bid * (embedding_size * (field_num + 1) * fw_field_num) +
           embedding_size * field_num * fw_field_num + tid] = Reg_square;

    T Reg_mean_0 = gmem_fw_cross_mean_sum[tid];
    T Reg_mean_1 = gmem_fw_cross_mean_sum[embedding_size * fw_field_num + tid];
    output[bid * (embedding_size * (field_num + 1) * fw_field_num) + tid] = Reg_mean_0;
    output[bid * (embedding_size * (field_num + 1) * fw_field_num) + embedding_size * fw_field_num +
           tid] = Reg_mean_1;
}

template <typename T = float, int32_t warp_num = 32>
__device__ void ProcessSamplePart(
    int32_t* smem_fw_field_map, int32_t* smem_fw_map_idx, T* gmem_cross_mean_sum,
    T* gmem_cross_square_sum, T* gmem_common_cross_mean_sum, T* gmem_common_cross_square_sum,
    int32_t* gmem_common_field_map, int32_t weight_size, int32_t field_num, int32_t fw_field_num,
    int32_t this_sample_feature_num, int32_t this_sample_feature_start_addr, const T* weight_tensor,
    const int32_t* field_tensor, T* smem_output, int32_t shared_mem_elements) {

    int32_t warp_id = threadIdx.y;
    int32_t lane_id = threadIdx.x;

    int32_t tid = threadIdx.x + threadIdx.y * blockDim.x;
    int32_t total_thread = warp_num * 32;

    for (int32_t i = 0; i < fw_field_num; i += warp_num * 32) {
        if (i + tid < fw_field_num) {
            smem_fw_field_map[i + tid] = gmem_common_field_map[i + tid];
        }
    }

    __syncthreads();

    // sample feature phase
    int32_t sample_start_row = warp_id + this_sample_feature_start_addr;
    int32_t sample_end_row = this_sample_feature_num + this_sample_feature_start_addr;
    for (int32_t wid = sample_start_row; wid < sample_end_row; wid += warp_num) {
        int32_t field_1 = field_tensor[wid * 2] - 1;
        int32_t fw_field_1 = field_tensor[wid * 2 + 1] - 1;
        if (fw_field_1 < 0 || fw_field_1 >= fw_field_num || field_1 < 0 || field_1 >= field_num)
            continue;

        if (lane_id == 0) smem_fw_field_map[fw_field_1] = field_1;

#pragma unroll
        for (int32_t field_2 = 0; field_2 < field_num; field_2++) {
            int32_t mem_field_offset = (field_2 * fw_field_num + fw_field_1) * weight_size;
#pragma unroll
            for (int32_t n = 0; n < (weight_size + 31) / 32 * 32; n += 32) {
                if (n + lane_id < weight_size) {
                    T reg = weight_tensor[wid * weight_size * field_num + field_2 * weight_size +
                                          n + lane_id];
                    atomicAdd(gmem_cross_mean_sum + mem_field_offset + n + lane_id, reg);
                }
            }
        }
        int32_t mem_field_offset = fw_field_1 * weight_size;
#pragma unroll
        for (int32_t n = 0; n < (weight_size + 31) / 32 * 32; n += 32) {
            if (n + lane_id < weight_size) {
                T reg = weight_tensor[wid * weight_size * field_num + field_1 * weight_size + n +
                                      lane_id];
                float square = reg * reg;
                atomicAdd(gmem_cross_square_sum + mem_field_offset + n + lane_id, square);
            }
        }
    }

    __syncthreads();
    if (warp_id == 0 && lane_id == 0) {
        int32_t cnt = 0;
        for (int32_t i = 0; i < fw_field_num; i++) {
            if (smem_fw_field_map[i] >= 0) {
                smem_fw_map_idx[cnt++] = i;
            }
        }
        smem_fw_map_idx[fw_field_num] = cnt;
    }
    __syncthreads();
}

template <typename T = float, int32_t warp_num = 32>
__device__ void ProcessOutput(int32_t weight_size, int32_t field_num, int32_t fw_field_num,
                              T* mem_cross_mean_sum, T* mem_cross_square_sum,
                              int32_t* smem_fw_field_map, int32_t* smem_fw_map_idx,
                              T* output_smem) {

    int32_t warp_id = threadIdx.y;
    int32_t lane_id = threadIdx.x;
    int32_t weight_size_pad = (weight_size + 31) / 32 * 32;

    int32_t total_vaild_fw_field = smem_fw_map_idx[fw_field_num];

    for (int32_t fw_field_1_idx = warp_id; fw_field_1_idx < total_vaild_fw_field;
         fw_field_1_idx += warp_num) {
        int32_t fw_field_1 = smem_fw_map_idx[fw_field_1_idx];
        int32_t field_1 = smem_fw_field_map[fw_field_1];

        int32_t fw_iter = (2 + fw_field_1) * (fw_field_1 + 1) / 2 - (fw_field_1 + 1);

        T reg_cross_mean_sum_tmp[2][6] = { 0 };  // weight_size <= 192

        for (int32_t n = 0; n < weight_size_pad; n += 32) {
            if (n + lane_id < weight_size) {
                reg_cross_mean_sum_tmp[0][n / 32] =
                    mem_cross_mean_sum[(0 * fw_field_num + fw_field_1) * weight_size + n + lane_id];
                reg_cross_mean_sum_tmp[1][n / 32] =
                    mem_cross_mean_sum[(1 * fw_field_num + fw_field_1) * weight_size + n + lane_id];
            }
        }

        for (int32_t fw_field_2_idx = 0; fw_field_2_idx < fw_field_1_idx; fw_field_2_idx++) {
            int32_t fw_field_2 = smem_fw_map_idx[fw_field_2_idx];
            int32_t field_2 = smem_fw_field_map[fw_field_2];

            int32_t index_1 = (field_1 * fw_field_num + fw_field_2) * weight_size;
            int32_t index_2 = field_2 * weight_size;

            T output_value = T(0);
            for (int32_t n = 0; n < (weight_size + 31) / 32 * 32; n += 32) {
                T reg_index_1 = T(0);
                T reg_index_2 = T(0);

                if (n + lane_id < weight_size) {
                    reg_index_1 = mem_cross_mean_sum[index_1 + n + lane_id];
                    if (field_2 == 0)
                        reg_index_2 = reg_cross_mean_sum_tmp[0][n / 32];
                    else
                        reg_index_2 = reg_cross_mean_sum_tmp[1][n / 32];

                    // reg_index_2 = mem_cross_mean_sum[index_2 + n + lane_id];
                }
                output_value += reg_index_1 * reg_index_2;
            }
            output_value = __reduce_sum_across_warp(output_value);
            // store here
            if (lane_id == 0) {
                output_smem[fw_iter + fw_field_2] = output_value;
            }
            __syncwarp(0xFFFFFFFF);
        }
        T output_value = T(0);
        for (int32_t n = 0; n < (weight_size + 31) / 32 * 32; n += 32) {
            int32_t index_1 = (field_1 * fw_field_num + fw_field_1) * weight_size;
            T reg_mean = T(0);
            T reg_square = T(0);

            if (n + lane_id < weight_size) {
                reg_mean = mem_cross_mean_sum[index_1 + n + lane_id];
                reg_square = mem_cross_square_sum[fw_field_1 * weight_size + n + lane_id];
            }
            output_value += T(0.5) * (reg_mean * reg_mean - reg_square);
        }
        output_value = __reduce_sum_across_warp(output_value);
        // store here
        if (lane_id == 0) output_smem[fw_iter + fw_field_1] = output_value;
        __syncwarp(0xFFFFFFFF);
    }
}

__global__ void ComputeBatchBoundary(const int32_t* index_tensor, int32_t total_feature_num,
                                     int32_t batch_size, int32_t* sample_feature_start_addr,
                                     int32_t* sample_feature_end_addr) {
    int32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < total_feature_num) {
        int32_t idx = index_tensor[tid];
        // atomicMin(sample_feature_start_addr + idx, tid);
        // atomicMax(sample_feature_end_addr + idx, tid + 1);
        if (tid > 0) {
            int32_t pre_idx = index_tensor[tid - 1];
            for (int32_t i = idx; i > pre_idx; --i) {
                sample_feature_start_addr[i] = tid;
            }
        } else {
            int32_t first_idx = index_tensor[0];
            for (int32_t i = 0; i <= first_idx; ++i) {
                sample_feature_start_addr[i] = 0;
            }
            int32_t last_idx = index_tensor[total_feature_num - 1];
            for (int32_t i = batch_size - 1; i > last_idx; --i) {
                sample_feature_start_addr[i] = total_feature_num;
            }
            sample_feature_start_addr[batch_size] = total_feature_num;
        }
    }
}

template <typename T = float, int32_t warp_num = 32>
__global__ void SparseFIPNNGpu(int32_t weight_size, int32_t field_num, int32_t fw_field_num,
                               int32_t* sample_feature_start_addr, int32_t* sample_feature_end_addr,
                               T* gmem_common_cross_mean_sum, T* gmem_common_cross_square_sum,
                               int32_t* gmem_common_field_map, const T* weight_tensor,
                               const int32_t* field_tensor, T* output_tensor,
                               T* workspace  // for mean_sum and square_sum
)

{
    int32_t batch_id = blockIdx.x;
    int32_t embedding_size = fw_field_num * (fw_field_num + 1) / 2;

    extern __shared__ float smem_pool[];

    int32_t* smem_fw_field_map = reinterpret_cast<int*>(smem_pool);
    int32_t* smem_fw_map_idx = smem_fw_field_map + fw_field_num;
    T* gmem_output = output_tensor + batch_id * embedding_size;

    // Use global memory in case of lacking atomicAdd float in shared mem
    T* mem_cross_mean_sum = workspace + batch_id * (weight_size * (field_num + 1) * fw_field_num);
    T* mem_cross_square_sum = mem_cross_mean_sum + weight_size * field_num * fw_field_num;

    int this_sample_feature_start_addr = sample_feature_start_addr[batch_id];
    int this_sample_feature_end_addr = sample_feature_start_addr[batch_id + 1];
    int this_sample_feature_num = this_sample_feature_end_addr - this_sample_feature_start_addr;

    ProcessSamplePart<T, warp_num>(
        smem_fw_field_map, smem_fw_map_idx, mem_cross_mean_sum, mem_cross_square_sum,
        gmem_common_cross_mean_sum, gmem_common_cross_square_sum, gmem_common_field_map,
        weight_size, field_num, fw_field_num, this_sample_feature_num,
        this_sample_feature_start_addr, weight_tensor, field_tensor, gmem_output, embedding_size);

    ProcessOutput<T, warp_num>(weight_size, field_num, fw_field_num, mem_cross_mean_sum,
                               mem_cross_square_sum, smem_fw_field_map, smem_fw_map_idx,
                               gmem_output);
    // postprocess<T>(smem_output, output_tensor + batch_id * embedding_size, embedding_size);
}

namespace functor {

template <typename T>
int32_t ComputeSparseFipnn(hipStream_t stream, const void* const* input, T* output,
                           void* workspace, const int32_t fw_field_num,
                           const int32_t sample_feature_size, const int32_t field_num,
                           const int32_t field_neuron_size, const int32_t batch_size) {
    const T* weight_tensor = static_cast<const T*>(input[0]);
    const int32_t* field_tensor = static_cast<const int32_t*>(input[1]);
    const int32_t* index_tensor = static_cast<const int32_t*>(input[2]);
    int32_t* sample_feature_start_addr = static_cast<int32_t*>(workspace);
    int32_t* sample_feature_end_addr = nullptr;
    int32_t* gmem_field_map = sample_feature_start_addr + batch_size + 1;
    T* gmem_cross_sum = reinterpret_cast<T*>(gmem_field_map + fw_field_num);
    T* gmem_common_cross_mean_sum = gmem_cross_sum + batch_size * (field_neuron_size * (field_num + 1) * fw_field_num);
    T* gmem_common_cross_square_sum = gmem_common_cross_mean_sum + field_neuron_size * field_num * fw_field_num;

    int32_t embedding_size = fw_field_num * (fw_field_num + 1) / 2;
    int32_t shared_mem_required_bytes = (fw_field_num * 2 + 1) * sizeof(int);
    dim3 block(32, 32);
    dim3 grid(batch_size);
    hipMemsetAsync(gmem_common_cross_mean_sum, 0,
                    sizeof(float) * (field_neuron_size * field_num * fw_field_num +
                                     field_neuron_size * fw_field_num),
                    stream);
    hipMemsetAsync(gmem_field_map, -1, sizeof(int) * (fw_field_num), stream);
    hipMemsetAsync(sample_feature_start_addr, 0, sizeof(int) * (batch_size + 1), stream);
    hipMemsetAsync(output, 0, sizeof(float) * (batch_size * embedding_size), stream);

    ComputeBatchBoundary<<<DIVUP(sample_feature_size, 1024), 1024, 0, stream>>>(
        index_tensor, sample_feature_size, batch_size, sample_feature_start_addr,
        sample_feature_end_addr);

    dim3 grid0(1);
    ProcessCommonPart<T, 32><<<grid0, block, 0, stream>>>(
        field_neuron_size, field_num, fw_field_num, sample_feature_start_addr, weight_tensor,
        field_tensor, gmem_common_cross_mean_sum, gmem_common_cross_square_sum, gmem_field_map);
    dim3 block_set(field_neuron_size);
    dim3 grid_set(batch_size * fw_field_num);
    BroadcastCommonPart<T>
        <<<grid_set, block_set, 0, stream>>>(
        batch_size, field_neuron_size, field_num, fw_field_num, gmem_common_cross_mean_sum,
        gmem_common_cross_square_sum, gmem_cross_sum);

    SparseFIPNNGpu<T, 32><<<grid, block, shared_mem_required_bytes, stream>>>(
        field_neuron_size, field_num, fw_field_num, sample_feature_start_addr,
        sample_feature_end_addr, gmem_common_cross_mean_sum, gmem_common_cross_square_sum, gmem_field_map,
        reinterpret_cast<const T*>(weight_tensor), field_tensor, output, gmem_cross_sum);

    return 1;
}

template int32_t ComputeSparseFipnn(hipStream_t stream, const void* const* input, float* output,
                                    void* workspace, const int32_t fw_field_num,
                                    const int32_t sample_feature_size, const int32_t field_num,
                                    const int32_t field_neuron_size, const int32_t batch_size);

template int32_t ComputeSparseFipnn(hipStream_t stream, const void* const* input, half* output,
                                    void* workspace, const int32_t fw_field_num,
                                    const int32_t sample_feature_size, const int32_t field_num,
                                    const int32_t field_neuron_size, const int32_t batch_size);

}  // namespace functor
}  // namespace sparse_fipnn
}  // namespace nvinfer1
